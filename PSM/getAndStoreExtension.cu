#include "hip/hip_runtime.h"
#ifndef __HIPCC__  
    #define __HIPCC__
#endif
#include <hip/device_functions.h>
#include "getAndStoreExtension.h"


__global__ void kernelPrintExtention(Extension *d_Extension,unsigned int n){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if (i<n){
		__syncthreads();
		printf("\n[%d]: DFS code:(%d,%d,%d,%d,%d)  (vgi,vgj):(%d,%d)\n",i,d_Extension[i].vi,d_Extension[i].vj,d_Extension[i].li,d_Extension[i].lij,d_Extension[i].lj,d_Extension[i].vgi,d_Extension[i].vgj);
	}

}


__global__ void kernelGetAndStoreExtension(int *d_O,int *d_LO,unsigned int numberOfElementd_O,int *d_N,int *d_LN,unsigned int numberOfElementd_N,Extension *d_Extension){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<numberOfElementd_O){
		if (d_O[i]!=-1){
			int j;
			int ek;
			//printf("\nThread:%d",i);	
			for(j=i+1;j<numberOfElementd_O;++j){					
				if(d_O[j]!=-1) {break;}				
			}			
			
			if (j==numberOfElementd_O) {
				ek=numberOfElementd_N;
			}
			else
			{
				ek=d_O[j];
			}
			//printf("\n[%d]:%d",i,ek);
			for(int k=d_O[i];k<ek;k++){
				//do something
				int index= k;
				d_Extension[index].vi=0;
				d_Extension[index].vj=0;
				d_Extension[index].li=d_LO[i];
				d_Extension[index].lij=d_LN[k];
				d_Extension[index].lj=d_LO[d_N[k]];
				d_Extension[index].vgi=i;
				d_Extension[index].vgj=d_N[k];
				//printf("\n[%d]:%d",i,index);
				/*printf("\n[%d]: DFS code:(%d,%d,%d,%d,%d)  (vgi,vgj):(%d,%d)\n",k,d_Extension[i].vi,d_Extension[i].vj,d_Extension[i].li,
					d_Extension[i].lij,d_Extension[i].lj,d_Extension[i].vgi,d_Extension[i].vgj);*/
			}
		}
	}
}


hipError_t getAndStoreExtension(Extension *d_Extension,int *d_O,int *d_LO,unsigned int numberOfElementd_O,int *d_N,int *d_LN,unsigned int numberOfElementd_N,unsigned int Le,unsigned int Lv){

	hipError_t cudaStatus;
	dim3 block(1024);
	dim3 grid((numberOfElementd_O+block.x-1)/block.x);

	

	kernelGetAndStoreExtension<<<grid,block>>>(d_O,d_LO,numberOfElementd_O,d_N,d_LN,numberOfElementd_N,d_Extension);
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize kernelGetAndStoreExtension failed",cudaStatus);
		goto labelError;
	}

	
	kernelPrintExtention<<<((numberOfElementd_N+block.x-1)/block.x),block>>>(d_Extension,numberOfElementd_N); //Số lượng phần tử của d_Extension bằng số lượng phần tử của d_N nhưng chúng có kích thước khác nhau vì mỗi phần tử của d_Extension là một cấu trúc trong khi d_N là một số int.
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize kernelPrintExtention failed",cudaStatus);
		goto labelError;
	}

labelError:

	return cudaStatus;
}
