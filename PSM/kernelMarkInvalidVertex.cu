#include "hip/hip_runtime.h"
#include "kernelMarkInvalidVertex.h"

__global__ void kernelMarkInvalidVertex(int *d_O,int *LO,unsigned int sizeLO,int *d_labelAmount,unsigned int sizeLabelAmount,unsigned int minsup){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if(i<sizeLabelAmount){
		if(d_labelAmount[i]<minsup){
			for (int j=0;j<sizeLO;++j){
				if(LO[j]==i) d_O[j]=-1;
			}
		}
	}
}