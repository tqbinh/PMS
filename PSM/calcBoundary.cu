#include "hip/hip_runtime.h"
#include "calcBoundary.h"

__global__ void kernelCalcBoundary(Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,int *d_B,unsigned int maxOfVer){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<noElem_d_ValidExtension-1){
		unsigned int graphIdAfter=d_ValidExtension[i+1].vgi/maxOfVer;
		unsigned int graphIdCurrent=d_ValidExtension[i].vgi/maxOfVer;
		unsigned int resultDiff=graphIdAfter-graphIdCurrent;
		d_B[i]=resultDiff;
	}

}


hipError_t calcBoundary(Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,int *d_B,unsigned int maxOfVer){
	hipError_t cudaStatus;

	dim3 block(1024);
	dim3 grid((noElem_d_ValidExtension+block.x)/block.x);

	kernelCalcBoundary<<<grid,block>>>(d_ValidExtension,noElem_d_ValidExtension,d_B,maxOfVer);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaDeviceSynchronize calcBoundary failed");
		goto Error;
	}

Error:
	return cudaStatus;
}
