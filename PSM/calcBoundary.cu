#include "hip/hip_runtime.h"
#include "calcBoundary.h"

__global__ void kernelCalcBoundary(Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,int *d_B,unsigned int Lv){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<noElem_d_ValidExtension-1){
		unsigned int graphIdAfter=d_ValidExtension[i+1].vgi/Lv;
		unsigned int graphIdCurrent=d_ValidExtension[i].vgi/Lv;
		unsigned int resultDiff=graphIdAfter-graphIdCurrent;
		d_B[i]=resultDiff;
	}

}


hipError_t calcBoundary(Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,int *d_B,unsigned int Lv){
	hipError_t cudaStatus;

	dim3 block(1024);
	dim3 grid((noElem_d_ValidExtension+block.x)/block.x);

	kernelCalcBoundary<<<grid,block>>>(d_ValidExtension,noElem_d_ValidExtension,d_B,Lv);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaDeviceSynchronize calcBoundary failed");
		goto Error;
	}

Error:
	return cudaStatus;
}
