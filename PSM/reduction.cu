#include "hip/hip_runtime.h"
#include "reduction.h"
#include "kernelPrintf.h"

#define funcCheck(stmt) {                                            \
    hipError_t err = stmt;                                          \
    if (err != hipSuccess)                                          \
    {                                                                \
        printf( "Failed to run stmt %d ", __LINE__);                 \
        printf( "Got CUDA error ...  %s ", hipGetErrorString(err)); \
        return cudaStatus;                                                   \
    }                                                                \
}

__global__  void total(float * input, float * output, int len) 
{
	// Load a segment of the input vector into shared memory
	__shared__ float partialSum[2*BLOCK_SIZE];
	int globalThreadId = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int t = threadIdx.x;
	unsigned int start = 2*blockIdx.x*blockDim.x;

	if ((start + t) < len)
	{
		partialSum[t] = input[start + t];      
	}
	else
	{       
		partialSum[t] = 0.0;
	}
	if ((start + blockDim.x + t) < len)
	{   
		partialSum[blockDim.x + t] = input[start + blockDim.x + t];
	}
	else
	{
		partialSum[blockDim.x + t] = 0.0;
	}

	// Traverse reduction tree
	for (unsigned int stride = blockDim.x; stride > 0; stride /= 2)
	{
		__syncthreads();
		if (t < stride)
			partialSum[t] += partialSum[t + stride];
	}
	__syncthreads();

	// Write the computed sum of the block to the output vector at correct index
	if (t == 0 && (globalThreadId*2) < len)
	{
		output[blockIdx.x] = partialSum[t];
	}
}


hipError_t reduction(float *deviceInput,int len,float &support){
	hipError_t cudaStatus;
	
	
    float * deviceOutput;

	int numInputElements = len; // number of elements in the input list
	int numOutputElements; // number of elements in the output list

	numOutputElements = numInputElements / (BLOCK_SIZE<<1);
	if (numInputElements % (BLOCK_SIZE<<1)) 
	{
		numOutputElements++;
	}
		
    funcCheck(hipMalloc((void **)&deviceOutput, numOutputElements * sizeof(float)));

	// Initialize the grid and block dimensions here
    dim3 DimGrid( numOutputElements, 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);

    // Launch the GPU Kernel here
    total<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, numInputElements);
	
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaDeviceSynchronize() reduction failed");
		goto Error;
	}
	printf("\n");
	printFloat(deviceOutput,numOutputElements);
	hipMemcpy(&support,deviceOutput,numOutputElements*sizeof(float),hipMemcpyDeviceToHost);

Error:
	return cudaStatus;
}
