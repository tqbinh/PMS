#include "hip/hip_runtime.h"
#include "kernelaccess_d_LO_from_idx_of_d_O.h"
__device__ void __syncthreads(void);

inline __global__ void kernelaccess_d_LO_from_idx_of_d_O(int *d_LO,int *d_N,int sizeOfArrayN){
	int i=threadIdx.x + blockDim.x*blockIdx.x;
	if(i<sizeOfArrayN){		
			printf("d_n[%d]:%d ",d_N[i],d_LO[d_N[i]]);		
	}
	__syncthreads();
	
}