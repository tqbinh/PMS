#include "hip/hip_runtime.h"
#include "extractUniqueEdge.h"
/*
	1. Tạo |d_N| threads mỗi threads tương ứng với 1 cạnh trong cơ sở dữ liệu
	2. Threads sẽ set value =1 tại vị trí tương ứng (Lij*Lv + Lj)
	note: nếu O[i]=-1 thì xem như đỉnh không hợp lệ, chúng ta bỏ qua cạnh liên quan đến đỉnh này.

*/

__global__ void kernelPrintFloat(float* A,int n){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if (i<n){
		printf("[%d]:%.0f ;",i,A[i]);
	}

}


extern "C" inline hipError_t extractUniqueEdge(int *d_O,int *d_LO,unsigned int sizeOfArrayO,int *d_N,int *d_LN, unsigned int sizeOfArrayN,
												int *d_singlePattern,unsigned int numberOfElementd_singlePattern,unsigned int Lv,unsigned int Le){
	hipError_t cudaStatus;
	//calculate block and grid
	printf("\nd_O:");
	kernelPrintf<<<1,32>>>(d_O,sizeOfArrayO);
	hipDeviceSynchronize();
	printf("\nd_N:");
	kernelPrintf<<<1,32>>>(d_N,sizeOfArrayN);
	hipDeviceSynchronize();
		printf("\nd_singlePattern:");
	kernelPrintf<<<1,512>>>(d_singlePattern,numberOfElementd_singlePattern);
	hipDeviceSynchronize();

	dim3 block(512);
	dim3 grid((sizeOfArrayO+block.x-1)/block.x);

	kernelExtractUniqueEdge<<<grid,block>>>(d_O,d_LO,sizeOfArrayO,d_N,d_LN,sizeOfArrayN,d_singlePattern,Lv,Le);
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"cudaDeviceSynchornize failed",cudaStatus);
		goto Error;
	}
	printf("\nElements of d_singlePattern: ");
	kernelPrintf<<<grid,block>>>(d_singlePattern,numberOfElementd_singlePattern);
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"cudaDeviceSynchornize failed",cudaStatus);
		goto Error;
	}

	//Khởi tạo biến g_odata để chứa dữ liệu kết quả của prescan
	//g_odata có kích thước bằng với kích thước của d_singlePattern
	//ban đầu g_odata chứa giá trị rác, sau khi thực thi xong prescan thì kết quả sẽ được cập nhật vào g_odata
	float *d_singlePatternFloat=NULL;
	hipMalloc((int**)&d_singlePatternFloat,numberOfElementd_singlePattern);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipMalloc g_odata failed",cudaStatus);
		return cudaStatus;
	}
	

	kernelCastingInt2Float<<<grid,block>>>(d_singlePatternFloat,d_singlePattern,numberOfElementd_singlePattern);
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"cudaDeviceSynchornize kernelCastingInt2Float failed",cudaStatus);
		goto Error;
	}
	printf("\n\nd_singlePatternFloat:");
	kernelPrintFloat<<<grid,block>>>(d_singlePatternFloat,numberOfElementd_singlePattern);

	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"cudaDeviceSynchornize kernelPrintFloat failed",cudaStatus);
		goto Error;
	}
	float *g_odata=NULL;
	hipMalloc((int**)&g_odata,numberOfElementd_singlePattern);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipMalloc g_odata failed",cudaStatus);
		return cudaStatus;
	}

	preallocBlockSums(numberOfElementd_singlePattern);


	prescanArray(g_odata,d_singlePatternFloat, numberOfElementd_singlePattern);
	
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"cudaDeviceSynchornize kernelCastingInt2Float failed",cudaStatus);
		goto Error;
	}
	
	printf("\n\n result:");
	kernelPrintFloat<<<grid,block>>>(g_odata,numberOfElementd_singlePattern);

	//kernelCastingFloat2Int<<<grid,block>>>(d_singlePattern,g_odata,numberOfElementd_singlePattern);
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"cudaDeviceSynchornize kernelCastingInt2Float failed",cudaStatus);
		goto Error;
	}
	
	/*printf("\nElements of g_odata: ");
	kernelPrintf<<<grid,block>>>(d_singlePattern,numberOfElementd_singlePattern);
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"cudaDeviceSynchornize failed",cudaStatus);
		goto Error;
	}
*/
Error:
	hipFree(g_odata);
	return cudaStatus;
}
