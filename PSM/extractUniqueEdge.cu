#include "extractUniqueEdge.h"
/*
	1. Tạo |d_N| threads mỗi threads tương ứng với 1 cạnh trong cơ sở dữ liệu
	2. Threads sẽ set value =1 tại vị trí tương ứng (Lij*Lv + Lj)
	note: nếu O[i]=-1 thì xem như đỉnh không hợp lệ, chúng ta bỏ qua cạnh liên quan đến đỉnh này.
*/
extern "C" inline hipError_t extractUniqueEdge(int *d_O,int *d_LO,unsigned int sizeOfArrayO,int *d_N,int *d_LN, unsigned int sizeOfArrayN,
												int *d_singlePattern,unsigned int numberOfElementd_singlePattern,unsigned int Lv,unsigned int Le){
	hipError_t cudaStatus;
	//calculate block and grid
	printf("\nd_O:");
	kernelPrintf<<<1,32>>>(d_O,sizeOfArrayO);
	hipDeviceSynchronize();
	printf("\nd_N:");
	kernelPrintf<<<1,32>>>(d_N,sizeOfArrayN);
	hipDeviceSynchronize();
		printf("\nd_singlePattern:");
	kernelPrintf<<<1,512>>>(d_singlePattern,numberOfElementd_singlePattern);
	hipDeviceSynchronize();

	dim3 block(512);
	dim3 grid((sizeOfArrayO+block.x-1)/block.x);

	kernelExtractUniqueEdge<<<grid,block>>>(d_O,d_LO,sizeOfArrayO,d_N,d_LN,sizeOfArrayN,d_singlePattern,Lv,Le);
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"cudaDeviceSynchornize failed",cudaStatus);
		goto Error;
	}
	printf("\nElements of d_singlePattern: ");
	kernelPrintf<<<grid,block>>>(d_singlePattern,numberOfElementd_singlePattern);
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"cudaDeviceSynchornize failed",cudaStatus);
		goto Error;
	}

	//Khởi tạo biến g_odata để chứa dữ liệu kết quả của prescan
	//g_odata có kích thước bằng với kích thước của d_singlePattern
	//ban đầu g_odata chứa giá trị rác, sau khi thực thi xong prescan thì kết quả sẽ được cập nhật vào g_odata
	int *g_odata=NULL;
	hipMalloc((int**)&g_odata,numberOfElementd_singlePattern-1);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipMalloc g_odata failed",cudaStatus);
		return cudaStatus;
	}

	/*
	prescan<<<1,8>>>(g_odata,d_singlePattern,numberOfElementd_singlePattern-1);
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaDeviceSynchornize prescan failed\n",cudaStatus);
		goto Error;
	}*/
	int loop=4;
	
	scan_bel<<<1,numberOfElementd_singlePattern>>>(d_singlePattern,loop,g_odata,numberOfElementd_singlePattern);
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaDeviceSynchornize scan_bel failed\n",cudaStatus);
		goto Error;
	}

	printf("\nElements of g_odata: ");
	kernelPrintf<<<grid,block>>>(g_odata,numberOfElementd_singlePattern-1);
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"cudaDeviceSynchornize failed",cudaStatus);
		goto Error;
	}

Error:
	hipFree(g_odata);
	return cudaStatus;
}
