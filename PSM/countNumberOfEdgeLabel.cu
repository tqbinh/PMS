#include "countNumberOfEdgeLabel.h"

hipError_t countNumberOfEdgeLabel(int *d_LN,unsigned int sizeOfarrayLN,unsigned int &numberOfDifferentEdgeLabel){
	hipError_t cudaStatus=	countNumberOfLabelVetex(d_LN,sizeOfarrayLN,numberOfDifferentEdgeLabel);
	if (cudaStatus!= hipSuccess){
		fprintf(stderr,"countNumberOfLabelVetex in countNumberOfEdgeLabel fail",cudaStatus);
		goto Error;
	}
Error:
	return cudaStatus;

}
