#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <string>
#include <map>
#include "conio.h"
#include "kernelPrintf.h"
#include "gspan.h"
#include "kernelCountLabelInGraphDB.h"
#include "kernelMarkInvalidVertex.h"
#include "markInvalidVertex.h"
#include "checkArray.h"
#include "displayArray.h"
#include "checkDataBetweenHostAndGPU.h"
#include "access_d_LO_from_idx_of_d_O.h"
#include "countNumberOfLabelVetex.h"
#include "countNumberOfEdgeLabel.h"
#include "extractUniqueEdge.h"
#include "ExtensionStructure.h"
#include "getAndStoreExtension.h"
#include "validEdge.h"
#include "scanV.h"
#include "getLastElement.h"
#include "getValidExtension.h"
#include "getUniqueExtension.h"
#include "calcLabelAndStoreUniqueExtension.h"
#include "calcBoundary.h"
#include "calcSupport.h"
//#include <thrust\device_vector.h>
//#include <thrust\host_vector.h>	
using namespace std;

#define CHECK(call) \
{ \
const hipError_t error = call; \
if (error != hipSuccess) \
{ \
printf("Error: %s:%d, ", __FILE__, __LINE__); \
printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
exit(1); \
} \
}

//declare prototype
//void displayArray(int*,const unsigned int);
//bool checkArray(int*, int*, const int);
//__device__ void __syncthreads(void);

int main(int argc, char * const  argv[])
{	



	//*************************** Load Graph database with some parameters ***********************

	//unsigned int minsup = 34;
	unsigned int minsup = 2;
	unsigned int maxpat = 2;
	//unsigned int maxpat = 0x00000000;
	unsigned int minnodes = 0;
	bool where = true;
	bool enc = false;
	bool directed = false;

	//int opt;
	char* fname;
	//fname = "Klesscus";
	fname = "Klessorigin";
	//fname = "KlessoriginCust1";
	//fname= "G0G1G2_custom";
	

	gSpan gspan;	
	ofstream fout("result.txt");

	//Chuyển dữ liệu từ fname sang TRANS
	gspan.run(fname,fout,minsup,maxpat,minnodes,enc,where,directed);


	unsigned int maxOfVer;
	unsigned int numberOfGraph;
	maxOfVer=gspan.findMaxVertices();
	numberOfGraph=gspan.noGraphs();
	int sizeOfarrayO=maxOfVer*numberOfGraph;

	//printf("\nMaximun number of vertices: %d",maxOfVer);

	int* arrayO = new int[sizeOfarrayO]; //Tạo mảng arrayO có kích thước D*m
	if(arrayO==NULL){
		printf("\n!!!Memory Problem ArrayO");
		exit(1);
	}else{
		memset(arrayO, -1, sizeOfarrayO*sizeof(int)); // gán giá trị cho các phần tử mảng bằng -1
	}

	unsigned int noDeg; //Tổng bậc của tất cả các đỉnh trong csdl đồ thị TRANS
	noDeg = gspan.sumOfDeg();
	//cout<<noDeg;
	unsigned int sizeOfArrayN=noDeg;
	int* arrayN = new int[sizeOfArrayN]; //Mảng arrayN lưu trữ id của các đỉnh kề với đỉnh tương ứng trong mảng arrayO.
	if(arrayN==NULL){ //kiểm tra cấp phát bộ nhớ cho mảng có thành công hay không
		printf("\n!!!Memory Problem ArrayN");
		exit(1);
	}else
	{
		memset(arrayN, -1, noDeg*sizeof(int));
	}


	int* arrayLO = new int[sizeOfarrayO]; //Mảng arrayLO lưu trữ label cho tất cả các đỉnh trong TRANS.
	if(arrayLO==NULL){ //kiểm tra cấp phát bộ nhớ cho mảng có thành công hay không
		printf("\n!!!Memory Problem ArrayLO");
		exit(1);
	}else
	{
		memset(arrayLO, -1, sizeOfarrayO*sizeof(int));
	}


	int* arrayLN = new int[noDeg]; //Mảng arrayLN lưu trữ label của tất cả các cạnh trong TRANS
	if(arrayLN==NULL){ //kiểm tra cấp phát bộ nhớ cho mảng có thành công hay không
		printf("\n!!!Memory Problem ArrayLN");
		exit(1);
	}else
	{
		memset(arrayLN, -1, noDeg*sizeof(int));
	}


	gspan.importDataToArray(arrayO,arrayLO,arrayN,arrayLN,sizeOfarrayO,noDeg,maxOfVer);

	cout<<"ArrayO:";
	displayArray(arrayO,sizeOfarrayO);
	cout<<"\nArrayLO:";
	displayArray(arrayLO,sizeOfarrayO);
	cout<<"\nArrayN:";
	displayArray(arrayN,noDeg);
	cout<<"\nArrayLN:";
	displayArray(arrayLN,noDeg);

	//kích thước của dữ liệu
	size_t nBytesO = sizeOfarrayO*sizeof(int);
	size_t nBytesLO = sizeOfarrayO*sizeof(int);
	size_t nBytesN = noDeg*sizeof(int);
	size_t nBytesLN = noDeg*sizeof(int);


	//****cấp phát vùng nhớ trên GPU***
	//1. khai báo biến trên GPU
	int *d_O;
	int *d_LO;
	int *d_N; //Số lượng phần tử của d_N bằng noDeg
	int *d_LN;

	//2. Kiểm tra lỗi khi cấp phát
	//Khai báo biến cudaStatusAllocate
	hipError_t cudaStatusAllocate;

	cudaStatusAllocate =hipMalloc((int**) &d_O,nBytesO);
	if (cudaStatusAllocate!=hipSuccess){
		fprintf(stderr, "hipMalloc failed!");
		goto labelError;
	}

	cudaStatusAllocate =hipMalloc((int**) &d_LO,nBytesLO);
	if (cudaStatusAllocate!=hipSuccess){
		fprintf(stderr, "hipMalloc failed!");
		goto labelError;
	}
	cudaStatusAllocate =hipMalloc((int**) &d_N,nBytesN);
	if (cudaStatusAllocate!=hipSuccess){
		fprintf(stderr, "hipMalloc failed!");
		goto labelError;
	}
	cudaStatusAllocate =hipMalloc((int**) &d_LN,nBytesLN);
	if (cudaStatusAllocate!=hipSuccess){
		fprintf(stderr, "hipMalloc failed!");
		goto labelError;
	}


	//chép dữ liệu từ bốn mảng O,LO,N,LN từ Host sang GPU. Đây chính là CSDL đồ thị dùng để khai thác trên GPU
	cudaStatusAllocate = hipMemcpy(d_O,arrayO,nBytesO,hipMemcpyHostToDevice);
	if (cudaStatusAllocate != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto labelError;
	}

	cudaStatusAllocate = hipMemcpy(d_LO,arrayLO,nBytesLO,hipMemcpyHostToDevice);
	if (cudaStatusAllocate != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto labelError;
	}

	cudaStatusAllocate = hipMemcpy(d_N,arrayN,nBytesN,hipMemcpyHostToDevice);
	if (cudaStatusAllocate != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto labelError;
	}

	cudaStatusAllocate = hipMemcpy(d_LN,arrayLN,nBytesLN,hipMemcpyHostToDevice);
	if (cudaStatusAllocate != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto labelError;
	}


	//Đồng bộ đồng thời kiểm tra xem đồng bộ có lỗi không
	cudaStatusAllocate= hipDeviceSynchronize();
	if (cudaStatusAllocate != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatusAllocate);
		goto labelError;
	}
	//xác định grid and block structure
	dim3 block(512);
	dim3 grid((nBytesO + block.x -1)/block.x);
	printf("grid %d; block %d;\n",grid.x,block.x);


	//********kiểm tra đảm bảo dữ liệu ở GPU giống với Host********

	hipError_t cudaStatus = checkDataBetweenHostAndGPU(d_O,d_LO,d_N,d_LN,sizeOfarrayO,noDeg,arrayO,arrayLO,arrayN,arrayLN,nBytesO,nBytesLO,nBytesN,nBytesLN);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "checkDataBetweenHostAndGPU failed!");
		return 1;
	}
	printf("\n***********Press the Enter key to continous**********");
	getch();
	/*
	//********Đếm số đỉnh song song và loại nhỏ những đỉnh nhỏ hơn minsup****
	//Nếu số đỉnh nhỏ hơn minSup thì đánh dấu đỉnh đó là -1 trong mảng O và mảng LO và các cạnh liên quan đến đỉnh đó cũng được đánh dấu là -1

	cudaStatus = markInvalidVertex(d_O,d_LO,sizeOfarrayO,minsup);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "markInvalidVertex failed!");
		return 1;
	}
	*/
	//**********Đếm số nhãn đỉnh khác nhau trong CSDL đồ thị************
	//Nhãn đỉnh được lưu trữ trong mảng d_LO. Nhãn không hợp lệ mang giá trị -1
	//1.Cấp phát một mảng số nguyên có kích thước bằng với kích thước mảng d_LO gọi là d_Lv
	//2.Cấp phát |d_LO| threads
	//3.thread thứ i sẽ đọc giá trị nhãn tại vị trí d_LO[i], rồi ghi 1 vào mảng d_Lv[d_LO[i]]
	//4. Reduction mảng d_Lv để thu được các nhãn phân biệt
	unsigned int Lv=0;
	cudaStatus = countNumberOfLabelVetex(d_LO,sizeOfarrayO,Lv);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "countNumberOfLabelVetex failed!");
		return 1;
	}
	printf("\nNumber of different label Lv is: %d ;",Lv);
	
	//*******Đếm các loại cạnh khác nhau trong CSDL đồ thị************
	//Nhãn của tất cả các cạnh được lưu trữ trong mảng d_LN.
	/*
		1. Cấp phát một mảng số nguyên có kích thước băng với kích thước mảng d_LN gọi là d_Ln
		2. Cấp phát |d_Ln| threads
		3. Thread thứ i sẽ đọc giá trị nhãn tại vị trí d_Ln[i], rồi ghi vào mảng d_Ln[d_LN[i]]
		4. Reduction mảng d_Ln để thu được các loại cạnh phân biệt
	*/
	unsigned int Le=0;
	cudaStatus = countNumberOfEdgeLabel(d_LN,sizeOfArrayN,Le);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "countNumberOfLabelVetex failed!");
		return 1;
	}
	printf("\nNumber of different label Le is: %d ;",Le);
	/*******Thu thập tất cả các pattern P có 1 cạnh và tất cả các embeddings của P ********
	1. Duyệt qua cơ sở dữ liệu và rút trích các cạnh phân biệt dựa vào mảng d_O,d_LO,d_N và d_LN và 
	set giá trị 1 trong mảng d_SinglePattern có kích thước là (Lv.LE)
	tương ứng (lij.Lv+lj).
	2. Reduction d_singlePattern để biết số lượng pattern(numberOfPattern) là bao nhiêu. Sau đó cấp 
	phát numberOfPattern threads để tìm embeddings cho pattern đó.
	   Kết quả được lưu vào mảng cấu trúc d_Ext có số lượng phần tử bằng với d_N. 
	   Các thông tin cần lưu gồm:
		i. DFS Code của pattern theo cấu trúc (vi,vj,li,lij,lj)
	   ii. Lưu trữ các Embeddings của pattern(vig,vjg)
      iii. Row pointer trỏ đến heading của embedding ở cột cuối cùng Qp //
	3. Cấp phát mảng B có kích thước bằng với d_Ext để ghi nhận thông tin boundary của d_Ext
	4. Exclusive scan mảng B để ánh xạ lại graphid
	Tính support của từng pattern P
	5. Cấp phát mảng F có kích thước bằng với số lượng đồ thị trong CSDL
	6. Cấp phát numberOfPattern threads để cập nhật mảng F tương ứng với giá trị của kết quả scan trên B
	7. Reduction F để có độ support của từng pattern.
	*/

	//int *d_singlePattern=NULL;
	////size_t nBytesd_singlePattern = Lv*Le*sizeof(int);
	////Kích thước của mảng d_singlePattern là =[Lv!/(k!(Lv-k)!+n]*Le = [((Lv-2+1)*(Lv-2+2))/2 + Lv]*Le = [((Lv-1)*Lv)/2 + Lv]*Le
	////Trong trường hợp này k luôn = 2 vì cạnh có 2 đầu nhãn đỉnh lấy từ tập nhãn đỉnh phân biệt. 
	////Le là tập nhãn cạnh phân biệt trong CSDL
	////Vậy chúng ta có công thức cho kích thước của d_singlePattern như sau:
	//unsigned int numberOfElementd_singlePattern=(((Lv-1)*Lv)/2 +Lv)*Le;
	//size_t nBytesd_singlePattern = numberOfElementd_singlePattern*sizeof(int);
	//cudaStatus=hipMalloc((int**)&d_singlePattern,nBytesd_singlePattern);
	//if(cudaStatus!=hipSuccess){
	//	fprintf(stderr,"hipMalloc d_singlePattern failed", cudaStatus);
	//	return 1;
	//}
	//else
	//{
	//	hipMemset(d_singlePattern,0,nBytesd_singlePattern);
	//}
	//
	//cudaStatus = extractUniqueEdge(d_O,d_LO,sizeOfarrayO,d_N,d_LN,sizeOfArrayN,d_singlePattern,numberOfElementd_singlePattern,Lv,Le);
	//if(cudaStatus!=hipSuccess){
	//	fprintf(stderr,"call extractUniqueEdge failed",cudaStatus);
	//	return 1;
	//}


	//*******Nhãn truy xuất nhãn LO theo index từ d_N; tên hàm đặt bị nhầm
	/*cudaStatus = access_d_LO_from_idx_of_d_O(d_LO,d_N,sizeOfArrayN);
	if(cudaStatus != hipSuccess){
	fprintf(stderr, "access_d_LO_from_idx_of_d_N");
	return 1;
	}
	*/

	/* //May/04/2017: Trích các cạnh từ CSDL và lưu vào d_Extension.
		1. Tạo một cấu trúc Extension để lưu trữ các mở rộng: DFSCode của cạnh mở rộng (vi,vj,li,lij,lj),global from vertex id(vgi),
		global to vertex id (vgj) và pointer trỏ đến header của embedding tương ứng với cạnh mở rộng.
		2. Tạo một mảng có kích thước bằng với kích thước của d_N để lưu trữ các cạnh mở rộng ban đầu, lúc chưa có bất kỳ một cạnh
		phổ biến nào (P=0).
		3. Tạo một kernel có số lượng threads bằng d_O, mỗi thread sẽ xử lý một đỉnh. Nhiệm vụ của thread là đọc các cạnh kề với nó rồi
		lưu trữ thông tin vào mảng Extension tương ứng tại vị trí.
	*/

	int numberOfElementd_N=noDeg;
	size_t nBytesOfArrayExtension = numberOfElementd_N*sizeof(Extension);
	Extension *d_Extension;
	cudaStatus= hipMalloc((Extension**)&d_Extension,nBytesOfArrayExtension);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"CudaMalloc d_Extension fail",cudaStatus);
		exit(1);
	}


	int numberOfElementd_O=sizeOfarrayO;	
	cudaStatus = getAndStoreExtension(d_Extension,d_O,d_LO,numberOfElementd_O,d_N,d_LN,numberOfElementd_N,Le,Lv);
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize getAndStoreExtension failed",cudaStatus);
		return 1;
	}

	CHECK(printfExtension(d_Extension,numberOfElementd_N));

	/* //05-May-2017: Khởi tạo mảng V với giá trị của các phần tử ban đầu là 0, để lưu trữ những mở rộng hợp lệ.
	1. Mở rộng hợp lệ là mở rộng có Lj<=Li
	2. Mảng V có số lượng phần tử bằng với số lượng phần tử của mảng d_Extension
	3. Tạo kernel với số lượng threads bằng với số lượng phần tử của d_Extension
		Mỗi thread sẽ xử lý một phần tử trong d_Extension. Kiểm tra nếu Lj<=Li thì gán V tại vị trí tương ứng là 1
	*/

	int numberElementd_Extension = numberOfElementd_N;
	int *V;
	size_t nBytesV= numberElementd_Extension*sizeof(int);

	cudaStatus=hipMalloc((int**)&V,nBytesV);
	if (cudaStatus!= hipSuccess){
		fprintf(stderr,"hipMalloc array V failed",cudaStatus);
		exit(1);
	}
	else
	{
		hipMemset(V,0,nBytesV);
	}

	cudaStatus=validEdge(d_Extension,V,numberElementd_Extension);
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize validEdge failed",cudaStatus);
		return 1;
	}

	/* //07-May-2017: Extract unique Edge from d_Extension
	1. Tiếp theo, chúng ta exclusive scan mảng V để thu được index chỉ vị trí của các valid edge trong d_Extension.
	2. Input data: mảng V
	3. Output data: mảng index
	Mảng Index có số lượng phần tử bằng với mảng V
	*/

	int* index;
	cudaStatus=hipMalloc((int**)&index,numberElementd_Extension*sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"Cuda Malloc failed",cudaStatus);
		return 1;
	}	

	cudaStatus = scanV(V,numberElementd_Extension,index);
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize getAndStoreExtension failed",cudaStatus);
		return 1;
	}

	printf("\n Scan Result index: ");
	kernelPrintf<<<grid,block>>>(index,numberElementd_Extension);

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize kernelPrintf failed",cudaStatus);
		exit(1);
	}

/*	//Khởi tạo một mảng d_Unique có kích thước bằng với kích thước của giá trị của phần tử index cuối cùng vừa mới scan được.
	1. Hàm getLastElement sẽ trả về giá trị của phần tử cuối của mảng index
	2. Viết hàm để trích và lưu trữ các mở rộng hợp lệ
		a. khởi tạo mảng có kích thước bằng với kích thước của phần tử cuối của mảng index
		b. Rút trích các mở rộng hợp lệ từ d_Extension tương ứng tại vị trí V=1 vào index tương ứng.
*/
	//1. Hàm getLastElement
	int noElem_d_ValidExtension=0;
	cudaStatus=getLastElement(index,numberElementd_Extension,noElem_d_ValidExtension);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"getLastElement failed",cudaStatus);
		return 1;
	}
	noElem_d_ValidExtension++;
	//printf("\n\nnumberElementd_UniqueExtension:%d",noElem_d_ValidExtension);

	/* //08-May-2017: getValidExtension */
	//2.Hàm extractValidExtension: Trích và lưu trữ các mở rộng hợp lệ
	//2.1. Cấp phát bộ nhớ cho d_ValidExtension
	Extension *d_ValidExtension;
	cudaStatus=hipMalloc((Extension**)&d_ValidExtension,noElem_d_ValidExtension*sizeof(Extension));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"cudamalloc failed",cudaStatus);
		return 1;
	}
	else
	{
		hipMemset(d_ValidExtension,0,noElem_d_ValidExtension*sizeof(Extension));
	}

	
	hipDeviceSynchronize();
	
	cudaStatus=getValidExtension(d_Extension,V,index,numberElementd_Extension,d_ValidExtension);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"getValidExtension failed",cudaStatus);
		return 1;
	}

	printf("\nNumber Element of d_ValidExtension:%d",noElem_d_ValidExtension);
	CHECK(printfExtension(d_ValidExtension,noElem_d_ValidExtension));

	/* //Hàm getUniqueExtension: Trích tra các cạnh duy nhất dựa vào nhãn Li, Lj và Lij của edge extension
	1. Tạo mảng d_allPossibleExtension có kích thước là noElem_allPossibleExtension=Le*Lv*Lv để lưu trữ 
		tất cả các mở rộng có thể có của tất cả các đỉnh. Các mở rộng có thể có từ 1 đỉnh trên righ most path có kích thước là Le*Lv.
	2. Viết hàm getUniqueExtension để gán giá trị là 1 tại vị trí Li Lj tương ứng của extension.
	*/
	unsigned int noElem_allPossibleExtension=Le*Lv*Lv;
	int *d_allPossibleExtension;

	cudaStatus=hipMalloc((int**)&d_allPossibleExtension,noElem_allPossibleExtension*sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc d_allPossibleExtension failed",cudaStatus);
		return 1;
	}

	/* //09-May-2017 */
	
	cudaStatus=getUniqueExtension(d_ValidExtension,noElem_d_ValidExtension,Lv,Le,d_allPossibleExtension);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"getUniqueExtension failed",cudaStatus);
		return 1;
	}
	printf("\n\nLe:%d Lv:%d",Le,Lv);
	printf("\nd_allPossibleExtension: ");
	printInt(d_allPossibleExtension,noElem_allPossibleExtension);
	
	/* //Tiếp theo chúng ta exclusive scan mảng d_allPossibleExtension để thu được index phục vụ cho việc
		lưu trữ các unique extension.
	1. Chúng ta khởi tạo một mảng d_allPossibleExtensionScanResult có kích thước bằng với kích thước của d_allPossibleExtension
		đồng thời khởi tạo giá trị cho các phần tử của nó là 0.
	2. Sau khi thu được kết quả scan, chúng ta tạo một mảng Extension* d_UniqueExtension có kích thước bằng với kích thước của giá trị phần tử cuối cùng
		trong mảng d_allPossibleExtensionScanResult cộng với 1.
	3. Dựa vào giá trị index trong mảng d_allPossibleExtensionScanResult để suy ra các nhãn Li, Lj và Lij của Extension và lưu trữ chúng vào d_UniqueExtension
	*/
	
	int *d_allPossibleExtensionScanResult;
	cudaStatus=hipMalloc((int**)&d_allPossibleExtensionScanResult,noElem_allPossibleExtension*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc d_allPossibleExtensionScanResult failed");
		return 1;
	}
	else
	{
		hipMemset(d_allPossibleExtensionScanResult,0,noElem_allPossibleExtension*sizeof(int));
	}

	cudaStatus=scanV(d_allPossibleExtension,noElem_allPossibleExtension,d_allPossibleExtensionScanResult);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\nscanV function failed",cudaStatus);
		return 1;
	}

	printf("\n\nd_allPossibleExtensionScanResult:\n");
	CHECK(printInt(d_allPossibleExtensionScanResult,noElem_allPossibleExtension));

	//Lấy giá trị của phần tử cuối cùng trong mảng d_allPossibleExtensionScanResult
	int noElem_d_UniqueExtension=0;

	cudaStatus=getLastElement(d_allPossibleExtensionScanResult,noElem_allPossibleExtension,noElem_d_UniqueExtension);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"getLastElement failed",cudaStatus);
		return 1;
	}
	//noElem_d_UniqueExtension++;
	//printf("\n\nnoElem_d_UniqueExtension:%d",noElem_d_UniqueExtension);
	Extension *d_UniqueExtension;
	cudaStatus=hipMalloc((Extension**)&d_UniqueExtension,noElem_d_UniqueExtension*sizeof(Extension));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"hipMalloc d_UniqueExtension failed",cudaStatus);
		return 1;
	}
	else
	{
		hipMemset(d_UniqueExtension,0,noElem_d_UniqueExtension*sizeof(Extension));
	}


	cudaStatus=calcLabelAndStoreUniqueExtension(d_allPossibleExtension,d_allPossibleExtensionScanResult,noElem_allPossibleExtension,d_UniqueExtension,noElem_d_UniqueExtension,Le,Lv);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n\ncalcLabelAndStoreUniqueExtension function failed",cudaStatus);
		return 1;
	}

	printf("\n\nUnique Extension:");
	printfExtension(d_UniqueExtension,noElem_d_UniqueExtension);

	/* //10-May-2017: Tính độ hỗ trợ
	1. Trước tiên, chúng ta cấp phát một mảng d_B, mảng này có số lượng phần tử bằng với số lượng phần tử của d_ValidExtension
		Mảng d_B dùng để đánh dấu vị trí biên (boundary: nơi tiếp giáp giữa 2 đồ thị)
	2. Exclusive scan mảng d_B và lưu kế quả vào d_scanB_result
	3. Khởi tạo mảng d_F có số lượng phần tử bằng với giá trị của phần tử cuối cùng của mảng d_scanB_Result cộng 1
	4. Tính độ hỗ trợ của từng phần tử trong mảng d_UniqueExtension dựa vào d_ValidExtension và ScanB_Result
	*/
	
	//1. Cấp phát một mảng d_B và gán các giá trị 0 cho mọi phần tử của d_B
	unsigned int noElement_d_B=noElem_d_ValidExtension;
	int* d_B;
	cudaStatus=hipMalloc((int**)&d_B,noElement_d_B*sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipMalloc d_B failed",cudaStatus);
		return 1;
	}
	else
	{
		hipMemset(d_B,0,noElement_d_B*sizeof(int));
	}

	cudaStatus=calcBoundary(d_ValidExtension,noElem_d_ValidExtension,d_B,Lv);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"calcBoundary function failed",cudaStatus);
		return 1;
	}

	printf("\n\nd_B:\n");
	printInt(d_B,noElement_d_B);

	//2. Exclusive Scan mảng d_B
	int* d_scanB_Result;
	cudaStatus=hipMalloc((int**)&d_scanB_Result,noElement_d_B*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"hipMalloc d_scanB_Result failed",cudaStatus);
		return 1;
	}
	else
	{
		hipMemset(d_scanB_Result,0,noElement_d_B*sizeof(int));
	}

	cudaStatus=scanV(d_B,noElement_d_B,d_scanB_Result);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\nscanB function failed",cudaStatus);
		return 1;
	}

	printf("\n\nd_scanB_Result:\n");
	printInt(d_scanB_Result,noElement_d_B);

	//3. Tính độ hỗ trợ cho các mở rộng trong d_UniqueExtension
	//3.1 Tạo mảng d_F có số lượng phần tử bằng với giá trị cuối cùng của mảng d_scanB_Result cộng 1 và gán giá trị 0 cho các phần tử.
	int noElement_F=0;
	cudaStatus=getLastElement(d_scanB_Result,noElement_d_B,noElement_F);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ngetLastElement function failed",cudaStatus);
		return 1;
	}

	noElement_F++;
	printf("\nnoElement_F:%d",noElement_F);

	float *d_F;
	cudaStatus=hipMalloc((int**)&d_F,noElement_F*sizeof(float));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc d_F failed",cudaStatus);
		return 1;
	}
	else
	{
		hipMemset(d_F,0,noElement_F*sizeof(float));
	}
		

	/* //Gọi hàm calcSupport để tính độ hỗ trợ cho các mở rộng trong mảng d_UniqueExtension đồng thời gọi hàm buildEmbedding để xây dựng embedding cho mở rộng thoả minsup*/
	cudaStatus=calcSupport(d_UniqueExtension,noElem_d_UniqueExtension,d_ValidExtension,noElem_d_ValidExtension,d_scanB_Result,d_F,noElement_F,minsup,d_O,d_LO,numberOfElementd_O,d_N,d_LN,numberOfElementd_N,Lv,Le,maxOfVer,numberOfGraph,noDeg);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncalcSupport function failed",cudaStatus);
		return 1;
	}



labelError:
	//giải phóng vùng nhớ của dữ liệu
	hipFree(d_O);
	hipFree(d_LO);
	hipFree(d_N);
	hipFree(d_LN);	
//	hipFree(d_singlePattern);
	hipFree(d_Extension);
	hipFree(V);
	hipFree(d_ValidExtension);	
	hipFree(d_UniqueExtension);
	hipFree(d_B);


	hipDeviceReset();	

	fout.close();
	//delete[] arrayO;
	delete[] arrayN;
	delete[] arrayLO;
	delete[] arrayLN;

	getch();
	return 0;
}
