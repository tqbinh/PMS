#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <vector>
#include <iostream>
#include <string>
#include <map>
#include "conio.h"
#include "kernelPrintf.h"
#include "gspan.h"
#include "kernelCountLabelInGraphDB.h"
#include "kernelMarkInvalidVertex.h"
#include "markInvalidVertex.h"
#include "checkArray.h"
#include "displayArray.h"
#include "checkDataBetweenHostAndGPU.h"
#include "access_d_LO_from_idx_of_d_O.h"
#include "countNumberOfLabelVetex.h"
#include "countNumberOfEdgeLabel.h"
#include "extractUniqueEdge.h"
#include "ExtensionStructure.h"
#include "getAndStoreExtension.h"
#include "validEdge.h"
#include "scanV.h"
#include "getLastElement.h"
#include "getValidExtension.h"
#include "getUniqueExtension.h"
#include "calcLabelAndStoreUniqueExtension.h"
#include "calcBoundary.h"
#include "calcSupport.h"
#include "getSatisfyEdge.h"
#include "header.h"

//#include <thrust\device_vector.h>
//#include <thrust\host_vector.h>	
using namespace std;

#define blocksize 512

#define CHECK(call) \
{ \
const hipError_t error = call; \
if (error != hipSuccess) \
{ \
printf("Error: %s:%d, ", __FILE__, __LINE__); \
printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
exit(1); \
} \
}

//declare prototype
//void displayArray(int*,const unsigned int);
//bool checkArray(int*, int*, const int);
//__device__ void __syncthreads(void);

int main(int argc, char * const  argv[])
{	

#pragma region "Load data in to database. OutPut: d_O,d_LO,d_N,d_LN"

	//*************************** Load Graph database with some parameters ***********************

	//unsigned int minsup = 34;
	unsigned int minsup = 2;
	unsigned int maxpat = 2;
	//unsigned int maxpat = 0x00000000;
	unsigned int minnodes = 0;
	bool where = true;
	bool enc = false;
	bool directed = false;

	//int opt;
	char* fname;
	//fname = "Klesscus";
	fname = "Klessorigin";
	//fname = "KlessoriginCust1";
	//fname= "G0G1G2_custom";
	

	gSpan gspan;	
	ofstream fout("result.txt");

	//Chuyển dữ liệu từ fname sang TRANS
	gspan.run(fname,fout,minsup,maxpat,minnodes,enc,where,directed);


	unsigned int maxOfVer;
	unsigned int numberOfGraph;
	maxOfVer=gspan.findMaxVertices();
	numberOfGraph=gspan.noGraphs();
	int sizeOfarrayO=maxOfVer*numberOfGraph;

	//printf("\nMaximun number of vertices: %d",maxOfVer);

	int* arrayO = new int[sizeOfarrayO]; //Tạo mảng arrayO có kích thước D*m
	if(arrayO==NULL){
		printf("\n!!!Memory Problem ArrayO");
		exit(1);
	}else{
		memset(arrayO, -1, sizeOfarrayO*sizeof(int)); // gán giá trị cho các phần tử mảng bằng -1
	}

	unsigned int noDeg; //Tổng bậc của tất cả các đỉnh trong csdl đồ thị TRANS
	noDeg = gspan.sumOfDeg();
	//cout<<noDeg;
	unsigned int sizeOfArrayN=noDeg;
	int* arrayN = new int[sizeOfArrayN]; //Mảng arrayN lưu trữ id của các đỉnh kề với đỉnh tương ứng trong mảng arrayO.
	if(arrayN==NULL){ //kiểm tra cấp phát bộ nhớ cho mảng có thành công hay không
		printf("\n!!!Memory Problem ArrayN");
		exit(1);
	}else
	{
		memset(arrayN, -1, noDeg*sizeof(int));
	}


	int* arrayLO = new int[sizeOfarrayO]; //Mảng arrayLO lưu trữ label cho tất cả các đỉnh trong TRANS.
	if(arrayLO==NULL){ //kiểm tra cấp phát bộ nhớ cho mảng có thành công hay không
		printf("\n!!!Memory Problem ArrayLO");
		exit(1);
	}else
	{
		memset(arrayLO, -1, sizeOfarrayO*sizeof(int));
	}


	int* arrayLN = new int[noDeg]; //Mảng arrayLN lưu trữ label của tất cả các cạnh trong TRANS
	if(arrayLN==NULL){ //kiểm tra cấp phát bộ nhớ cho mảng có thành công hay không
		printf("\n!!!Memory Problem ArrayLN");
		exit(1);
	}else
	{
		memset(arrayLN, -1, noDeg*sizeof(int));
	}


	gspan.importDataToArray(arrayO,arrayLO,arrayN,arrayLN,sizeOfarrayO,noDeg,maxOfVer);

	cout<<"ArrayO:";
	displayArray(arrayO,sizeOfarrayO);
	cout<<"\nArrayLO:";
	displayArray(arrayLO,sizeOfarrayO);
	cout<<"\nArrayN:";
	displayArray(arrayN,noDeg);
	cout<<"\nArrayLN:";
	displayArray(arrayLN,noDeg);

	//kích thước của dữ liệu
	size_t nBytesO = sizeOfarrayO*sizeof(int);
	size_t nBytesLO = sizeOfarrayO*sizeof(int);
	size_t nBytesN = noDeg*sizeof(int);
	size_t nBytesLN = noDeg*sizeof(int);


	//****cấp phát vùng nhớ trên GPU***
	//1. khai báo biến trên GPU
	int *d_O;
	int *d_LO;
	int *d_N; //Số lượng phần tử của d_N bằng noDeg
	int *d_LN;

	//2. Kiểm tra lỗi khi cấp phát
	//Khai báo biến cudaStatusAllocate
	hipError_t cudaStatusAllocate;

	cudaStatusAllocate =hipMalloc((int**) &d_O,nBytesO);
	if (cudaStatusAllocate!=hipSuccess){
		fprintf(stderr, "hipMalloc failed!");
		//goto labelError;
		exit(1);
	}

	cudaStatusAllocate =hipMalloc((int**) &d_LO,nBytesLO);
	if (cudaStatusAllocate!=hipSuccess){
		fprintf(stderr, "hipMalloc failed!");
		//goto labelError;
		exit(1);
	}
	cudaStatusAllocate =hipMalloc((int**) &d_N,nBytesN);
	if (cudaStatusAllocate!=hipSuccess){
		fprintf(stderr, "hipMalloc failed!");
		//goto labelError;
		exit(1);
	}
	cudaStatusAllocate =hipMalloc((int**) &d_LN,nBytesLN);
	if (cudaStatusAllocate!=hipSuccess){
		fprintf(stderr, "hipMalloc failed!");
		//goto labelError;
		exit(1);
	}


	//chép dữ liệu từ bốn mảng O,LO,N,LN từ Host sang GPU. Đây chính là CSDL đồ thị dùng để khai thác trên GPU
	cudaStatusAllocate = hipMemcpy(d_O,arrayO,nBytesO,hipMemcpyHostToDevice);
	if (cudaStatusAllocate != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		//goto labelError;
		exit(1);
	}

	cudaStatusAllocate = hipMemcpy(d_LO,arrayLO,nBytesLO,hipMemcpyHostToDevice);
	if (cudaStatusAllocate != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		//goto labelError;
		exit(1);
	}

	cudaStatusAllocate = hipMemcpy(d_N,arrayN,nBytesN,hipMemcpyHostToDevice);
	if (cudaStatusAllocate != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		//goto labelError; 
		exit(1);
	}

	cudaStatusAllocate = hipMemcpy(d_LN,arrayLN,nBytesLN,hipMemcpyHostToDevice);
	if (cudaStatusAllocate != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		//goto labelError;
		exit(1);
	}


	//Đồng bộ đồng thời kiểm tra xem đồng bộ có lỗi không
	cudaStatusAllocate= hipDeviceSynchronize();
	if (cudaStatusAllocate != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatusAllocate);
		//goto labelError;
		exit(1);
	}
	//xác định grid and block structure
	dim3 block(blocksize);
	dim3 grid((nBytesO + block.x -1)/block.x);
	printf("grid %d; block %d;\n",grid.x,block.x);


	//********kiểm tra đảm bảo dữ liệu ở GPU giống với Host********

	hipError_t cudaStatus = checkDataBetweenHostAndGPU(d_O,d_LO,d_N,d_LN,sizeOfarrayO,noDeg,arrayO,arrayLO,arrayN,arrayLN,nBytesO,nBytesLO,nBytesN,nBytesLN);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "checkDataBetweenHostAndGPU failed!");
		return 1;
	}
	printf("\n***********Finished: Database has been copied from host to device. Next: count different label of vertex in all graph in database **********");
	printf("\n***********Press the Enter key to continous**********\n");
	getch();

#pragma endregion

	//don't use this snippet 

	/*
	//********Đếm số đỉnh song song và loại nhỏ những đỉnh nhỏ hơn minsup****
	//Nếu số đỉnh nhỏ hơn minSup thì đánh dấu đỉnh đó là -1 trong mảng O và mảng LO và các cạnh liên quan đến đỉnh đó cũng được đánh dấu là -1

	cudaStatus = markInvalidVertex(d_O,d_LO,sizeOfarrayO,minsup);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "markInvalidVertex failed!");
		return 1;
	}
	*/ 

#pragma region "Get Lv: Distinct vertex labels and Le: Distinct edge labels in graph database"
	//**********Đếm số nhãn đỉnh khác nhau trong CSDL đồ thị************
	//Nhãn đỉnh được lưu trữ trong mảng d_LO. Nhãn không hợp lệ mang giá trị -1
	//1.Cấp phát một mảng số nguyên có kích thước bằng với kích thước mảng d_LO gọi là d_Lv
	//2.Cấp phát |d_LO| threads
	//3.thread thứ i sẽ đọc giá trị nhãn tại vị trí d_LO[i], rồi ghi 1 vào mảng d_Lv[d_LO[i]]
	//4. Reduction mảng d_Lv để thu được các nhãn phân biệt
	unsigned int Lv=0;
	cudaStatus = countNumberOfLabelVetex(d_LO,sizeOfarrayO,Lv);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "countNumberOfLabelVetex failed!");
		return 1;
	}
	printf("\nNumber of different label Lv is: %d ;",Lv);
	printf("\n***********Finished: count different label of vertex in all graph in database. Next: count different label of edge in all graph in database **********");
	printf("\n***********Press the Enter key to continous**********\n");
	//getch();

	//*******Đếm các loại cạnh khác nhau trong CSDL đồ thị************
	//Nhãn của tất cả các cạnh được lưu trữ trong mảng d_LN.
	/*
		1. Cấp phát một mảng số nguyên có kích thước băng với kích thước mảng d_LN gọi là d_Ln
		2. Cấp phát |d_Ln| threads
		3. Thread thứ i sẽ đọc giá trị nhãn tại vị trí d_Ln[i], rồi ghi vào mảng d_Ln[d_LN[i]]
		4. Reduction mảng d_Ln để thu được các loại cạnh phân biệt
	*/
	unsigned int Le=0;
	cudaStatus = countNumberOfEdgeLabel(d_LN,sizeOfArrayN,Le);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "countNumberOfLabelVetex failed!");
		return 1;
	}
	printf("\nNumber of different label Le is: %d ;",Le);
	printf("\n***********Finished: count different label of edge in all graph in database. Next: get and store all single edge extension into d_Extension **********");
	printf("\n***********Press the Enter key to continous**********\n");
	//getch();

#pragma endregion

	//don't use this snippet

	/*******Thu thập tất cả các pattern P có 1 cạnh và tất cả các embeddings của P ********
	1. Duyệt qua cơ sở dữ liệu và rút trích các cạnh phân biệt dựa vào mảng d_O,d_LO,d_N và d_LN và 
	set giá trị 1 trong mảng d_SinglePattern có kích thước là (Lv.LE)
	tương ứng (lij.Lv+lj).
	2. Reduction d_singlePattern để biết số lượng pattern(numberOfPattern) là bao nhiêu. Sau đó cấp 
	phát numberOfPattern threads để tìm embeddings cho pattern đó.
	   Kết quả được lưu vào mảng cấu trúc d_Ext có số lượng phần tử bằng với d_N. 
	   Các thông tin cần lưu gồm:
		i. DFS Code của pattern theo cấu trúc (vi,vj,li,lij,lj)
	   ii. Lưu trữ các Embeddings của pattern(vig,vjg)
      iii. Row pointer trỏ đến heading của embedding ở cột cuối cùng Qp //
	3. Cấp phát mảng B có kích thước bằng với d_Ext để ghi nhận thông tin boundary của d_Ext
	4. Exclusive scan mảng B để ánh xạ lại graphid
	Tính support của từng pattern P
	5. Cấp phát mảng F có kích thước bằng với số lượng đồ thị trong CSDL
	6. Cấp phát numberOfPattern threads để cập nhật mảng F tương ứng với giá trị của kết quả scan trên B
	7. Reduction F để có độ support của từng pattern.
	*/


	//don't use this snippet

	//int *d_singlePattern=NULL;
	////size_t nBytesd_singlePattern = Lv*Le*sizeof(int);
	////Kích thước của mảng d_singlePattern là =[Lv!/(k!(Lv-k)!+n]*Le = [((Lv-2+1)*(Lv-2+2))/2 + Lv]*Le = [((Lv-1)*Lv)/2 + Lv]*Le
	////Trong trường hợp này k luôn = 2 vì cạnh có 2 đầu nhãn đỉnh lấy từ tập nhãn đỉnh phân biệt. 
	////Le là tập nhãn cạnh phân biệt trong CSDL
	////Vậy chúng ta có công thức cho kích thước của d_singlePattern như sau:
	//unsigned int numberOfElementd_singlePattern=(((Lv-1)*Lv)/2 +Lv)*Le;
	//size_t nBytesd_singlePattern = numberOfElementd_singlePattern*sizeof(int);
	//cudaStatus=hipMalloc((int**)&d_singlePattern,nBytesd_singlePattern);
	//if(cudaStatus!=hipSuccess){
	//	fprintf(stderr,"hipMalloc d_singlePattern failed", cudaStatus);
	//	return 1;
	//}
	//else
	//{
	//	hipMemset(d_singlePattern,0,nBytesd_singlePattern);
	//}
	//
	//cudaStatus = extractUniqueEdge(d_O,d_LO,sizeOfarrayO,d_N,d_LN,sizeOfArrayN,d_singlePattern,numberOfElementd_singlePattern,Lv,Le);
	//if(cudaStatus!=hipSuccess){
	//	fprintf(stderr,"call extractUniqueEdge failed",cudaStatus);
	//	return 1;
	//}


	//*******Nhãn truy xuất nhãn LO theo index từ d_N; tên hàm đặt bị nhầm
	/*cudaStatus = access_d_LO_from_idx_of_d_O(d_LO,d_N,sizeOfArrayN);
	if(cudaStatus != hipSuccess){
	fprintf(stderr, "access_d_LO_from_idx_of_d_N");
	return 1;
	}
	*/



#pragma region "Extract all edge in database and store them into d_Extension"
	//Giải thuật:
	/* //May/04/2017: Trích tất cả các cạnh từ CSDL và lưu vào d_Extension.
	Cách làm:
		1. Tạo một cấu trúc Extension để lưu trữ các mở rộng: DFSCode của cạnh mở rộng (vi,vj,li,lij,lj),global from vertex id(vgi),
		global to vertex id (vgj).
		2. Tạo một mảng có kích thước bằng với kích thước của d_N để lưu trữ các cạnh mở rộng ban đầu, lúc chưa có bất kỳ một cạnh
		phổ biến nào (P=0).
		3. Tạo một kernel có số lượng threads bằng d_O, mỗi thread sẽ xử lý một đỉnh. Nhiệm vụ của thread là đọc các cạnh kề với nó rồi
		lưu trữ thông tin vào mảng Extension tương ứng tại vị trí.
		
	*/
	

	//cấp phát bộ nhớ cho d_Extension
	int numberOfElementd_N=noDeg;
	size_t nBytesOfArrayExtension = numberOfElementd_N*sizeof(Extension);
	Extension *d_Extension;
	cudaStatus= hipMalloc((Extension**)&d_Extension,nBytesOfArrayExtension);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"CudaMalloc d_Extension fail",cudaStatus);
		exit(1);
	}

	//Trích tất cả các cạnh từ database rồi lưu vào d_Extension
	int numberOfElementd_O=sizeOfarrayO;	
	cudaStatus = getAndStoreExtension(d_Extension,d_O,d_LO,numberOfElementd_O,d_N,d_LN,numberOfElementd_N,Le,Lv);
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize getAndStoreExtension failed",cudaStatus);
		return 1;
	}

	CHECK(printfExtension(d_Extension,numberOfElementd_N));

	
	printf("\n***********Finished: get and store all single edge extension in to d_Extension. Next: set 1 for all valid single edge extension in V array **********");
	printf("\n***********Press the Enter key to continous**********\n");
	//getch();

#pragma endregion

#pragma region "Mark valid edge from d_Extension and set 1 in array V at the corresponding position"
	//Giải thuật:
	/* //05-May-2017: Khởi tạo mảng V với giá trị của các phần tử ban đầu là 0, để lưu trữ những mở rộng hợp lệ.
	1. Mở rộng hợp lệ là mở rộng có Lj<=Li
	2. Mảng V có số lượng phần tử bằng với số lượng phần tử của mảng d_Extension
	3. Tạo kernel với số lượng threads bằng với số lượng phần tử của d_Extension
		Mỗi thread sẽ xử lý một phần tử trong d_Extension. Kiểm tra nếu Lj<=Li thì gán V tại vị trí tương ứng là 1
	*/

	int numberElementd_Extension = numberOfElementd_N;
	int *V;
	size_t nBytesV= numberElementd_Extension*sizeof(int);

	cudaStatus=hipMalloc((int**)&V,nBytesV);
	if (cudaStatus!= hipSuccess){
		fprintf(stderr,"hipMalloc array V failed",cudaStatus);
		exit(1);
	}
	else
	{
		hipMemset(V,0,nBytesV);
	}

	cudaStatus=validEdge(d_Extension,V,numberElementd_Extension);
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize validEdge failed",cudaStatus);
		return 1;
	}

	
	printf("\n***********Finished: set 1 for all valid single edge extension in V array. Next: Extract all valid extension  from d_Extension base on V array **********");
	printf("\n***********Press the Enter key to continous**********\n");
	//getch();

#pragma endregion
	
#pragma region "Exclusive scan V array and then store scan result into index array"

	//Giải thuật:
	/* //07-May-2017: Extract unique Edge from d_Extension
	1. Tiếp theo, chúng ta exclusive scan mảng V để thu được index chỉ vị trí của các valid edge trong d_Extension.
	2. Input data: mảng V
	3. Output data: mảng index
	Mảng Index có số lượng phần tử bằng với mảng V
	*/

	//cấp phát bộ nhớ cho mảng index
	int* index;
	cudaStatus=hipMalloc((int**)&index,numberElementd_Extension*sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"Cuda Malloc failed",cudaStatus);
		return 1;
	}	
	//Exclusive scan mảng V và lưu kết quả scan vào mảng index
	cudaStatus = scanV(V,numberElementd_Extension,index);
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize getAndStoreExtension failed",cudaStatus);
		return 1;
	}
	//Hiển thị nội dung mảng index
	printf("\n Scan Result index: ");
	kernelPrintf<<<grid,block>>>(index,numberElementd_Extension);

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize kernelPrintf failed",cudaStatus);
		exit(1);
	}

#pragma endregion

#pragma region "Replying on the index array to extract the valid edges from d_Extension to d_ValidExtension"

/*	//Khởi tạo một mảng d_Unique có kích thước bằng với kích thước của giá trị của phần tử index cuối cùng vừa mới scan được.
	1. Hàm getLastElement sẽ trả về giá trị của phần tử cuối của mảng index
	2. Viết hàm để trích và lưu trữ các mở rộng hợp lệ
		a. khởi tạo mảng có kích thước bằng với kích thước của phần tử cuối của mảng index
		b. Rút trích các mở rộng hợp lệ từ d_Extension tương ứng tại vị trí V=1 vào index tương ứng.
*/
	//1. Hàm getLastElement
	int noElem_d_ValidExtension=0;
	cudaStatus=getLastElement(index,numberElementd_Extension,noElem_d_ValidExtension);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"getLastElement failed",cudaStatus);
		return 1;
	}
	noElem_d_ValidExtension++;
	//printf("\n\nnumberElementd_UniqueExtension:%d",noElem_d_ValidExtension);

	/* //08-May-2017: getValidExtension */
	//2.Hàm extractValidExtension: Trích và lưu trữ các mở rộng hợp lệ
	//2.1. Cấp phát bộ nhớ cho d_ValidExtension
	Extension *d_ValidExtension;
	cudaStatus=hipMalloc((Extension**)&d_ValidExtension,noElem_d_ValidExtension*sizeof(Extension));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"cudamalloc failed",cudaStatus);
		return 1;
	}
	else
	{
		hipMemset(d_ValidExtension,0,noElem_d_ValidExtension*sizeof(Extension));
	}

	
	hipDeviceSynchronize();
	
	//Trích những cạnh hợp lệ từ mảng d_Extension sang d_ValidExtension
	cudaStatus=getValidExtension(d_Extension,V,index,numberElementd_Extension,d_ValidExtension);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"getValidExtension failed",cudaStatus);
		return 1;
	}

	printf("\nNumber Element of d_ValidExtension:%d",noElem_d_ValidExtension);
	CHECK(printfExtension(d_ValidExtension,noElem_d_ValidExtension));


	printf("\n***********Finished: Extract all valid extension  from d_Extension base on V array and put result in d_ValidExtension. Next: Extract unique extension from d_ValidExtension base on label of vertex and label of edge **********");
	printf("\n***********Press the Enter key to continous**********\n");
	//getch();

#pragma endregion

#pragma region "Extract the unique edges from d_ValidExtension replying on their label. Note: d_allPossibleExtension"

	//Giải thuật
	/* //Hàm getUniqueExtension: Trích tra các cạnh duy nhất dựa vào nhãn Li, Lj và Lij của edge extension
	1. Tạo mảng d_allPossibleExtension có kích thước là noElem_allPossibleExtension=Le*Lv*Lv để lưu trữ 
		tất cả các mở rộng có thể có của tất cả các đỉnh. Các mở rộng có thể có từ 1 đỉnh trên righ most path có kích thước là Le*Lv.
	2. Viết hàm getUniqueExtension để gán giá trị là 1 tại vị trí Li Lj tương ứng của extension.
	*/
	//Tính số lượng tất cả các cạnh có thể có dựa vào nhãn của chúng
	unsigned int noElem_allPossibleExtension=Le*Lv*Lv;
	int *d_allPossibleExtension;

	//cấp phát bộ nhớ cho mảng d_allPossibleExtension
	cudaStatus=hipMalloc((int**)&d_allPossibleExtension,noElem_allPossibleExtension*sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc d_allPossibleExtension failed",cudaStatus);
		return 1;
	}

	/* //09-May-2017 */
	//Trích các cạnh duy nhất dựa vào nhãn cạnh
	cudaStatus=getUniqueExtension(d_ValidExtension,noElem_d_ValidExtension,Lv,Le,d_allPossibleExtension);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"getUniqueExtension failed",cudaStatus);
		return 1;
	}
	printf("\n\nLe:%d Lv:%d",Le,Lv);
	printf("\nd_allPossibleExtension: ");
	printInt(d_allPossibleExtension,noElem_allPossibleExtension);
	
	printf("\n***********Finished: Extract unique extension from d_ValidExtension base on label of vertex and label of edge and set 1 as result in d_allPossibleExtension array. Next: Mapping label of vertex and edge into edge and store them in d_UniqueExtension **********");
	printf("\n***********Press the Enter key to continous**********\n");
	//getch();

	/* //Tiếp theo chúng ta exclusive scan mảng d_allPossibleExtension để thu được index phục vụ cho việc
		lưu trữ các unique extension.
	1. Chúng ta khởi tạo một mảng d_allPossibleExtensionScanResult có kích thước bằng với kích thước của d_allPossibleExtension
		đồng thời khởi tạo giá trị cho các phần tử của nó là 0.
	2. Sau khi thu được kết quả scan, chúng ta tạo một mảng Extension* d_UniqueExtension có kích thước bằng với kích thước của giá trị phần tử cuối cùng
		trong mảng d_allPossibleExtensionScanResult cộng với 1.
	3. Dựa vào giá trị index trong mảng d_allPossibleExtensionScanResult để suy ra các nhãn Li, Lj và Lij của Extension và lưu trữ chúng vào d_UniqueExtension
	*/

	//Cấp phát bộ nhớ cho mảng d_allPossibleExtensionScanResult có kích thước bằng với mảng d_allPossibleExtension
	int *d_allPossibleExtensionScanResult;
	cudaStatus=hipMalloc((int**)&d_allPossibleExtensionScanResult,noElem_allPossibleExtension*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc d_allPossibleExtensionScanResult failed");
		return 1;
	}
	else
	{
		hipMemset(d_allPossibleExtensionScanResult,0,noElem_allPossibleExtension*sizeof(int));
	}

	//Exclusive scan mảng d_allPossibleExtension và lưu kết quả vào mảng d_allPossibleExtensionScanResult
	cudaStatus=scanV(d_allPossibleExtension,noElem_allPossibleExtension,d_allPossibleExtensionScanResult);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\nscanV function failed",cudaStatus);
		return 1;
	}

	printf("\n\nd_allPossibleExtensionScanResult:\n");
	CHECK(printInt(d_allPossibleExtensionScanResult,noElem_allPossibleExtension));

	/*	Tính kích thước cho mảng d_UniqueExtension
	*	Lấy giá trị của phần tử cuối cùng trong mảng d_allPossibleExtensionScanResult và lưu vào biến noElem_d_UniqueExtension
	*	Nếu phần tử cuối cùng của mảng d_allPossibleExtension có giá trị 1 thì phải tăng biến noElem_d_UniqueExtension lên 1
	*/
	int noElem_d_UniqueExtension=0;
	//Tính kích thước của mảng d_UniqueExtension dựa vào kết quả exclusive scan
	cudaStatus=getSizeBaseOnScanResult(d_allPossibleExtension,d_allPossibleExtensionScanResult,noElem_allPossibleExtension,noElem_d_UniqueExtension);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"getLastElement failed",cudaStatus);
		return 1;
	}
	// printf("\n\nnoElem_d_UniqueExtension:%d",noElem_d_UniqueExtension);

	//Tạo mảng d_UniqueExtension với kích thước mảng vừa tính được
	UniEdge *d_UniqueExtension;
	cudaStatus=hipMalloc((void**)&d_UniqueExtension,noElem_d_UniqueExtension*sizeof(UniEdge));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"hipMalloc d_UniqueExtension failed",cudaStatus);
		return 1;
	}
	else
	{
		hipMemset(d_UniqueExtension,0,noElem_d_UniqueExtension*sizeof(UniEdge));
	}

	//Ánh xạ ngược lại từ vị trí trong d_allPossibleExtension thành cạnh và lưu kết quả vào d_UniqueExtension
	cudaStatus=calcLabelAndStoreUniqueExtension(d_allPossibleExtension,d_allPossibleExtensionScanResult,noElem_allPossibleExtension,d_UniqueExtension,noElem_d_UniqueExtension,Le,Lv);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n\ncalcLabelAndStoreUniqueExtension function failed",cudaStatus);
		return 1;
	}

	printf("\n\nUnique Extension:");
	printfUniEdge(d_UniqueExtension,noElem_d_UniqueExtension);

	printf("\n***********Finished: Mapping label of vertex and edge into edge and store them in d_UniqueExtension . Next: compute support for valid unique extension in d_UniqueExtension **********");
	printf("\n***********Press the Enter key to continous**********\n");
	//getch();


#pragma endregion


#pragma region "Caculating support for each edge in d_UniqueExtension"
	
	//Giải thuật:
	/* //10-May-2017: Tính độ hỗ trợ
	1. Trước tiên, chúng ta cấp phát một mảng d_B, mảng này có số lượng phần tử bằng với số lượng phần tử của d_ValidExtension
		Mảng d_B dùng để đánh dấu vị trí biên (boundary: nơi tiếp giáp giữa 2 đồ thị)
	2. Exclusive scan mảng d_B và lưu kế quả vào d_scanB_result
	3. Khởi tạo mảng d_F có số lượng phần tử bằng với giá trị của phần tử cuối cùng của mảng d_scanB_Result cộng 1
	4. Tính độ hỗ trợ của từng phần tử trong mảng d_UniqueExtension dựa vào d_ValidExtension và ScanB_Result
	*/
	
	/* Xây dựng Boundary cho mảng d_ValidExtension */
	//1. Cấp phát một mảng d_B và gán các giá trị 0 cho mọi phần tử của d_B
	unsigned int noElement_d_B=noElem_d_ValidExtension;
	int* d_B;
	cudaStatus=hipMalloc((int**)&d_B,noElement_d_B*sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipMalloc d_B failed",cudaStatus);
		return 1;
	}
	else
	{
		hipMemset(d_B,0,noElement_d_B*sizeof(int));
	}

	//Gián giá trị boundary cho d_B
	cudaStatus=calcBoundary(d_ValidExtension,noElem_d_ValidExtension,d_B,maxOfVer);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"calcBoundary function failed",cudaStatus);
		return 1;
	}

	printf("\n\nd_B:\n");
	printInt(d_B,noElement_d_B);
	printf("\n***********Finished: set Boundary for d_ValidExtension . Next: compute support for valid unique extension in d_UniqueExtension **********");
	printf("\n***********Press the Enter key to continous**********\n");
	//getch();


	//2. Exclusive Scan mảng d_B
	int* d_scanB_Result;
	cudaStatus=hipMalloc((int**)&d_scanB_Result,noElement_d_B*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"hipMalloc d_scanB_Result failed",cudaStatus);
		return 1;
	}
	else
	{
		hipMemset(d_scanB_Result,0,noElement_d_B*sizeof(int));
	}

	cudaStatus=scanV(d_B,noElement_d_B,d_scanB_Result);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\nscanB function failed",cudaStatus);
		return 1;
	}

	printf("\n\nd_scanB_Result:\n");
	printInt(d_scanB_Result,noElement_d_B);

	//3. Tính độ hỗ trợ cho các mở rộng trong d_UniqueExtension
	//3.1 Tạo mảng d_F có số lượng phần tử bằng với giá trị cuối cùng của mảng d_scanB_Result cộng 1 và gán giá trị 0 cho các phần tử.
	int noElemF=0;
	cudaStatus=getLastElement(d_scanB_Result,noElement_d_B,noElemF);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ngetLastElement function failed",cudaStatus);
		return 1;
	}

	noElemF++;
	printf("\nnoElement_F:%d",noElemF);

	float *d_F;
	cudaStatus=hipMalloc((int**)&d_F,noElem_d_UniqueExtension*noElemF*sizeof(float));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc d_F failed",cudaStatus);
		return 1;
	}
	else
	{
		hipMemset(d_F,0,noElemF*sizeof(float));
	}
		
	
	/* //Gọi hàm calcSupport để tính độ hỗ trợ cho các mở rộng trong mảng d_UniqueExtension đồng thời gọi hàm buildEmbedding để xây dựng embedding cho mở rộng thoả minsup*/
	//Mở rộng nào phổ biến sẽ được ghi nhận lại vào mảng h_frequentEdge (Có số lượng phần tử bằng với d_uniqueExtension)
	//Tại vị trí tương ứng của cạnh lớn hơn bằng minsup sẽ được set là 1.
	/*int numberEle_h_frequentEdge=noElem_d_UniqueExtension; //không dùng h_frequentEdge để hi nhận những mở rộng thoả minSup
	int *h_frequentEdge = (int*) malloc(numberEle_h_frequentEdge*sizeof(int));
	if(h_frequentEdge==NULL){
		printf("\n Malloc array h_frequentEdge failed");
		exit(1);
	}
	else
	{
		memset(h_frequentEdge,0,numberEle_h_frequentEdge*sizeof(int));
	}*/
	//vector<int> h_satisfyEdge;
	//vector<int> h_satisfyEdgeSupport;
	
	//Hàm calcSupport tính độ hỗ trợ của tất cả các cạnh trong d_UniqueExtension
	//Nó  trả về vị trí index của d_UniqueExtension mà tại đó thoả minSup
	//Nó  trả về giá trị minSup
	//cudaStatus=calcSupport(d_UniqueExtension,noElem_d_UniqueExtension,d_ValidExtension,noElem_d_ValidExtension,d_scanB_Result,d_F,noElement_F,minsup,d_O,d_LO,numberOfElementd_O,d_N,d_LN,numberOfElementd_N,Lv,Le,maxOfVer,numberOfGraph,noDeg,h_satisfyEdge,h_satisfyEdgeSupport);
	//if (cudaStatus!=hipSuccess){
	//	fprintf(stderr,"\ncalcSupport function failed",cudaStatus);
	//	return 1;
	//}

	float *h_resultSup=nullptr;
	cudaStatus=computeSupport(d_UniqueExtension,noElem_d_UniqueExtension,d_ValidExtension,noElem_d_ValidExtension,d_scanB_Result,d_F,noElemF,h_resultSup);
	
	//In độ hỗ trợ cho các cạnh tương ứng trong mảng kết quả h_resultSup
	for (int i = 0; i < noElem_d_UniqueExtension; i++)
	{
		printf("\n resultSup[%d]:%.1f",i,h_resultSup[i]);
	}
	
#pragma endregion

#pragma region "count number of edge in each graph in database"

	numberOfGraph;//Có bao nhiêu đồ thị

	//Tạo ra một mảng để lưu trữ số lượng cạnh của các đồ thị trong CSDL
	int noElem_hNumberEdgeInEachGraph=numberOfGraph;
	int *hNumberEdgeInEachGraph; /* mảng này ở bộ nhớ host */
	int *dNumberEdgeInEachGraph=nullptr; /* mảng này ở bộ nhớ trên device */
	cudaStatus = getNumberOfEdgeInGraph(d_O,numberOfElementd_N,maxOfVer,hNumberEdgeInEachGraph,dNumberEdgeInEachGraph,numberOfGraph);
		if(cudaStatus != hipSuccess){
			fprintf(stderr,"\n getNumberOfEdgeInGraph() in kernel.cu failed",cudaStatus);
			goto Error;
		}

		printf("\n ************ hNumberEdgeInEachGraph **************\n");
		for (int i = 0; i < numberOfGraph; i++)
		{
			printf("\n hNumberEdgeInEachGraph[%d]:%d",i,hNumberEdgeInEachGraph[i]);
		}
#pragma endregion


#pragma region "Build DFS_CODE for the valid edge in d_UniqueExtension && CHECK minDFS_CODE && Create Embedding Column && find Extension from all Embedding"

	/*	Build DFS_Code for the valid Extension********************************************************************
	*	Duyệt qua các cạnh trong mảng d_UniqueExtension và đối chiếu với độ hỗ trợ của cạnh trong mảng h_resultSup
	*	Nếu độ hỗ trợ >= minsup thì sẽ ghi tạo DFS_code cho nó.
	*/
	//Cấp phát bộ nhớ tạm để lấy nhãn cạnh từ device
		UniEdge *h_tempEdge=nullptr;
		h_tempEdge=(UniEdge*)malloc(sizeof(UniEdge));
		if(h_tempEdge==NULL){
			printf("\n malloc h_tempEdge in kernel.cu failed");
			exit(1);
		}
		
	//Duyệt  và kiểm tra xem độ hỗ trợ  của các phần tử trong mảng Unique *d_UniqueExtension có thoả minSup hay không?
	// mảng h_resultSup lưu giá trị support của cạnh tương ứng trong d_UniqueExtension
	for (int i = 0; i < noElem_d_UniqueExtension; i++)
	{
		#pragma region "check minsup statification"

		if(h_resultSup[i]>=minsup){ /*Nếu phần tử nào có độ hỗ trợ lớn hơn minsup thì mới chép nhãn của cạnh đó sang host để kiểm tra minDFS_Code */
			hipMemcpy(h_tempEdge,&d_UniqueExtension[i],sizeof(UniEdge),hipMemcpyDeviceToHost); /* chép cạnh trong d_UniqueExtension sang mảng h_tempEdge */
			
			/* Lấy nhãn của cạnh li,lij và lj để xây dựng DFS_CODE */			
			int li,lij,lj;
			li =h_tempEdge[0].li;
			lij = h_tempEdge[0].lij;
			lj=h_tempEdge[0].lj;
			//printf("\n (%d,%d,%d)",h_tempEdge[0].li,h_tempEdge[0].lij,h_tempEdge[0].lj); 
			

			gspan.DFS_CODE.push(0,1,li,lij,lj); /* Xây dựng DFS_CODE ban đầu cho cạnh */

			int minLabel=li; /* lấy minLabel để phục vụ cho quá trình mở rộng cạnh */
			int maxid = 1; /* id lớn nhất của DFS_CODE */
			
			/*	 Kiểm tra xem DFS_CODE có phải là nhỏ nhất hay không. Nếu DFS_CODE là nhỏ nhất thì mới ghi kết quả DFS_CODE vào file result.txt
			 *	 Nếu thoả minDFS_CODE thì quá trình khai thác sẽ được lặp đi lặp lại cho đến khi nào không thể khai thác trên nhánh đó được nữa.
			*/

#pragma region "check graphismin"

			if(gspan.is_min()){
				int *hArrGraphId;
				int noElem_hArrGraphId=0;
				hArrGraphId=(int*)malloc(sizeof(int)*noElemF);	

				/* Trước khi ghi kết quả thì phải biết đồ thị phổ biến đó tồn tại ở những graphId nào. Hàm getGraphIdContainEmbedding dùng để làm việc này
				* 3 tham số đầu tiên của hàm là nhãn cạnh của phần tử d_UniqueExtension đang xét */
				cudaStatus =getGraphIdContainEmbedding(li,lij,lj,d_ValidExtension,noElem_d_ValidExtension,hArrGraphId,noElem_hArrGraphId,maxOfVer);
				if (cudaStatus!=hipSuccess){
					fprintf(stderr,"\n getGraphIdContainEmbedding in kernel.cu failed",cudaStatus);
					exit(1);
				}
				
				//In nội dung mảng hArrGraphId
				/* 
				printf("\n ************** hArrGraphId ****************\n");
				for (int j = 0; j < noElem_hArrGraphId; j++)
				{
					printf("%d ",hArrGraphId[j]);
				}*/

				/*	Ghi kết quả DFS_CODE vào file result.txt ************************************************************
				 *	Hàm report sẽ chuyển DFS_CODE pattern sang dạng đồ thị, sau đó sẽ ghi đồ thị đó xuống file result.txt
				 *	Hàm report gồm 3 tham số:
				 *	Tham số thứ 1: mảng chứa danh sách các graphID chứa DFS_CODE pattern
				 *	Tham số thứ 2: số lượng mảng
				 *	Tham số thứ 3: độ hỗ trợ của DFS_CODE pattern *******************************************************/
				gspan.report(hArrGraphId,noElem_hArrGraphId,h_resultSup[i]);

				//Giải phóng bộ nhớ hArrGraphId
				free(hArrGraphId);
				

				/* Tạo Embedding cho DFS_CODE **************************************
				 * Mỗi một cột Q được mô tả bởi 3 mảng: dArrPointerEmbedding,dArrSizedQ,dArrPrevQ
				 * Dựa vào d_ValidExtension để xây dựng Embedding cho các DFS_CODE */
				Embedding **dArrPointerEmbedding;
				int noElem_dArrPointerEmbedding=0; //Số lượng cột Q
				int *dArrSizedQ=nullptr; //Số lượng phần tử của từng cột Q
				int noElem_dArrSizedQ=0;
				
				//Không dùng mảng dArrPrevQ nữa
				/*int *dArrPrevQ; //Liên kết với PrevQ nào
				int noElem_dArrPrevQ=0;
				cudaStatus = createEmbeddingRoot(dArrPointerEmbedding,noElem_dArrPointerEmbedding,dArrSizedQ,noElem_dArrSizedQ,dArrPrevQ,noElem_dArrPrevQ,d_ValidExtension,noElem_d_ValidExtension,li,lij,lj);
				if(cudaStatus!=hipSuccess){
					fprintf(stderr,"\n createEmbeddingRoot in kernel.cu failed");
					goto Error;
				}
				*/

				//Tạo embedding root không có prevQ
				cudaStatus = createEmbeddingRoot1(dArrPointerEmbedding,noElem_dArrPointerEmbedding,dArrSizedQ,noElem_dArrSizedQ,d_ValidExtension,noElem_d_ValidExtension,li,lij,lj);
				if(cudaStatus!=hipSuccess){
					fprintf(stderr,"\n createEmbeddingRoot in kernel.cu failed");
					goto Error;
				}

				//In nội dung của Embedding column.
				cudaStatus = printAllEmbeddingColumn(dArrPointerEmbedding,dArrSizedQ,noElem_dArrPointerEmbedding);	
				if(cudaStatus!=hipSuccess){
					fprintf(stderr,"\n printAllEmbeddingColumn() in kernel.cu failed");
					goto Error;
				}
				

				/*Cần ít nhất 2 tham số để truy xuất toàn bộ các đỉnh trên embedding
				*	- Vị trí trong mảng dArrPointerEmbedding (posColumn)
				*	- vị trí trong mảng dArrSizedQ (posRow)
				*/
				/*
				int posColumn =1;
				int posRow = 2;
				cudaStatus = printEmbeddingFromPos(dArrPointerEmbedding,posColumn,posRow);
				if(cudaStatus!=hipSuccess){
					fprintf(stderr,"\n printEmbeddingFromPos() in kernel.cu failed");
					goto Error;
				}
				*/
				/* Tạo dRMPath cho Embedding Column. Nó lưu trữ index của dArrPointerEmbedding mà tại đó Q column thuộc right most path 
				 * Làm sao để cập nhật dRMPath khi mở rộng embedding? 
				 * Nếu có embedding và thông tin cột cuối cùng của embedding column 
				 * thì chúng ta có thể lần ngược về các cột phía trước của embedding
				 * và đồng thời cập nhật lại dRMPath
				 * Cần viết một hàm để tính kích thước của dRMPath
				 * Sau đó viết một hàm nữa để cập nhật giá trị cho dRMPath */

				int *dRMPath=nullptr;
				int noElem_dRMPath=0;		
				cudaStatus = createRMPath(dRMPath,noElem_dRMPath);
				if (cudaStatus!=hipSuccess){
					fprintf(stderr,"\n createRMPath() in kernel.cu failed",cudaStatus);
					goto Error;
				}

				//cudaStatus = printRMPath(dRMPath,noElem_dRMPath);
				//if (cudaStatus!=hipSuccess){
				//	fprintf(stderr,"\n printRMPath() in kernel.cu failed",cudaStatus);
				//	goto Error;
				//}

				/* Tiếp theo là tìm các mở rộng hợp lệ từ các embedding 
			* Các mở rộng hợp lệ có thể là mở rộng forward hoặc là mở rộng backward

				Backward edge extension:
				Trong đó: 
				- ĐK1: id của đỉnh "to" của cạnh cuối cùng bằng với id của đỉnh from thuộc cạnh right most path đang xét (Hay nói cách khác, đỉnh "to" của cạnh mở rộng phải thuộc right most path).
				- ĐK2: Nhãn cạnh của mở rộng lớn hơn nhãn cạnh của đỉnh from của cạnh thuộc right most path
				- ĐK3: Gồm 3.1 và 3.2
				o ĐK3.1: Nhãn cạnh của mở rộng bằng với nhãn cạnh của right most path
				o ĐK3.2: Nhãn đỉnh to của cạnh cuối cùng (đỉnh from của cạnh mở rộng) của DFS_CODE lớn hơn hoặc bằng nhãn đỉnh to của right most path.

				Forward edge extension:
				1.2.1 Tìm tất cả các mở rộng forward edge từ đỉnh phải cùng của DFS_CODE (get_forward_pure function)
				- Chỉ lấy những mở rộng mà có nhãn đỉnh "to" lớn hơn hoặc bằng nhãn đỉnh minlabel (minlabel là nhãn đỉnh from của cạnh đầu tiên trong DFS_CODE) và đỉnh to của cạnh mở 	rộng chưa thuộc DFS_CODE.
				1.2.2 Tìm tất cả các mở rộng forward edge từ các đỉnh còn lại thuộc right most path (get_forward_rmpath function)
				- Loại bỏ mở rộng khi:
				o id đỉnh to của mở rộng bằng với id đỉnh to của cạnh right most path, vì lúc này cạnh mở rộng trùng với right most path.
				o hoặc nhãn đỉnh to của mở rộng nhỏ hơn nhãn đỉnh minlabel
				o hoặc id của đỉnh to của mở rộng đã thuộc DFS_CODE rồi.
				- Chỉ lấy các mở rộng khi:
				o Nhãn cạnh của right most path nhỏ hơn nhãn cạnh mở rộng
				o hoặc nhãn cạnh của right most path bằng với nhãn cạnh mở rộng và nhãn đỉnh to của righ most path nhỏ hơn hoặc bằng nhãn đỉnh to của mở rộng.
			* Với điều kiện như trên thì dữ liệu đầu vào cần phải có những gì:
			Input:
				- Embedding column: Gồm 3 mảng và các thông tin mô tả cho chúng
				- Right Most Path: là một mảng lưu trữ những Q column thuộc right most path
				- Database: gồm mảng d_O,d_LO,d_N,d_LN và thông tin kích thước của từng mảng
			Output:
				- Extension: mảng d_ForwardExtension và thông tin mô tả kích thước.
			* Mở rộng phải được thực hiện theo trình tự: Backward Extension --> forward Extension (từ đỉnh cuối lần ngược lên đỉnh root của embedding).
			* Mở rộng backward chỉ tồn tại ở đỉnh cuối và right most path có nhiều hơn 1 cạnh
			* Mở rộng forward từ đỉnh cuối:
				 
				 Nhưng từ đỉnh cuối thì làm sao mà lần ra được tất cả các đỉnh thuộc embedding? Nếu từ cột cuối thì chỉ có thể lần ra right most path của embedding mà thôi.
				 Cho nên chúng ta không thể kiểm tra được đỉnh hoặc cạnh đó có thuộc embedding hay không.
				 ==> Chúng ta cần xây dựng GraphHistory, nó cho biết những đỉnh nào và cạnh nào đã thuộc embedding và những đỉnh nào và cạnh nào thuộc right most path.
			Làm sao để xây dựng graphHistory?
			 - Có bao nhiêu Embedding thì xây dựng bấy nhiêu graphHistory. Phải dựa vào kích thước của cột Q cuối cùng để biết số lượng Embedding.
			 - Các graphHistory có số lượng đỉnh bằng nhau (maxOfVer).
			 - Các graphHistory có số lượng cạnh khác nhau
			 - Duyệt qua embedding và gán các giá trị tương ứng cho graphHistory tương ứng là 2(ý muốn nói những đỉnh và cạnh đó thuộc embedding
			 - Mỗi khi có embedding mới, thì ta cần phải cập nhật lại graphHistory tương ứng, chuyển những giá trị 2 thành giá trị 1. Sau đó, duyệt qua embedding
			   mới và gán lại giá trị là 2 tại những vị trí thuộc right most path.
				 
			*/

///*************Tạm gác lại vấn đề khai thác dựa trên graphHistory *******************/
///****************Sẽ quay lại sau, để so sánh hiệu năng *****************************/
			#pragma region "graphHistory is temporary stopping here"
								/*
								////Xây dựng graphHistory
								///*  Mỗi Embedding có một graphHistory gắn liền với nó, mô tả những cạnh và đỉnh đã thuộc embedding
								//*	Khi mở rộng embedding thì phải cập nhật lại graphHistory nếu mở rộng đó là phổ biến
								//*	Ngược lại, nếu không có mở rộng phổ biến nào được phát triển từ embedding đó thì graphHistory của nó cũng phải được giải phóng bộ nhớ.
								//*	==> graphHistory phải được xây dựng bên trong quá trình mở rộng embedding và được cập nhật cho đến khi không còn mở rộng phổ biến nào tồn tại.
								//Nhưng ở đây các mở rộng từ các embedding phải được thực hiện một các song song. Tức là chúng ta phải có một mảng các graphHistory 
								// */


				
								/*
								//graphHistory là sự kết hợp gồm 3 mảng bên dưới. Số lượng phần tử của các mảng này bằng nhau và bằng số lượng embedding.
								int noElem_dArrPointerdHO = 0;
								int **dArrPointerdHO=nullptr; //Lưu trữ pointer của mảng các đỉnh trên device
								int **dArrPointerdHLN=nullptr; //Lưu trữ pointer của mảng các cạnh trên device
								int *dArrNumberEdgeOfEachdHLN=nullptr; //số lượng phần tử của mảng các cạnh trên device
								*/
								/*Tạo graphHistory
								*	Input: Các cột embedding (bộ 6 thành phần), CSDL (d_O,d_N,d_LO,d_LN) và số lượng phần tử của chúng
								*	Output: Bộ 3 thành phần graphHistory
								*/
								/*
								cudaStatus = createGraphHistory(dArrPointerEmbedding,dArrSizedQ,dArrPrevQ,noElem_dArrPointerEmbedding,noElem_dArrSizedQ,noElem_dArrPrevQ,d_O,d_LO,numberOfElementd_O,d_N,d_LN,numberOfElementd_N,maxOfVer,dArrPointerdHO,noElem_dArrPointerdHO,dArrPointerdHLN,dArrNumberEdgeOfEachdHLN,hNumberEdgeInEachGraph,noElem_hNumberEdgeInEachGraph,dNumberEdgeInEachGraph);
								if(cudaStatus!=hipSuccess){
									fprintf(stderr,"\n createGraphHistory() in kernel.cu failed",cudaStatus);
									goto Error;
								}
								*/
								//In nội dung của graphHistory
								//printf("\n noElem_dArrPointerdHO:%d",noElem_dArrPointerdHO);
 								//printf("\n ********** dArrPointerdHO *****************\n");
								//printDoublePointerInt(dArrPointerdHO,noElem_dArrPointerdHO,maxOfVer);
								//if(cudaStatus!=hipSuccess){
								//	fprintf(stderr,"\n printDoublePointerInt() in kernel.cu failed",cudaStatus);
								//	goto Error;
								//}
								//printf("\n**************dArrNumberEdgeOfEachdHLN**************\n");
								//printInt(dArrNumberEdgeOfEachdHLN,noElem_dArrPointerdHO);
								//printf("\n**************dArrPointerdHLN**************\n");
								//printDoublePointerInt(dArrPointerdHLN,noElem_dArrPointerdHO,dArrNumberEdgeOfEachdHLN);
				#pragma endregion

//Tìm các mở rộng hợp lệ từ các đỉnh của embedding. 
#pragma region "forward Extension"

				/* Mở rộng từ tất cả các đỉnh của embedding một cách song song.
				* Kết quả là các mở rộng hợp lệ được lưu trữ vào mảng dArrPointerExt
				* dArrPointerExt là một mảng lưu trữ pointer trỏ đến các mảng dExt
				* dArrPointerExt có số lượng phần tử bằng với kích thước của RightMostPath
				*/
				EXT** dArrPointerExt=nullptr;
				 int *dArrNoElemPointerExt = nullptr;
				 

				 int noElem_dArrPointerExt=noElem_dRMPath;
				cudaStatus = forwardExtension(dArrPointerEmbedding,noElem_dArrPointerEmbedding,dArrSizedQ,noElem_dArrSizedQ,dRMPath,noElem_dRMPath,d_O,d_LO,d_N,d_LN,numberOfElementd_O,numberOfElementd_N,maxOfVer,dArrPointerExt,noElem_dArrPointerExt,minLabel,maxid,dArrNoElemPointerExt);
				 if(cudaStatus!=hipSuccess){
					 fprintf(stderr,"\n forwardExtension() in kernel.cu failed",cudaStatus);
					 goto Error;
				 }					


				cudaStatus = printInt(dArrNoElemPointerExt,noElem_dArrPointerExt);
				 if(cudaStatus!=hipSuccess){
					 fprintf(stderr,"\n printInt() in kernel.cu failed",cudaStatus);
					 goto Error;
				 }	

				 //In nội dung mảng dArrPointerExt dựa vào mảng kích thước của từng phần tử dArrNoElemPointerExt 
				 printf("\n*******dArrPointerExt******\n");
				cudaStatus = printdArrPointerExt(dArrPointerExt,dArrNoElemPointerExt,noElem_dArrPointerExt);
				if(cudaStatus!=hipSuccess){
					 fprintf(stderr,"\n printdArrPointerExt() in kernel.cu failed",cudaStatus);
					 goto Error;
				 }	
#pragma endregion

#pragma region "Unique Extension Extraction"				
				
				//Quá trình rút trích này không cần dùng đến Embedding
				UniEdge **dArrPointerUniEdge=nullptr;
				int noElem_dArrPointerUniEdge = noElem_dRMPath;
				int *dArrNoELemPointerUniEdge=nullptr; //Mảng này có số lượng phần tử bằng với dArrPointerUniEdge, nó cho biết số lượng phần tử tương ứng mối phần tử trong dArrPointerUniEdge
				//Duyệt qua các EXTk và trả về pointer UniEdge* dUniEdge

				 int *hArrNoElemPointerExt= (int*)malloc(sizeof(int)*noElem_dArrPointerExt);
				 if (hArrNoElemPointerExt==NULL){
					 printf("\nMalloc hArrNoElemPointerExt in kernel.cu failed");
					 exit(1);
				 }
				 else
				 {
					 hipMemcpy(hArrNoElemPointerExt,dArrNoElemPointerExt,sizeof(int)*noElem_dArrPointerExt,hipMemcpyDeviceToHost);
				 }
				 //Gọi hàm trích các mở rộng duy nhất
				cudaStatus = extractUniExtension(dArrPointerExt,noElem_dArrPointerExt,Lv,Le,dArrPointerUniEdge,noElem_dArrPointerUniEdge,dArrNoELemPointerUniEdge,hArrNoElemPointerExt,dArrNoElemPointerExt);
				if(cudaStatus!=hipSuccess){
					fprintf(stderr,"\n extractUniExtension() in kernel.cu failed",cudaStatus);
					goto Error;
				}

				printf("\n***********dArrNoELemPointerUniEdge***********\n");
				cudaStatus =printInt(dArrNoELemPointerUniEdge,noElem_dArrPointerUniEdge);
				if(cudaStatus!=hipSuccess){
					fprintf(stderr,"\n printInt dArrNoELemPointerUniEdge in kernel.cu failed",cudaStatus);
					goto Error;
				}

				printf("\n***********dArrPointerUniEdge***********\n");
				cudaStatus =printArrPointerUniEdge(dArrPointerUniEdge,dArrNoELemPointerUniEdge,noElem_dArrPointerUniEdge);
				if(cudaStatus!=hipSuccess){
					fprintf(stderr,"\n printArrPointerUniEdge dArrPointerUniEdge in kernel.cu failed",cudaStatus);
					goto Error;
				}

#pragma endregion


#pragma region "Compute Support"
				
				/* 
					Tính độ hỗ trợ của các cạnh trong dArrPointerUniEdge
					Giải thuật:
					1. Duyệt qua từng phần tử trong dArrPointerUniEdge. 
					2. 
				*/
				
				int *hArrNoELemPointerUniEdge =(int*)malloc(sizeof(int)*noElem_dArrPointerUniEdge);
				if(hArrNoELemPointerUniEdge==NULL){
					printf("\nmalloc hArrNoElemPointerUniEdge in kernel.cu failed");
					exit(1);
				}
				else
				{
					hipMemcpy(hArrNoELemPointerUniEdge,dArrNoELemPointerUniEdge,sizeof(int)*noElem_dArrPointerUniEdge,hipMemcpyDeviceToHost);
				}

				//Các biến sau dùng để lưu trữ kết quả tính độ hỗ trợ của các Unique Edge trong mảng dArrPointerUniEdge.
				unsigned int **hArrPointerSupport=nullptr;
				unsigned int *hArrNoElemPointerSupport=nullptr;
				unsigned int noElem_hArrPointerSupport=noElem_dArrPointerUniEdge;
				//Gọi hàm computeSupportv2 để tính độ hỗ trợ và lưu kết quả vào hArrPointerSupport
				cudaStatus=computeSupportv2(dArrPointerExt,dArrNoElemPointerExt,hArrNoElemPointerExt,noElem_dArrPointerExt,dArrPointerUniEdge,dArrNoELemPointerUniEdge,hArrNoELemPointerUniEdge,noElem_dArrPointerUniEdge,hArrPointerSupport,hArrNoElemPointerSupport,noElem_hArrPointerSupport,maxOfVer);
				if(cudaStatus!=hipSuccess){
					fprintf(stderr,"\n computeSupportv2 in kernel.cu failed",cudaStatus);
					goto Error;
				}
				
				for (int i = 0; i < noElem_hArrPointerSupport; i++)
				{
					int noElem = hArrNoElemPointerSupport[i];
					unsigned int *dArr = hArrPointerSupport[i];
					for (int j = 0; j < noElem; j++)
					{
						printf("\nSupport of i:%d in j:%d:%d",i,j,dArr[j]);
					}
				}

#pragma endregion "Ending of compute support"


				free(hArrNoELemPointerUniEdge);
				//Giải phóng bộ nhớ mảng UniEdge **dArrPointerUniEdge
				cudaStatus = cudaFreeArrPointerUniEdge(dArrPointerUniEdge,dArrNoELemPointerUniEdge,noElem_dArrPointerUniEdge);		
				if(cudaStatus!=hipSuccess){
					fprintf(stderr,"\n cudaFreeArrPointerExt() in kernel.cu failed",cudaStatus);
					goto Error;
				}
				//Giải phóng bộ nhớ mảng dArrPointerExt
				cudaStatus = cudaFreeArrPointerExt(dArrPointerExt,dArrNoElemPointerExt,noElem_dArrPointerExt);		
				if(cudaStatus!=hipSuccess){
					fprintf(stderr,"\n cudaFreeArrPointerExt() in kernel.cu failed",cudaStatus);
					goto Error;
				}

				//Giải phóng bộ nhớ mảng dArrPointerEmbedding
				cudaStatus = cudaFreeArrPointerEmbedding(dArrPointerEmbedding,dArrSizedQ,noElem_dArrPointerEmbedding);		
				if(cudaStatus!=hipSuccess){
					fprintf(stderr,"\n cudaFreeArrPointerEmbedding() in kernel.cu failed",cudaStatus);
					goto Error;
				}

				//Giải phóng bộ nhớ mảng hArrPointerSupport
				

			} //endif check graphismin
#pragma endregion "checkgraphismin"


			gspan.DFS_CODE.pop();			
		}//endif check minsup statification	
#pragma endregion "check minsup statification"
	} //end for. Kết thúc việc duyệt qua mảng d_UniqueExtension để tính độ hỗ trợ

#pragma endregion
	
	
#pragma region "Tam gac lai van de nay, se quay lai sau"
	///*
	//for (int i = 0; i < h_satisfyEdge.size(); i++)
	//{
	//	printf("\n h_satisfyEdge[%d]:%d",i,h_satisfyEdge[i]);
	//	printf("\n h_satisfyEdgeSupport[%d]:%d",i,h_satisfyEdgeSupport[i]);
	//}



	//getch();

	////Tiếp theo chúng ta dựa vào mảng h_frequentEdge để trích ra những cạnh phổ biến và xây dựng DFS_CODE cho chúng.
	////Sau khi xây dựng DFS_Code, chúng ta sẽ chuyển chúng sang đồ thị và ghi đồ thị đó vào file kết quả result.txt

	///*gspan.DFS_CODE.push(0,1,0,0,1);	
	//bool min = gspan.is_min();
	//printf("\n min:%d",min);*/

	////Gọi P là pattern và EP là các embedding của pattern P
	////Làm sao để có P? ==> Dựa vào h_frequentEdge để lấy cạnh trong d_UniqueExtension xây dựng DFS_Code P
	//for (int i = 0; i < h_satisfyEdge.size(); i++)
	//{		
	//	int li;
	//	int lij;
	//	int lj;
	//	int indexOfSatisfyEdge=h_satisfyEdge[i];
	//	int *d_arr_edgeLabel=nullptr;
	//	cudaStatus = getSatisfyEdge(d_UniqueExtension,noElem_d_UniqueExtension,indexOfSatisfyEdge,li,lij,lj,d_arr_edgeLabel);
	//	if(cudaStatus != hipSuccess){
	//		fprintf(stderr,"\n getSatisfyEdge failed",cudaStatus);
	//		//goto labelError;
	//		exit(1);
	//	}

	//	int *h_arr_graphIdContainEmbedding=nullptr;
	//	int noElem_h_arr_graphIdContainEmbedding=0;
	//cudaStatus =getGraphIdContainEmbedding(li,lij,lj,d_ValidExtension,noElem_d_ValidExtension,h_arr_graphIdContainEmbedding,noElem_h_arr_graphIdContainEmbedding,maxOfVer); //hàm này được để trong calcSupport file
	//	if(cudaStatus != hipSuccess){
	//		fprintf(stderr,"\n getGraphIdContainEmbedding failed",cudaStatus);
	//		//goto labelError;
	//		exit(1);
	//	}
	//	
	//	//printf("\n i:%d (li:%d, lij:%d, lj:%d)",i,li,lij,lj);
	//	//printInt(d_arr_edgeLabel,3);			
	//	//1.Xây dựng DFS_CODE, đồng thời ghi nhận lại minLabel và maxtoc của DFS_CODE
	//	gspan.DFS_CODE.push(0,1,li,lij,lj); //Cạnh đầu tiên của DFS_Code luôn có (vi,vj)=(0,1), khi mở rộng DFS_CODE thì 
	//	int minLabel = 0;
	//	int maxtoc = 1; //là id của đỉnh cuối cùng trên rmpath của DFS_CODE_MIN
	//					//tuỳ vào backward hay forward để tính (vi,vj). 
	//	//2. Ở đây các cạnh đã thoả minDFS_CODE, nên không cần xét minDFS_CODE trong trường hợp này.

	//	//3. Chuyển DFS_CODE sang đồ thị và ghi kết quả vào tập tin			
	//	//int graph[3]={0,1,2};
		//gspan.report(h_arr_graphIdContainEmbedding,noElem_h_arr_graphIdContainEmbedding,h_satisfyEdgeSupport[i]);
	//	//4. Tìm các Embedding của DFS_CODE
	//	//xây dựng embedding cho mở rộng thoả minsup (dùng struct_Q *device_arr_Q=NULL; để lưu trữ các cột Q của embeddings)
	//		struct_Q *device_arr_Q=nullptr; //các cột Q và thông tin của nó được lưu trữ trong mảng cấu trúc struct_Q *device_arr_Q;
	//		printf("\n***********support of (%d,%d,%d) >= %d --> create embeddings for DFS_CODE************",li,lij,lj,minsup);
	//		cudaStatus=createForwardEmbedding(d_ValidExtension,noElem_d_ValidExtension,li,lij,lj,d_O,d_LO,numberOfElementd_O,d_N,d_LN,numberOfElementd_N,Lv,Le,minsup,maxOfVer,numberOfGraph,noDeg,device_arr_Q);
	//		if (cudaStatus!=hipSuccess){
	//			fprintf(stderr,"\ncreateForwardEmbedding failed");
	//			exit(1);
	//		}

	//		//Lấy số lượng phần tử của một cột Q bất kỳ trong mảng device_arr_Q
	//		/*
	//		printf("\nPrint information of size of the last element of d_arr_Q:");	
	//		int positionLastElement = 1;
	//		int *dsizeOfLastElement;
	//		int hsizeOfLastElement=0;
	//		
	//		cudaStatus = hipMalloc((void**)&dsizeOfLastElement,sizeof(int));
	//		if(cudaStatus!=hipSuccess){
	//			fprintf(stderr,"\n hipMalloc dsizeOfLastElement failed");
	//			//goto Error;
	//			exit(1);
	//		}
	//		else
	//		{
	//			hipMemset(dsizeOfLastElement,0,sizeof(int));
	//		}
	//		*/
	//		//Hàm kernelGetInformationLastElement sẽ lấy kích thước của cột Q trong mảng device_arr_Q và lưu kết quả vào biến
	//		/*
	//		kernelGetInformationLastElement<<<1,1>>>(device_arr_Q,positionLastElement,dsizeOfLastElement); 
	//		hipDeviceSynchronize();
	//		hipMemcpy(&hsizeOfLastElement,dsizeOfLastElement,sizeof(int),hipMemcpyDeviceToHost);
	//		printf("\nhsizeOfLastElement:%d",hsizeOfLastElement);
	//		*/
	//		//Truy xuất tất cả các Embeddings khi truyền vào một mảng cấu trúc struct_Q: device_arr_Q
	//		/*
	//		printf("\n\nPrint all embedding from the last element of device_arr_Q");
	//		PrintAllEmbedding<<<1,hsizeOfLastElement>>>(device_arr_Q,1,hsizeOfLastElement);
	//		hipDeviceSynchronize();
	//		*/
	//		
	//		//11.5.2 Tìm các mở rộng cho các Embedding của DFS_CODE
	//		/* 
	//		- Sau khi đã xây dựng được các Embedding columns để biểu diễn embeddings cho các frequent 1-edge extension.
	//		- Cụ thể các cột embedding ở đây là một mảng device_arr_Q, mỗi phần tử của device_arr_Q là một cột Q, với chỉ số 
	//		được tính bắt đầu từ 0 (Q0, Q1, Q2,...).
	//		- Mảng RMPath: dùng để lưu trữ index của device_arr_Q mà tại đó cột Q thuộc Right Most Path. 
	//		- Biến lastColumn: lữu trữ index của cột cuối cùng (tức là đỉnh phải nhất của Embedding). Từ cột này chúng ta có
	//		thể lần ngược để duyệt qua tất cả các Q thuộc Right Most Path
	//		1. Viết hàm getExtension
	//		- Input: device_arr_Q,lastColumn,RMPath,d_O,d_LO,numberOfElementd_O,d_N,d_LN,numberOfElementd_N,Lv,Le,minsup.
	//		- Output: RMPath
	//		*/
	//		int lastColumn=1; //ở đây các embedding chỉ có 1 cạnh, nên Q cuối cùng nằm ở vị trí index=1 trong mảng device_arr_Q,
	//							//Khi mở rộng embedding và bổ sung thêm Q mới vào sau mảng device_arr_Q thì chúng ta phải cập nhật lại lastColumn						
	//		vector<int> RighMostPath(2); //chứa index của mảng device_arr_Q mà tại đó cột Q thuộc right most path
	//		RighMostPath.at(0)=0;							// Tương tự lastColumn, Khi mở rộng embedding và bổ sung thêm Q mới vào sau mảng device_arr_Q thì chúng ta phải cập nhật lại RightMostPath	
	//		RighMostPath.at(1)=1;
	//		cHistory **dH=nullptr; //hàm getExtension sẽ trả về dH và số lượng phần tử của dH (chính bằng số lượng embedding) của DFS_CODE đang xét
	//		int numberElem_dH=0; //Số lượng embeddings của DFS_CODE
	//		cudaStatus = getExtension(device_arr_Q,lastColumn,RighMostPath,d_O,d_LO,numberOfElementd_O,d_N,d_LN,numberOfElementd_N,Lv,Le,minsup,maxOfVer,numberOfGraph,noDeg,dH,numberElem_dH); 
	//		if(cudaStatus !=hipSuccess){
	//			fprintf(stderr,"\n getExtension failed",cudaStatus);
	//			exit(1);
	//		}

	//		//Duyệt qua từng embedding trong device_arr_Q và đánh dấu những đỉnh và cạnh thuộc embedding trong dH là 2
	//		cudaStatus = markEmbedding(dH,device_arr_Q,lastColumn,numberElem_dH,maxOfVer,d_O,d_N);
	//		if (cudaStatus!=hipSuccess){
	//			fprintf(stderr,"\n markEmbedding function has been failed.",cudaStatus);
	//			/*goto labelError;*/
	//			exit(1);
	//		}			

	//		printf("\n****************Display history of all embedding in dH array***********"); //kiểm tra thử dữ liệu của mảng dH trên device sau khi đã đánh dấu các embedding thuộc right most path
	//		kernelPrintdeviceH<<<1,1>>>(dH,numberElem_dH);
	//		hipDeviceSynchronize();
	//		
	//		//Duyệt qua từng cột Q thuộc RightMostPath trong device_arr_Q và tìm những mở rộng hợp lệ từ các đỉnh trên cột Q
	//		for (int i = RighMostPath.size()-1; i >=0; i--)
	//		{					
	//			Extension *d_arrE=nullptr;
	//			int numberElement_d_arrE=0;
	//			cudaStatus=getValidExtensionFromEmbeding(d_arrE,numberElement_d_arrE,device_arr_Q,RighMostPath[i],dH,numberElem_dH,maxOfVer,d_O,d_LO,d_N,d_LN,numberOfElementd_O,numberOfElementd_N,lastColumn);
	//			if(cudaStatus!=hipSuccess){
	//				fprintf(stderr,"\ngetValidExtensionFromEmbedding failed",cudaStatus);
	//				//goto labelError;
	//				exit(1);
	//			}

	//			printfExtension(d_arrE,numberElement_d_arrE);
	//			hipDeviceSynchronize();
	//		}

	//		//5. Xây dựng hàm để lặp lại quá trình khai thác 
	//		gspan.DFS_CODE.pop();
	//}

	//printf("\n***********Finished: compute support for valid unique extension in d_UniqueExtension **********");
	//printf("\n***********Press the Enter key to continous**********\n");

	//getch();
#pragma endregion

Error:
	//giải phóng vùng nhớ của dữ liệu
	free(h_resultSup);
	free(hNumberEdgeInEachGraph);
	hipFree(dNumberEdgeInEachGraph);
	hipFree(d_O);
	hipFree(d_LO);
	hipFree(d_N);
	hipFree(d_LN);	
	//	hipFree(d_singlePattern);
	hipFree(d_Extension);
	hipFree(V);
	hipFree(index);
	hipFree(d_ValidExtension);	
	hipFree(d_allPossibleExtension);
	hipFree(d_allPossibleExtensionScanResult);
	hipFree(d_UniqueExtension);
	hipFree(d_B);
	
	hipDeviceReset();	

	fout.close();
	//delete[] arrayO;
	free(h_tempEdge);
	delete[] arrayN;
	delete[] arrayLO;
	delete[] arrayLN;

	getch();
	return 0;
}
