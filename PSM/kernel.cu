#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <string>
#include <map>
#include "conio.h"
#include "kernelPrintf.h"
#include "gspan.h"
#include "kernelCountLabelInGraphDB.h"
#include "kernelMarkInvalidVertex.h"
#include "markInvalidVertex.h"
#include "checkArray.h"
#include "displayArray.h"
#include "checkDataBetweenHostAndGPU.h"
#include "access_d_LO_from_idx_of_d_O.h"
#include "countNumberOfLabelVetex.h"
#include "countNumberOfEdgeLabel.h"
#include "extractUniqueEdge.h"
#include "ExtensionStructure.h"
#include "getAndStoreExtension.h"
#include "validEdge.h"
#include "scanV.h"
#include "getLastElement.h"

using namespace std;

//declare prototype
//void displayArray(int*,const unsigned int);
//bool checkArray(int*, int*, const int);
//__device__ void __syncthreads(void);

int main(int argc, char * const  argv[])
{	

	//*************************** Load Graph database with some parameters ***********************

	//unsigned int minsup = 34;
	unsigned int minsup = 2;
	unsigned int maxpat = 2;
	//unsigned int maxpat = 0x00000000;
	unsigned int minnodes = 0;
	bool where = true;
	bool enc = false;
	bool directed = false;

	//int opt;
	char* fname;
	//fname = "Klesscus";
	fname = "Klessorigin";

	gSpan gspan;	
	ofstream fout("result.txt");

	//Chuyển dữ liệu từ fname sang TRANS
	gspan.run(fname,fout,minsup,maxpat,minnodes,enc,where,directed);


	unsigned int maxOfVer;
	unsigned int numberOfGraph;
	maxOfVer=gspan.findMaxVertices();
	numberOfGraph=gspan.noGraphs();
	int sizeOfarrayO=maxOfVer*numberOfGraph;

	//printf("\nMaximun number of vertices: %d",maxOfVer);

	int* arrayO = new int[sizeOfarrayO]; //Tạo mảng arrayO có kích thước D*m
	if(arrayO==NULL){
		printf("\n!!!Memory Problem ArrayO");
		exit(1);
	}else{
		memset(arrayO, -1, sizeOfarrayO*sizeof(int)); // gán giá trị cho các phần tử mảng bằng -1
	}

	unsigned int noDeg; //Tổng bậc của tất cả các đỉnh trong csdl đồ thị TRANS
	noDeg = gspan.sumOfDeg();
	//cout<<noDeg;
	unsigned int sizeOfArrayN=noDeg;
	int* arrayN = new int[sizeOfArrayN]; //Mảng arrayN lưu trữ id của các đỉnh kề với đỉnh tương ứng trong mảng arrayO.
	if(arrayN==NULL){ //kiểm tra cấp phát bộ nhớ cho mảng có thành công hay không
		printf("\n!!!Memory Problem ArrayN");
		exit(1);
	}else
	{
		memset(arrayN, -1, noDeg*sizeof(int));
	}


	int* arrayLO = new int[sizeOfarrayO]; //Mảng arrayLO lưu trữ label cho tất cả các đỉnh trong TRANS.
	if(arrayLO==NULL){ //kiểm tra cấp phát bộ nhớ cho mảng có thành công hay không
		printf("\n!!!Memory Problem ArrayLO");
		exit(1);
	}else
	{
		memset(arrayLO, -1, sizeOfarrayO*sizeof(int));
	}


	int* arrayLN = new int[noDeg]; //Mảng arrayLN lưu trữ label của tất cả các cạnh trong TRANS
	if(arrayLN==NULL){ //kiểm tra cấp phát bộ nhớ cho mảng có thành công hay không
		printf("\n!!!Memory Problem ArrayLN");
		exit(1);
	}else
	{
		memset(arrayLN, -1, noDeg*sizeof(int));
	}


	gspan.importDataToArray(arrayO,arrayLO,arrayN,arrayLN,sizeOfarrayO,noDeg,maxOfVer);

	cout<<"ArrayO:";
	displayArray(arrayO,sizeOfarrayO);
	cout<<"\nArrayLO:";
	displayArray(arrayLO,sizeOfarrayO);
	cout<<"\nArrayN:";
	displayArray(arrayN,noDeg);
	cout<<"\nArrayLN:";
	displayArray(arrayLN,noDeg);

	//kích thước của dữ liệu
	size_t nBytesO = sizeOfarrayO*sizeof(int);
	size_t nBytesLO = sizeOfarrayO*sizeof(int);
	size_t nBytesN = noDeg*sizeof(int);
	size_t nBytesLN = noDeg*sizeof(int);


	//****cấp phát vùng nhớ trên GPU***
	//1. khai báo biến trên GPU
	int *d_O;
	int *d_LO;
	int *d_N; //Số lượng phần tử của d_N bằng noDeg
	int *d_LN;

	//2. Kiểm tra lỗi khi cấp phát
	//Khai báo biến cudaStatusAllocate
	hipError_t cudaStatusAllocate;

	cudaStatusAllocate =hipMalloc((int**) &d_O,nBytesO);
	if (cudaStatusAllocate!=hipSuccess){
		fprintf(stderr, "hipMalloc failed!");
		goto labelError;
	}

	cudaStatusAllocate =hipMalloc((int**) &d_LO,nBytesLO);
	if (cudaStatusAllocate!=hipSuccess){
		fprintf(stderr, "hipMalloc failed!");
		goto labelError;
	}
	cudaStatusAllocate =hipMalloc((int**) &d_N,nBytesN);
	if (cudaStatusAllocate!=hipSuccess){
		fprintf(stderr, "hipMalloc failed!");
		goto labelError;
	}
	cudaStatusAllocate =hipMalloc((int**) &d_LN,nBytesLN);
	if (cudaStatusAllocate!=hipSuccess){
		fprintf(stderr, "hipMalloc failed!");
		goto labelError;
	}


	//chép dữ liệu từ bốn mảng O,LO,N,LN từ Host sang GPU. Đây chính là CSDL đồ thị dùng để khai thác trên GPU
	cudaStatusAllocate = hipMemcpy(d_O,arrayO,nBytesO,hipMemcpyHostToDevice);
	if (cudaStatusAllocate != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto labelError;
	}

	cudaStatusAllocate = hipMemcpy(d_LO,arrayLO,nBytesLO,hipMemcpyHostToDevice);
	if (cudaStatusAllocate != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto labelError;
	}

	cudaStatusAllocate = hipMemcpy(d_N,arrayN,nBytesN,hipMemcpyHostToDevice);
	if (cudaStatusAllocate != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto labelError;
	}

	cudaStatusAllocate = hipMemcpy(d_LN,arrayLN,nBytesLN,hipMemcpyHostToDevice);
	if (cudaStatusAllocate != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto labelError;
	}


	//Đồng bộ đồng thời kiểm tra xem đồng bộ có lỗi không
	cudaStatusAllocate= hipDeviceSynchronize();
	if (cudaStatusAllocate != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatusAllocate);
		goto labelError;
	}
	//xác định grid and block structure
	dim3 block(512);
	dim3 grid((nBytesO + block.x -1)/block.x);
	printf("grid %d; block %d;\n",grid.x,block.x);


	//********kiểm tra đảm bảo dữ liệu ở GPU giống với Host********

	hipError_t cudaStatus = checkDataBetweenHostAndGPU(d_O,d_LO,d_N,d_LN,sizeOfarrayO,noDeg,arrayO,arrayLO,arrayN,arrayLN,nBytesO,nBytesLO,nBytesN,nBytesLN);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "checkDataBetweenHostAndGPU failed!");
		return 1;
	}
	/*
	//********Đếm số đỉnh song song và loại nhỏ những đỉnh nhỏ hơn minsup****
	//Nếu số đỉnh nhỏ hơn minSup thì đánh dấu đỉnh đó là -1 trong mảng O và mảng LO và các cạnh liên quan đến đỉnh đó cũng được đánh dấu là -1

	cudaStatus = markInvalidVertex(d_O,d_LO,sizeOfarrayO,minsup);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "markInvalidVertex failed!");
		return 1;
	}
	*/
	//**********Đếm số nhãn đỉnh khác nhau trong CSDL đồ thị************
	//Nhãn đỉnh được lưu trữ trong mảng d_LO. Nhãn không hợp lệ mang giá trị -1
	//1.Cấp phát một mảng số nguyên có kích thước bằng với kích thước mảng d_LO gọi là d_Lv
	//2.Cấp phát |d_LO| threads
	//3.thread thứ i sẽ đọc giá trị nhãn tại vị trí d_LO[i], rồi ghi 1 vào mảng d_Lv[d_LO[i]]
	//4. Reduction mảng d_Lv để thu được các nhãn phân biệt
	unsigned int Lv=0;
	cudaStatus = countNumberOfLabelVetex(d_LO,sizeOfarrayO,Lv);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "countNumberOfLabelVetex failed!");
		return 1;
	}
	printf("\nNumber of different label Lv is: %d ;",Lv);
	
	//*******Đếm các loại cạnh khác nhau trong CSDL đồ thị************
	//Nhãn của tất cả các cạnh được lưu trữ trong mảng d_LN.
	/*
		1. Cấp phát một mảng số nguyên có kích thước băng với kích thước mảng d_LN gọi là d_Ln
		2. Cấp phát |d_Ln| threads
		3. Thread thứ i sẽ đọc giá trị nhãn tại vị trí d_Ln[i], rồi ghi vào mảng d_Ln[d_LN[i]]
		4. Reduction mảng d_Ln để thu được các loại cạnh phân biệt
	*/
	unsigned int Le=0;
	cudaStatus = countNumberOfEdgeLabel(d_LN,sizeOfArrayN,Le);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "countNumberOfLabelVetex failed!");
		return 1;
	}
	printf("\nNumber of different label Le is: %d ;",Le);
	/*******Thu thập tất cả các pattern P có 1 cạnh và tất cả các embeddings của P ********
	1. Duyệt qua cơ sở dữ liệu và rút trích các cạnh phân biệt dựa vào mảng d_O,d_LO,d_N và d_LN và 
	set giá trị 1 trong mảng d_SinglePattern có kích thước là (Lv.LE)
	tương ứng (lij.Lv+lj).
	2. Reduction d_singlePattern để biết số lượng pattern(numberOfPattern) là bao nhiêu. Sau đó cấp 
	phát numberOfPattern threads để tìm embeddings cho pattern đó.
	   Kết quả được lưu vào mảng cấu trúc d_Ext có số lượng phần tử bằng với d_N. 
	   Các thông tin cần lưu gồm:
		i. DFS Code của pattern theo cấu trúc (vi,vj,li,lij,lj)
	   ii. Lưu trữ các Embeddings của pattern(vig,vjg)
      iii. Row pointer trỏ đến heading của embedding ở cột cuối cùng Qp //
	3. Cấp phát mảng B có kích thước bằng với d_Ext để ghi nhận thông tin boundary của d_Ext
	4. Exclusive scan mảng B để ánh xạ lại graphid
	Tính support của từng pattern P
	5. Cấp phát mảng F có kích thước bằng với số lượng đồ thị trong CSDL
	6. Cấp phát numberOfPattern threads để cập nhật mảng F tương ứng với giá trị của kết quả scan trên B
	7. Reduction F để có độ support của từng pattern.
	*/
	int *d_singlePattern=NULL;
	//size_t nBytesd_singlePattern = Lv*Le*sizeof(int);
	//Kích thước của mảng d_singlePattern là =[Lv!/(k!(Lv-k)!+n]*Le = [((Lv-2+1)*(Lv-2+2))/2 + Lv]*Le = [((Lv-1)*Lv)/2 + Lv]*Le
	//Trong trường hợp này k luôn = 2 vì cạnh có 2 đầu nhãn đỉnh lấy từ tập nhãn đỉnh phân biệt. 
	//Le là tập nhãn cạnh phân biệt trong CSDL
	//Vậy chúng ta có công thức cho kích thước của d_singlePattern như sau:
	unsigned int numberOfElementd_singlePattern=(((Lv-1)*Lv)/2 +Lv)*Le;
	size_t nBytesd_singlePattern = numberOfElementd_singlePattern*sizeof(int);
	cudaStatus=hipMalloc((int**)&d_singlePattern,nBytesd_singlePattern);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"hipMalloc d_singlePattern failed", cudaStatus);
		return 1;
	}
	else
	{
		hipMemset(d_singlePattern,0,nBytesd_singlePattern);
	}
	
	cudaStatus = extractUniqueEdge(d_O,d_LO,sizeOfarrayO,d_N,d_LN,sizeOfArrayN,d_singlePattern,numberOfElementd_singlePattern,Lv,Le);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"call extractUniqueEdge failed",cudaStatus);
		return 1;
	}


	//*******Nhãn truy xuất nhãn LO theo index từ d_N; tên hàm đặt bị nhầm
	/*cudaStatus = access_d_LO_from_idx_of_d_O(d_LO,d_N,sizeOfArrayN);
	if(cudaStatus != hipSuccess){
	fprintf(stderr, "access_d_LO_from_idx_of_d_N");
	return 1;
	}
	*/

	/* //May/04/2017: Trích các cạnh từ CSDL và lưu vào d_Extension.
		1. Tạo một cấu trúc Extension để lưu trữ các mở rộng: DFSCode của cạnh mở rộng (vi,vj,li,lij,lj),global from vertex id(vgi),
		global to vertex id (vgj) và pointer trỏ đến header của embedding tương ứng với cạnh mở rộng.
		2. Tạo một mảng có kích thước bằng với kích thước của d_N để lưu trữ các cạnh mở rộng ban đầu, lúc chưa có bất kỳ một cạnh
		phổ biến nào (P=0).
		3. Tạo một kernel có số lượng threads bằng d_O, mỗi thread sẽ xử lý một đỉnh. Nhiệm vụ của thread là đọc các cạnh kề với nó rồi
		lưu trữ thông tin vào mảng Extension tương ứng tại vị trí.
	*/

	int numberOfElementd_N=noDeg;
	size_t nBytesOfArrayExtension = numberOfElementd_N*sizeof(Extension);
	Extension *d_Extension;
	cudaStatus= hipMalloc((Extension**)&d_Extension,nBytesOfArrayExtension);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"CudaMalloc d_Extension fail",cudaStatus);
		exit(1);
	}


	int numberOfElementd_O=sizeOfarrayO;	
	cudaStatus = getAndStoreExtension(d_Extension,d_O,d_LO,numberOfElementd_O,d_N,d_LN,numberOfElementd_N,Le,Lv);
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize getAndStoreExtension failed",cudaStatus);
		return 1;
	}


	/* //05-May-2017: Khởi tạo mảng V với giá trị của các phần tử ban đầu là 0, để lưu trữ những mở rộng hợp lệ.
	1. Mở rộng hợp lệ là mở rộng có Lj<=Li
	2. Mảng V có số lượng phần tử bằng với số lượng phần tử của mảng d_Extension
	3. Tạo kernel với số lượng threads bằng với số lượng phần tử của d_Extension
		Mỗi thread sẽ xử lý một phần tử trong d_Extension. Kiểm tra nếu Lj<=Li thì gán V tại vị trí tương ứng là 1
	*/

	int numberElementd_Extension = numberOfElementd_N;
	int *V;
	size_t nBytesV= numberElementd_Extension*sizeof(int);

	cudaStatus=hipMalloc((int**)&V,nBytesV);
	if (cudaStatus!= hipSuccess){
		fprintf(stderr,"hipMalloc array V failed",cudaStatus);
		exit(1);
	}
	else
	{
		hipMemset(V,0,nBytesV);
	}

	cudaStatus=validEdge(d_Extension,V,numberElementd_Extension);
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize validEdge failed",cudaStatus);
		return 1;
	}

	/* //07-May-2017: Extract unique Edge from d_Extension
	1. Tiếp theo, chúng ta exclusive scan mảng V để thu được index chỉ vị trí của các valid edge trong d_Extension.
	2. Input data: mảng V
	3. Output data: mảng index
	Mảng Index có số lượng phần tử bằng với mảng V
	*/

	int* index;
	cudaStatus=hipMalloc((int**)&index,numberElementd_Extension*sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"Cuda Malloc failed",cudaStatus);
		return 1;
	}	

	cudaStatus = scanV(V,numberElementd_Extension,index);
	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize getAndStoreExtension failed",cudaStatus);
		return 1;
	}

	//Khởi tạo một mảng d_Unique có kích thước bằng với kích thước của giá trị của phần tử index cuối cùng vừa mới scan được.
	int numberElementd_UniqueExtension=0;
	getLastElement(index,numberElementd_Extension,numberElementd_UniqueExtension);

	printf("\n\nnumberElementd_UniqueExtension:%d",numberElementd_UniqueExtension);

	//gspan.graphMining(arrayO,arrayLO,arrayN,arrayLN,minsup);
labelError:
	//giải phóng vùng nhớ của dữ liệu
	hipFree(d_O);
	hipFree(d_LO);
	hipFree(d_N);
	hipFree(d_LN);	
	hipFree(d_singlePattern);
	hipFree(d_Extension);
	hipFree(V);
	hipDeviceReset();	

	fout.close();
	delete[] arrayO;
	delete[] arrayN;
	delete[] arrayLO;
	delete[] arrayLN;

	getch();
	return 0;
}
