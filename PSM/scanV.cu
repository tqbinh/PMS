#include "scanV.h"
#include "kernelPrintf.h"

hipError_t scanV(int *V,unsigned int numberElementV,int *index){
	hipError_t cudaStatus;

	
	//Khởi tạo một mảng indexFloat
	float* indexFloat; //mảng này dùng để chứa kết quả trả về của phép scan
   cudaStatus=hipMalloc((float**)&indexFloat,numberElementV*sizeof(float));
	if (cudaStatus != hipSuccess){
		fprintf(stderr,"hipMalloc failed",cudaStatus);
		goto Error;
	}

	float* VFloat; //mảng này dùng để chứa kết quả của phép chuyển từ kiểu int sang float của mảng V
	cudaStatus = hipMalloc((float**)&VFloat,numberElementV*sizeof(float));
	if (cudaStatus != hipSuccess){
		fprintf(stderr,"hipMalloc failed",cudaStatus);
		goto Error;
	}
	
	dim3 block(512);
	dim3 grid((numberElementV+block.x-1)/block.x);


	kernelCastingInt2Float<<<grid,block>>>(VFloat,V,numberElementV);

	

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize failed",cudaStatus);
		goto Error;
	}

	
	
	preallocBlockSums(numberElementV);
	prescanArray(indexFloat,VFloat,numberElementV);

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize prescanArray failed",cudaStatus);
		goto Error;
	}

	printf("\n Scan Result float: ");
	kernelPrintFloat<<<grid,block>>>(VFloat,numberElementV);

	kernelCastingFloat2Int<<<grid,block>>>(index,indexFloat,numberElementV);


	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize kernelCastingFloat2Int failed",cudaStatus);
		goto Error;
	}

	/*printf("\n Scan Result int: ");
	kernelPrintf<<<grid,block>>>(index,numberElementV);

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize kernelPrintf failed",cudaStatus);
		goto Error;
	}

*/

Error:
	hipFree(VFloat);
	hipFree(indexFloat);

	return cudaStatus;
}
