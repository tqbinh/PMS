#include "hip/hip_runtime.h"
#include "getValidExtension.h"

__global__ void kernelGetValidExtension(Extension *d_Extension,int *V,int *index,unsigned int numberElementd_Extension,Extension *d_ValidExtension){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<numberElementd_Extension){
		if(V[i]==1){

			//printf("\nV[%d]:%d, index[%d]:%d,d_Extension[%d], d_Extension[%d]:%d\n",i,V[i],i,index[i],i,i,d_Extension[i].vgi);
			d_ValidExtension[index[i]].li=d_Extension[i].li;
			d_ValidExtension[index[i]].lj=d_Extension[i].lj;
			d_ValidExtension[index[i]].lij=d_Extension[i].lij;
			d_ValidExtension[index[i]].vgi=d_Extension[i].vgi;
			d_ValidExtension[index[i]].vgj=d_Extension[i].vgj;
			d_ValidExtension[index[i]].vi=d_Extension[i].vi;
			d_ValidExtension[index[i]].vj=d_Extension[i].vj;
		}

	}

}



extern "C" inline hipError_t getValidExtension(Extension *d_Extension,int *V,int *index,unsigned int numberElementd_Extension,Extension *d_ValidExtension){
	hipError_t cudaStatus;
	
	//printfExtension(d_Extension,numberElementd_Extension);

	dim3 block(1024);
	dim3 grid((numberElementd_Extension+block.x)/block.x);

	kernelGetValidExtension<<<grid,block>>>(d_Extension,V,index,numberElementd_Extension,d_ValidExtension);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if (cudaStatus != hipSuccess){
		fprintf(stderr,"\nkernelGetValidExtension failed");
		goto Error;
	}

	//printfExtension(d_ValidExtension,16);

Error:
	/*hipFree(d_Extension);
	hipFree(index);
	hipFree(d_ValidExtension);*/
	return cudaStatus;
}

