#include "hip/hip_runtime.h"
#include "getValidForwardExtensionFromTheLastQ.h"

__global__ void kernelPrintd_arr_V(struct_V *d_arr_V,int numberElementOf_d_arr_V){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i<numberElementOf_d_arr_V){
		//if(d_arr_V[i].valid==1){
			printf("\n Thread %d: valid: %d, d_backward: %d",i,d_arr_V[i].valid,d_arr_V[i].backward);
		//}		
	}
}


__global__ void kernelFindValidForwardFromLastQ(struct_Q *device_arr_Q,int indexOfQ,cHistory **dH,int n,int *d_O,int *d_LO,int *d_N,struct_V *d_arr_V,float *d_arr_degreeOfVerticesInQColumn, int maxOfVer,int m){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<n){
		int minLabel = d_LO[device_arr_Q[0]._d_arr_Q[0].vid];
		printf("\n minLabel: %d",minLabel);
		// diplay array dH
		/*		
		//dH[i]->printmn();
		printf("\n dh[%d]->m:%d",i,dH[i]->m);
		printf("\n dh[%d]->n:%d",i,dH[i]->n);
		for (int j = 0; j < dH[i]->n; j++) //display d_arr_HO
		{
		printf("\n dH[%d]->d_arr_HO[%d]:%d",i,j,dH[i]->d_arr_HO[j]);
		}
		for (int j = 0; j < dH[i]->m; j++) //display d_arr_HLN
		{
		printf("\n dH[%d]->d_arr_HLN[%d]:%d",i,j,dH[i]->d_arr_HLN[j]);
		}
		*/
		int vid = device_arr_Q[indexOfQ]._d_arr_Q[i].vid; //lấy vid của cột Q
		//int indexOfPrevQ = device_arr_Q[indexOfQ]._d_arr_Q[i].idx; //Tạm thời không lấy index của Q phía trước
		int degreeVid = __float2int_rn(d_arr_degreeOfVerticesInQColumn[i]); //lấy bậc của vid đó, do bậc là kiểu float nên phải convert sang kiểu int
		printf("\n Thread %d: vid:%d have degree: %d",i,vid,degreeVid);
		//Duyệt qua các đỉnh kề với đỉnh vid dựa vào số lần duyệt là bậc
		int indexToVidIndN=d_O[vid];
		int toVid;
		int labelToVid;
		for (int j = 0; j < degreeVid; j++,indexToVidIndN++) //Duyệt qua tất cả các đỉnh kề với đỉnh vid, nếu đỉnh không thuộc embedding thì --> cạnh cũng không thuộc embedding vì đây là Q cuối
		{			
			toVid=d_N[indexToVidIndN]; //Lấy vid của đỉnh cần kiểm tra
			labelToVid = d_LO[toVid]; //lấy label của đỉnh cần kiểm tra
			//printf("\nThread %d, j: %d has ToVidLabel:%d",i,j,labelToVid);
			//kiểm tra xem đỉnh toVid đã tồn tại trong embedding hay chưa (khác zero là thuộc embedding)
			int indexOfToVidInEmbedding=(toVid%maxOfVer);
			//printf("\n Thread %d, for j: %d, dH[%d]->d_arr_HO[%d]:%d",i,j,i,indexOfToVidInEmbedding,dH[i]->d_arr_HO[indexOfToVidInEmbedding]);
			if(dH[i]->d_arr_HO[indexOfToVidInEmbedding]==0){ //Nếu giá trị tương ứng trên Embedding bằng zero thì xét xem label của nó có thoả lớn hơn hoặc bằng minLabel hay không
				if(labelToVid>=minLabel){ //nếu thoả thì sẽ set mảng V tương ứng là 1 và chỉ định nó là forward
					int indexOfd_arr_V=i*m+j;
					d_arr_V[indexOfd_arr_V].valid=1;					
				}
			}
		}
	}
}


__global__ void find_maximum_kernel(float *array, float *max, int *mutex, unsigned int n)
{
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int stride = gridDim.x*blockDim.x;
	unsigned int offset = 0;

	__shared__ float cache[256];


	float temp = -1.0;
	while(index + offset < n){
		temp = fmaxf(temp, array[index + offset]);

		offset += stride;
	}

	cache[threadIdx.x] = temp;

	__syncthreads();


	// reduction
	unsigned int i = blockDim.x/2;
	while(i != 0){
		if(threadIdx.x < i){
			cache[threadIdx.x] = fmaxf(cache[threadIdx.x], cache[threadIdx.x + i]);
		}

		__syncthreads();
		i /= 2;
	}

	if(threadIdx.x == 0){
		while(atomicCAS(mutex,0,1) != 0);  //lock
		*max = fmaxf(*max, cache[0]);
		atomicExch(mutex, 0);  //unlock
	}
}


__global__ void kernelFindDegreeOfVertex(int *d_O,int *d_N,int numberOfElementd_O,int numberOfElementd_N,struct_Q *device_arr_Q,int indexOfQ,int n,float *d_arr_degreeOfVerticesInQColumn,int maxOfVer){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if(i<n){
		float degreeOfV =0;
		int nextVid;
		int graphid;
		int lastGraphId=(numberOfElementd_O-1)/maxOfVer;
		int vid =device_arr_Q[indexOfQ]._d_arr_Q[i].vid;
		if(d_O[vid]==-1){
			printf("\ndevice_arr_Q is not correct, vertex id %vid is not exist in database");
			return;
		}

		if (vid==numberOfElementd_O-1){ //nếu như đây là đỉnh cuối cùng trong d_O
			degreeOfV=numberOfElementd_N-d_O[vid]; //thì bậc của đỉnh vid chính bằng tổng số cạnh trừ cho giá trị của d_O[vid].
		}
		else
		{
			nextVid = vid+1; //xét đỉnh phía sau có khác 1 hay không?
			graphid=vid/maxOfVer;
			if(d_O[nextVid]==-1 && graphid==lastGraphId){
				degreeOfV=numberOfElementd_N-d_O[vid];
			}
			else if(d_O[nextVid]==-1 && graphid!=lastGraphId){
				nextVid=(graphid+1)*maxOfVer;
				degreeOfV=d_O[nextVid]-d_O[vid];
			}
			else
			{
				degreeOfV=d_O[nextVid]-d_O[vid];
			}							
		}		
		//printf("\nThread:%d : Degree of %d is %f",i,vid,degreeOfV);
		d_arr_degreeOfVerticesInQColumn[i]=degreeOfV;
		//printf("\nThread %d: d_arr_degreeOfVerticesInQColumn[%d]:%f",i,i,d_arr_degreeOfVerticesInQColumn[i]);
	}		
}


hipError_t getValidForwardExtensionFromTheLastQ(struct_Q *device_arr_Q,int indexOfQ,cHistory **dH,int n,unsigned int maxOfVer,int *d_O,int *d_LO,int *d_N,int *d_LN,int numberOfElementd_O,int numberOfElementd_N){
	hipError_t cudaStatus;

	dim3 block(1024);
	dim3 grid((n+block.x-1)/block.x);

	//1. Tìm bậc lớn nhất m của các vid thuộc device_arr_Q[indexOfQ] đang xét.
	//1.1 Khởi tạo một mảng số nguyên có kích thước bằng số lượng embedding
	float *d_arr_degreeOfVerticesInQColumn;
	cudaStatus = hipMalloc((void**)&d_arr_degreeOfVerticesInQColumn,n*sizeof(float));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc d_arr_degreeOfVerticeInQColumn failed");
		goto Error;
	}
	else
	{
		hipMemset(d_arr_degreeOfVerticesInQColumn,0,n*sizeof(float));
	}

	//1.2 Tính bậc của các đỉnh vid trong Q column và lưu vào d_arr_OfVerticeInQColumn
	kernelFindDegreeOfVertex<<<grid,block>>>(d_O,d_N,numberOfElementd_O,numberOfElementd_N,device_arr_Q,indexOfQ,n,d_arr_degreeOfVerticesInQColumn,maxOfVer);

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaDeviceSynchronize kernelFindDegreeOfVertex failed");
		goto Error;
	}

	//2. Tìm bậc lớn nhất của vid trong Q column chính là tìm giá trị lớn nhất trong mảng d_arr_degreeOfVerticesInQColumn
	float *h_max;
	h_max = (float*)malloc(sizeof(float));
	if(h_max==NULL){
		printf("\nMalloc h_max failed");
		exit(1);
	}

	float *d_max;
	int *d_mutex;
	cudaStatus=hipMalloc((void**)&d_max,sizeof(float));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc d_max failed");
		goto Error;
	}
	else
	{
		hipMemset(d_max,0,sizeof(float));
	}

	cudaStatus=hipMalloc((void**)&d_mutex,sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc d_mutex failed");
		goto Error;
	}
	else
	{
		hipMemset(d_mutex,0,sizeof(int));
	}

	dim3 gridSize = 256;
	dim3 blockSize = 256;
	find_maximum_kernel<<<gridSize, blockSize>>>(d_arr_degreeOfVerticesInQColumn, d_max, d_mutex, n);

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaDeviceSynchronize find_maximum_kernel failed");
		goto Error;
	}

	// copy from device to host
	hipMemcpy(h_max, d_max, sizeof(float), hipMemcpyDeviceToHost);

	//report results
	int m = (int)(*h_max); //bậc lớn nhất của các đỉnh trong 1 cột Q
	printf("\nMax degree of vid in Q column is: %d",m);

	/*
	//3. Tạo mảng d_arr_V có kích thước: maxDegree_vid_Q * |Q|
			Lưu ý, mảng d_arr_V phải có dạng cấu trúc đủ thể hiện cạnh mở rộng có hợp lệ hay không và là forward extension hay backward extension.
			struct struct_V
			{
				int valid; //default: 0, valid: 1
				int backward; //default: 0- forward; backward: 1
			}
			*/
	struct_V *d_arr_V;
	int numberElementOf_d_arr_V=m*n;
	cudaStatus=hipMalloc((void**)&d_arr_V,numberElementOf_d_arr_V*sizeof(struct_V));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc d_arr_V failed");
		goto Error;
	}
	else
	{
		hipMemset(d_arr_V,0,numberElementOf_d_arr_V*sizeof(struct_V));
	}

	/*
	//4. Tìm các mở rộng của vid và đánh dấu những mở rộng hợp lệ vào mảng d_arr_V
		o Bậc của các đỉnh trong Q column được lưu trữ trong mảng d_arr_degreeOfVerticesInQColumn--> chúng ta không cần tính bậc của vid
		o cHistory được lưu trữ trong dH là một cấu trúc gồm mảng d_HO và d_HLN cho biết cạnh và đỉnh đã thuộc embedding
		o Thread thứ i sẽ sử dụng các phần tử tương ứng index_d_arr_V từ [i*m,(i+1)*m - 1]
		o Mỗi lần lặp bậc của vid thì biến tạm sẽ tăng lên 1 để chỉ vùng nhớ tương ứng trên d_arr_V
		o Nếu đỉnh phải cùng của DFS_Code kết nối trực tiếp với đỉnh đầu tiên của DFS_Code thì không tồn tại backward edge (chỉ đúng trong đơn đồ thị vô hướng).
	*/
	kernelFindValidForwardFromLastQ<<<grid,block>>>(device_arr_Q,indexOfQ,dH,n,d_O,d_LO,d_N,d_arr_V,d_arr_degreeOfVerticesInQColumn,maxOfVer,m);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaDeviceSynchronize kernelFindValidForwardFromLastQ failed");
		goto Error;
	}
	
	//Hiển thị kết quả mảng d_arr_V với số lượng phần tử numberElementOf_d_arr_V
	kernelPrintd_arr_V<<<1,numberElementOf_d_arr_V>>>(d_arr_V,numberElementOf_d_arr_V);
	
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaDeviceSynchronize getValidExtensionFromEmbedding failed");
		goto Error;
	}
Error:

	return cudaStatus;
}
