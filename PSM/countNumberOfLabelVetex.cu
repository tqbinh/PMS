#include "countNumberOfLabelVetex.h"
#include <iostream>
	//1.Cấp phát một mảng số nguyên có kích thước bằng với kích thước mảng d_LO gọi là d_Lv
	//2.Cấp phát |d_LO| threads
	//3.thread thứ i sẽ đọc giá trị nhãn tại vị trí d_LO[i], rồi ghi 1 vào mảng d_Lv[d_LO[i]]
	//4. Reduction mảng d_Lv để thu được các nhãn phân biệt

hipError_t countNumberOfLabelVetex(int* d_LO,unsigned int sizeOfArrayLO, unsigned int &numberOfSaperateVertex){
	hipError_t cudaStatus;
	numberOfSaperateVertex=0;
	size_t nBytesLv = sizeOfArrayLO*sizeof(int);
	//cấp phát mảng d_Lv trên device
	int *d_Lv;
	cudaStatus=hipMalloc((int**)&d_Lv,nBytesLv);
		if(cudaStatus!=hipSuccess){
			fprintf(stderr,"hipMalloc d_Lv failed");
			goto Error;
		}
		else
		{
			hipMemset(d_Lv,0,nBytesLv);
		}

		//Cấp phát threads
		dim3 block(32);
		dim3 grid((sizeOfArrayLO+block.x-1)/block.x);
		kernelCountNumberOfLabelVertex<<<grid,block>>>(d_LO,d_Lv,sizeOfArrayLO);
		
		hipDeviceSynchronize();
		printf("\nElements of d_Lv:");
		kernelPrintf<<<grid,block>>>(d_Lv,sizeOfArrayLO);

		int* h_Lv=NULL;
		h_Lv=(int*)malloc(nBytesLv);
		if(h_Lv==NULL){
			printf("h_Lv malloc memory fail");
			exit(1);
		}
		hipMemcpy(h_Lv,d_Lv,nBytesLv,hipMemcpyDeviceToHost);
		cudaStatus=hipDeviceSynchronize();
		if(cudaStatus!=hipSuccess){
			fprintf(stderr,"hipDeviceSynchronize fail",cudaStatus);
			goto Error;
		}
		int result=0;
		sumUntilReachZero(h_Lv,sizeOfArrayLO,result);
		numberOfSaperateVertex=result;
		/*printf("Number of label is: %d ;",numberOfSaperateVertex);*/
	//unsigned int size=sizeOfArrayLO;

	//// execution configuration
	//int blocksize = 3; // initial block size
	//
	// block.x=blocksize;
	// grid.x=(size+block.x-1)/block.x;
	//printf("grid %d block %d\n",grid.x, block.x);
	//// allocate host memory
	//size_t bytes = size * sizeof(int);
	//int *h_odata = (int *) malloc(grid.x*sizeof(int));
	//if (h_odata == NULL) {
	//	printf("\nMallocation memory h_odata failure\n");
	//	exit(1);
	//}
	//else
	//{
	//	memset(h_odata,0,grid.x*sizeof(int));
	//}



	//// allocate device memory
	//int *d_odata = NULL;
	//hipMalloc((void **) &d_odata, grid.x*sizeof(int));

	//kernelReduce<<<grid, block>>>(d_Lv, d_odata, size);
	//hipDeviceSynchronize();
	//cudaStatus=hipGetLastError();
	//if(cudaStatus!= hipSuccess){
	//	fprintf(stderr,"hipDeviceSynchronize returned error code %d after launching addKernel!\n",cudaStatus );
	//	goto Error;
	//}


	//hipMemcpy(h_odata, d_odata, grid.x*sizeof(int), hipMemcpyDeviceToHost);

	//int gpu_sum = 0;
	//for (int i=0; i<grid.x; i++) gpu_sum += h_odata[i];
	//printf("\ngpu Neighbored gpu_sum: %d <<<grid %d block %d>>>\n",gpu_sum,grid.x,block.x);
	//hipDeviceSynchronize();

	//	printf("\nElements of d_Lv after reduction:");
	//	kernelPrintf<<<1,32>>>(d_Lv,sizeOfArrayLO);

	//	//kernelPrintf<<<grid,block>>>(result,1);


Error:
	hipFree(d_Lv);
	/// free host memory
	//free(h_odata);
	// free device memory
	//hipFree(d_odata);
	return cudaStatus;
	return cudaStatus;
}
