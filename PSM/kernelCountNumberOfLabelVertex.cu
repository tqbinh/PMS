#include "hip/hip_runtime.h"
#include "countNumberOfLabelVetex.h"
__global__ void kernelCountNumberOfLabelVertex(int *d_LO,int *d_Lv,unsigned int sizeOfArrayLO){
	int i= blockDim.x*blockIdx.x + threadIdx.x;
	if(i<sizeOfArrayLO){
		if(d_LO[i]!=-1){
			d_Lv[d_LO[i]]=1;
		}
	}
}
