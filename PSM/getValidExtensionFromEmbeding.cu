#include "getExtensionFromEmbedding.h"

hipError_t getValidExtensionFromEmbeding(Extension *&d_arrE,int &numberElement_d_arrE,struct_Q *device_arr_Q,int indexOfQ,cHistory **dH,int n,unsigned int maxOfVer,int *d_O,int *d_LO,int *d_N,int *d_LN,int numberOfElementd_O,int numberOfElementd_N,int lastColumn){
	hipError_t cudaStatus;
	//Có bao nhiêu embedding n thì tạo bấy nhiêu thread để xử lý embedding tương ứng
	//printf("\n number of embedding:%d",n);
	//dim3 block(1024);
	//dim3 grid((n+block.x-1)/block.x);
	
	//cần tạo ra bao nhiêu Ext để lưu kết quả trả về và mỗi Ext có kích thước là bao nhiêu

	if(indexOfQ==lastColumn){
		//printf("\nThis is the last Q column");
		getValidForwardExtensionFromTheLastQ(d_arrE,numberElement_d_arrE,device_arr_Q,indexOfQ,dH,n,maxOfVer,d_O,d_LO,d_N,d_LN,numberOfElementd_O,numberOfElementd_N);
	}
	else
	{
		printf("\nThis is not last Q column");
	}



	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaDeviceSynchronize getValidExtensionFromEmbedding failed");
		goto Error;
	}
Error:

	return cudaStatus;
}
