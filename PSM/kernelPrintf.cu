#include "hip/hip_runtime.h"
#include "kernelPrintf.h"


//__device__ void __syncthreads(void);
__global__ void kernelPrintf(int *O,int sizeO){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	
	if(i<sizeO){
		printf("[%d]:%d ; ",i,O[i]);
	}

}
