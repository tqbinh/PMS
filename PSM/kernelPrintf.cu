#include "hip/hip_runtime.h"
#include "kernelPrintf.h"


//__device__ void __syncthreads(void);
__global__ void kernelPrintf(int *O,int sizeO){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	
	if(i<sizeO){			
		printf("[%d]:%d ; ",i,O[i]);
	}

}


hipError_t printInt(int* d_array,int noElem_d_Array){
	hipError_t cudaStatus;

	dim3 block(1024);
	dim3 grid((noElem_d_Array+block.x-1)/block.x);

	kernelPrintf<<<grid,block>>>(d_array,noElem_d_Array);
	hipDeviceSynchronize();

	cudaStatus=hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"\nkernelPrintExtention failed");
		goto Error;
	}
Error:
	
	return cudaStatus;
}



__global__ void kernelPrintFloat(float* A,int n){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if (i<n){
		printf("[%d]:%.0f ;",i,A[i]);
	}

}

hipError_t printFloat(float* d_array,int numberElementOfArray){
	hipError_t cudaStatus;

	dim3 block(1024);
	dim3 grid((numberElementOfArray+block.x-1)/block.x);

	kernelPrintFloat<<<grid,block>>>(d_array,numberElementOfArray);
	hipDeviceSynchronize();

	cudaStatus=hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"\nkernelPrintExtention failed");
		goto Error;
	}
Error:
	
	return cudaStatus;
}



__global__ void kernelPrintExtention(Extension *d_Extension,unsigned int n){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if (i<n){		
		printf("\n[%d]: DFS code:(%d,%d,%d,%d,%d)  (vgi,vgj):(%d,%d)\n",i,d_Extension[i].vi,d_Extension[i].vj,d_Extension[i].li,d_Extension[i].lij,d_Extension[i].lj,d_Extension[i].vgi,d_Extension[i].vgj);
	}

}


hipError_t printfExtension(Extension *d_E,unsigned int noElem_d_E){
	hipError_t cudaStatus;

	dim3 block(1024);
	dim3 grid((noElem_d_E+block.x-1)/block.x);

	kernelPrintExtention<<<grid,block>>>(d_E,noElem_d_E);
	hipDeviceSynchronize();

	cudaStatus=hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"\nkernelPrintExtention failed");
		goto Error;
	}
Error:
	
	return cudaStatus;
}

__global__ void kernelPrintEmbedding(struct_Embedding *d_Embedding,int noElem_Embedding){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if (i<noElem_Embedding){		
		printf("\n[%d]: (idx:%d, vid:%d)",i,d_Embedding[i].idx,d_Embedding[i].vid);
	}
}


hipError_t printEmbedding(struct_Embedding *d_Embedding,int noElem_Embedding){
	hipError_t cudaStatus;

	dim3 block(1024);
	dim3 grid((noElem_Embedding+block.x-1)/block.x);

	kernelPrintEmbedding<<<grid,block>>>(d_Embedding,noElem_Embedding);
	hipDeviceSynchronize();

	cudaStatus=hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"\nkernelPrintEmbedding failed");
		goto Error;
	}
Error:
	
	return cudaStatus;
}
