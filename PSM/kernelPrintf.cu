#include "hip/hip_runtime.h"
#include "kernelPrintf.h"


//__device__ void __syncthreads(void);
__global__ void kernelPrintf(int *O,int sizeO){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	
	if(i<sizeO){			
		printf("[%d]:%d ; ",i,O[i]);
	}

}



__global__ void kernelPrintFloat(float* A,int n){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if (i<n){
		printf("[%d]:%.0f ;",i,A[i]);
	}

}