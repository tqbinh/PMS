#include "hip/hip_runtime.h"
#include "kernelPrintf.h"


//__device__ void __syncthreads(void);
__global__ void kernelPrintf(int *O,int sizeO){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	
	if(i<sizeO){			
		printf("[%d]:%d ; ",i,O[i]);
	}

}


hipError_t printInt(int* d_array,int noElem_d_Array){
	hipError_t cudaStatus;

	dim3 block(1024);
	dim3 grid((noElem_d_Array+block.x-1)/block.x);

	kernelPrintf<<<grid,block>>>(d_array,noElem_d_Array);
	hipDeviceSynchronize();

	cudaStatus=hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"\nkernelPrintInt failed");
		goto Error;
	}
Error:
	
	return cudaStatus;
}

__global__ void kernelprintUnsignedInt(unsigned int *O,int sizeO){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	
	if(i<sizeO){			
		printf("[%d]:%d ; ",i,O[i]);
	}

}

inline hipError_t printUnsignedInt(unsigned int* d_array,int noElem_d_Array){
	hipError_t cudaStatus;

	dim3 block(1024);
	dim3 grid((noElem_d_Array+block.x-1)/block.x);

	kernelprintUnsignedInt<<<grid,block>>>(d_array,noElem_d_Array);
	hipDeviceSynchronize();

	cudaStatus=hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"\nkernelPrintInt failed");
		goto Error;
	}
Error:
	
	return cudaStatus;
}



__global__ void kernelPrintFloat(float* A,int n){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if (i<n){
		printf("[%d]:%.0f ;",i,A[i]);
	}

}

hipError_t printFloat(float* d_array,int numberElementOfArray){
	hipError_t cudaStatus;

	dim3 block(1024);
	dim3 grid((numberElementOfArray+block.x-1)/block.x);

	kernelPrintFloat<<<grid,block>>>(d_array,numberElementOfArray);
	hipDeviceSynchronize();

	cudaStatus=hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"\nkernelPrintExtention failed");
		goto Error;
	}
Error:
	
	return cudaStatus;
}



__global__ void kernelPrintExtention(Extension *d_Extension,unsigned int n){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if (i<n){		
		printf("\n[%d]: DFS code:(%d,%d,%d,%d,%d)  (vgi,vgj):(%d,%d)\n",i,d_Extension[i].vi,d_Extension[i].vj,d_Extension[i].li,d_Extension[i].lij,d_Extension[i].lj,d_Extension[i].vgi,d_Extension[i].vgj);
	}

}


hipError_t printfExtension(Extension *d_E,unsigned int noElem_d_E){
	hipError_t cudaStatus;

	dim3 block(1024);
	dim3 grid((noElem_d_E+block.x-1)/block.x);

	kernelPrintExtention<<<grid,block>>>(d_E,noElem_d_E);
	hipDeviceSynchronize();

	cudaStatus=hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"\nkernelPrintExtention failed");
		goto Error;
	}
Error:
	
	return cudaStatus;
}


__global__ void kernelPrintUniEdge(UniEdge *d_UniqueEdge,unsigned int n){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if (i<n){		
		int li = d_UniqueEdge[i].li;
		int lij = d_UniqueEdge[i].lij;
		int lj = d_UniqueEdge[i].lj;
		printf("\n Edge %d: (%d,%d,%d)",i,li,lij,lj);		
	}
}


hipError_t printfUniEdge(UniEdge *d_E,unsigned int noElem_d_E){
	hipError_t cudaStatus;

	dim3 block(1024);
	dim3 grid((noElem_d_E+block.x-1)/block.x);

	kernelPrintUniEdge<<<grid,block>>>(d_E,noElem_d_E);
	hipDeviceSynchronize();

	cudaStatus=hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"\nkernelPrintUniEdge failed");
		goto Error;
	}
Error:
	
	return cudaStatus;
}

__global__ void kernelPrintEmbedding(struct_Embedding *d_Embedding,int noElem_Embedding){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if (i<noElem_Embedding){		
		printf("\n[%d]: (idx:%d, vid:%d)",i,d_Embedding[i].idx,d_Embedding[i].vid);
	}
}


hipError_t printEmbedding(struct_Embedding *d_Embedding,int noElem_Embedding){
	hipError_t cudaStatus;

	dim3 block(1024);
	dim3 grid((noElem_Embedding+block.x-1)/block.x);

	kernelPrintEmbedding<<<grid,block>>>(d_Embedding,noElem_Embedding);
	hipDeviceSynchronize();

	cudaStatus=hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"\nkernelPrintEmbedding failed");
		goto Error;
	}
Error:
	
	return cudaStatus;
}
