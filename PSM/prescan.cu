#include "hip/hip_runtime.h"
#include "prescan.h"

//__global__ void prescan(int *g_odata,int *g_idata,unsigned int n){
//	extern __shared__ int temp[];  // allocated on invocation  
//	int thid = threadIdx.x;  
//	int offset = 1;
//	temp[2*thid] = g_idata[2*thid]; // load input into shared memory  
//	temp[2*thid+1] = g_idata[2*thid+1];  
//	for (int d = n>>1; d > 0; d >>= 1)     // build sum in place up the tree  
//	{   
//		__syncthreads();  
//		if (thid < d)  
//		{  
//			int ai = offset*(2*thid+1)-1;  
//			int bi = offset*(2*thid+2)-1;  
//			temp[bi] += temp[ai];  
//		}  
//		offset *= 2; 
//	}		
//		if (thid == 0) { temp[n - 1] = 0; } // clear the last element  
//	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan  
//	{  
//		offset >>= 1;  
//		__syncthreads();  
//		if (thid < d)                       
//		{  
//			int ai = offset*(2*thid+1)-1;  
//			int bi = offset*(2*thid+2)-1;  
//			float t = temp[ai];  
//			temp[ai] = temp[bi];  
//			temp[bi] += t;   
//		}  
//	}  
//	__syncthreads(); 
//	g_odata[2*thid] = temp[2*thid]; // write results to device memory  
//	g_odata[2*thid+1] = temp[2*thid+1];  
//
//}

__global__ void scan_bel(int* inputarray,int loop,int* outputarray,int number)
{
	unsigned int thIdx = blockIdx.x * blockDim.x + threadIdx.x;

	int divisor = 2;
	int adder = 1;
	int temp;

	for(int i=0;i<loop;i++)
	{
		if(thIdx%(divisor) == divisor-1)
		{
			outputarray[thIdx] = outputarray[thIdx-adder]+outputarray[thIdx];
		}
		__syncthreads();
		divisor*=2;
		adder*=2;
	}

	divisor = number;
	adder = divisor/2;

	outputarray[number-1] = 0;
	for(int i=0;i<loop;i++)
	{
		if(thIdx%(divisor) == divisor-1)
		{
			temp = outputarray[thIdx];
			outputarray[thIdx] = outputarray[thIdx-adder]+outputarray[thIdx];
			outputarray[thIdx-adder] = temp;
		}
		__syncthreads();
		divisor/=2;
		adder/=2;
	}
}