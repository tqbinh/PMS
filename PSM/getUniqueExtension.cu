#include "hip/hip_runtime.h"
#include "getUniqueExtension.h"
//#include "kernelPrintf.h"
__global__ void kernelGetUniqueExtension(Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,unsigned int Lv,unsigned int Le,int *d_allPossibleExtension){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<noElem_d_ValidExtension){
		int index=	d_ValidExtension[i].li*Lv*Le + d_ValidExtension[i].lij*Lv + d_ValidExtension[i].lj;
		d_allPossibleExtension[index]=1;
	}

}



hipError_t getUniqueExtension(Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,unsigned int Lv,unsigned int Le,int *d_allPossibleExtension){
	hipError_t cudaStatus;
	
	dim3 block(1024);
	dim3 grid((noElem_d_ValidExtension+block.x-1)/block.x);

	kernelGetUniqueExtension<<<grid,block>>>(d_ValidExtension,noElem_d_ValidExtension,Lv,Le,d_allPossibleExtension);
	hipDeviceSynchronize();

	
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize failed");
		goto Error;
	}

Error:
	return cudaStatus;
}
