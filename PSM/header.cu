#include "hip/hip_runtime.h"
#pragma once
#include "header.h"

//kernel khởi tạo bộ nhớ và tạo nội dung cho dQ
__global__ void kernelInitializeDataEmbedding(Embedding *dQ,int sizedQ){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<sizedQ){
		dQ[i].idx=i;
		dQ[i].vid=i+100;
	}

}

//Hàm khởi tạo bộ nhớ và tạo nội dung cho dQ
inline hipError_t createEmbeddingElement(Embedding *&dQ,int sizedQ,int &first){
	hipError_t cudaStatus;

	//Khởi tạo bộ nhớ cho dQ1 trên device
	size_t nBytes = sizedQ*sizeof(Embedding);
	cudaStatus=hipMalloc((void**)&dQ,nBytes);
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc dQ of createEmbeddingElement failed",cudaStatus);
		goto Error;
	}
	if(first==0)
	{
		hipMemset(dQ,-1,nBytes);
		++first;
		return cudaStatus;
	}

	//Khởi tạo dữ liệu bất kỳ cho dQ
	dim3 block(blocksize);
	dim3 grid((sizedQ + block.x - 1)/block.x);
	kernelInitializeDataEmbedding<<<grid,block>>>(dQ,sizedQ);



	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize of createEmbeddingElement failed",cudaStatus);
		goto Error;
	}
Error:

	return cudaStatus;
}


__global__ void kernelPrint(Embedding *dQ,int sizedQ){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<sizedQ){

		printf("\n Thread %d: %p (idx:%d,vid:%d) (%p,%p)",i,dQ,dQ[i].idx,dQ[i].vid,&(dQ[i].idx),&(dQ[i].vid));
	}
}


//Hàm in nội dung Embedding *dQ khi biết kích thước
inline hipError_t print(Embedding *dQ,int sizedQ){
	hipError_t cudaStatus;

	dim3 block(blocksize);
	dim3 grid((sizedQ + block.x - 1)/block.x);

	kernelPrint<<<grid,block>>>(dQ,sizedQ);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize of createEmbeddingElement failed",cudaStatus);
		goto Error;
	}


Error:

	return cudaStatus;
}

//kernel In phần tử Embedding **pdQ
__global__ void kernelPrint(Embedding **pdQ,int *d_arrSizedQ,int *d_arrPrevQ,int sizepdQ){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < sizepdQ){
		printf("\n Thread %d: Value of pdQ:%p",i,pdQ+i);
		Embedding *dQ = pdQ[i];
		//printf("\n Thread %d: PrevQ: %d",i,d_arrPrevQ[i]);
		int prevQ = d_arrPrevQ[i];
		for (int j = 0; j < d_arrSizedQ[i]; j++)
		{
			printf("\n i=%d %p PrevQ:%d (idx:%d, vid:%d) ",i,dQ,prevQ,dQ[j].idx,dQ[j].vid);
		}
	}
}

//Hàm in phần tử Embedding **pdQ khi biết kích thước của dQ trong mảng h_arrSizedQ tương ứng
inline hipError_t print(Embedding **pdQ,int *h_arrSizedQ,int *d_arrPrevQ,int sizepdQ){
	hipError_t cudaStatus;
	dim3 block(blocksize);
	dim3 grid((sizepdQ+block.x-1)/block.x);
	printf("\n\n Array pdQ:\n");
	kernelPrint<<<grid,block>>>(pdQ,h_arrSizedQ,d_arrPrevQ,sizepdQ);
	printf("\n");
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize of kernelPrint failed",cudaStatus);
		goto Error;
	}


Error:

	return cudaStatus;
}

//kernel lấy pointer của dQ lưu vào pdQ
__global__ void kernelgetPointer(Embedding **pdQ,Embedding *dQ){
	*pdQ=dQ;

}


__global__ void kernelCopyEmbedding(Embedding **pdQ,int sizepdQ,Embedding **d_temp){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<sizepdQ){
		d_temp[i]=pdQ[i];
	}
}

__global__ void kernelPrintDoubleEmbedding(Embedding **d_temp,int newsize){
	int i=blockDim.x * blockIdx.x + threadIdx.x;
	if (i<newsize){
		printf("\n Thread %d: %p",i,d_temp[i]);

	}

}

__global__ void kernelCopyLastEmbedding(Embedding **d_temp,Embedding *dQ,int newsize){
	d_temp[newsize-1]=dQ;
}


//Hàm lấy pointer của phần tử Embedding *dQ bằng hàm hipMemcpy
inline hipError_t getPointer(Embedding **&pdQ,int &sizepdQ,Embedding *dQ){
	hipError_t cudaStatus;

	//
	int currentsize = sizepdQ;
	int newsize = ++sizepdQ;
	if (currentsize==0){
		cudaStatus=hipMalloc((void**)&pdQ,newsize*sizeof(Embedding*));
		if(cudaStatus!=hipSuccess){
			fprintf(stderr,"\n hipMalloc pdQ failed",cudaStatus);
			goto Error;
		}
		else
		{
			kernelgetPointer<<<1,1>>>(pdQ,dQ);
			hipDeviceSynchronize();
		}
		goto Error;
	}

	//Khởi tạo mảng tạm 
	Embedding **d_temp=nullptr;
	size_t nBytes=newsize*sizeof(Embedding*);
	cudaStatus=hipMalloc((void**)&d_temp,nBytes);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc d_temp in getPointer failed",cudaStatus);
		goto Error;
	}
	else
	{
		hipMemset(d_temp,0,nBytes);
	}

	//chép mảng hiện tại 
	kernelCopyEmbedding<<<1,currentsize>>>(pdQ,currentsize,d_temp);
	hipDeviceSynchronize();

	//chép phần tử cần thêm vào cuối mảng d_temp
	kernelCopyLastEmbedding<<<1,1>>>(d_temp,dQ,newsize);
	hipDeviceSynchronize();

	//Hiển thị nội dung mảng d_temp

	kernelPrintDoubleEmbedding<<<1,sizepdQ>>>(d_temp,sizepdQ);
	hipDeviceSynchronize();


	//Cấp phát lại bộ nhớ cho mảng chính với kích thước lớn hơn 1 và chép mảng d_temp vào mảng chính
	hipFree(pdQ);

	cudaStatus=hipMalloc((void**)&pdQ,nBytes);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc pdQ in getPointer failed",cudaStatus);
		goto Error;
	}
	else
	{
		hipMemset(pdQ,-1,nBytes);
	}

	kernelCopyEmbedding<<<1,sizepdQ>>>(d_temp,sizepdQ,pdQ);
	hipDeviceSynchronize();

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize kernelgetPointer of getPointer failed",stderr);
		goto Error;
	}

Error:

	return cudaStatus;
}

//kernel chép dữ liệu kiểu int từ device sang device
__global__ void kernelCopyInt(int *d_arrSizedQ,int *d_tempArrSizedQ,int currentSize){
	int i=blockIdx.x * blockDim.x + threadIdx.x;
	if(i<currentSize){
		d_tempArrSizedQ[i]=d_arrSizedQ[i];
	}
}


__global__ void kernelCopyLastInt(int *temp,int *d_tempArrSizedQ,int newsize){
	d_tempArrSizedQ[newsize-1]=*temp;
}


inline hipError_t copyDeviceToDeviceInt(int *d_FromIntArray,int *d_ToIntArray,int size){
	hipError_t cudaStatus;

	dim3 block(blocksize);
	dim3 grid((size + block.x)/block.x);

	kernelCopyInt<<<grid,block>>>(d_FromIntArray,d_ToIntArray,size);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize in copyDeviceToDeviceInt failed",stderr);
		goto Error;
	}

Error:

	return cudaStatus;
}


//Trả về mảng kích thước là một h_arrSizedQ trên device
inline hipError_t getSizedQ(int *&d_arrSizedQ,int &sized_arrSizedQ,int sizedQ){
	hipError_t cudaStatus;

	//Mở rộng kích thước mảng d_arrSizedQ
	int currentSize = sized_arrSizedQ;
	int newsize =++sized_arrSizedQ;
	if(currentSize==0){
		cudaStatus = hipMalloc((void**)&d_arrSizedQ,newsize*sizeof(int));
		if(cudaStatus!=hipSuccess){
			fprintf(stderr,"\n hipMalloc d_arrSizedQ in getPointer failed",cudaStatus);
			goto Error;
		}
		else
		{
			hipMemcpy(d_arrSizedQ,&sizedQ,sizeof(int),hipMemcpyHostToDevice);
		}


		goto Error;
	}


	size_t nBytes = newsize*sizeof(int);
	int *d_tempArrSizedQ;
	cudaStatus=hipMalloc((void**)&d_tempArrSizedQ,nBytes);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc d_tempArrSizedQ in getPointer failed",cudaStatus);
		goto Error;
	}
	else
	{
		hipMemset(d_tempArrSizedQ,-1,nBytes);
	}

	//Chép mảng cũ qua mảng mới
	cudaStatus = copyDeviceToDeviceInt(d_arrSizedQ,d_tempArrSizedQ,currentSize);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n copyDeviceToDeviceInt in getSizedQ failed");
		goto Error;
	}



	//Kiểm tra thử kết quả trên mảng tạm
	//print(d_tempArrSizedQ,currentSize);

	//Tạo một biết temp để cấp phát phần tử kiểu int trên device và chép sizedQ sang biến tạm
	int * temp;
	cudaStatus=hipMalloc((void**)&temp,sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc temp in getPointer failed",cudaStatus);
		goto Error;
	}
	else
	{
		hipMemcpy(temp,&sizedQ,sizeof(int),hipMemcpyHostToDevice);
	}


	kernelCopyLastInt<<<1,1>>>(temp,d_tempArrSizedQ,newsize);
	hipDeviceSynchronize();

	hipFree(d_arrSizedQ);
	hipMalloc((void**)&d_arrSizedQ,nBytes);
	hipMemset(d_arrSizedQ,0,nBytes);


	copyDeviceToDeviceInt(d_tempArrSizedQ,d_arrSizedQ,newsize);
	/*
	printf("\n\n value of d_arrSizedQ array on device\n");
	print(d_arrSizedQ,newsize);*/

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize getPointer failed",cudaStatus);
		goto Error;
	}

Error:
	return cudaStatus;

}

//kernel in mảng kiểu int trên device
__global__ void kernelPrintInt(int *dArray,int sizedArray){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<sizedArray){
		printf("\n Thread %d: dArray:%d",i,dArray[i]);
	}
}

//Hàm in mảng kiểu int trên device
inline hipError_t print(int *dArray,int sizedArray){
	hipError_t cudaStatus;

	dim3 block(blocksize);
	dim3 grid((sizedArray + block.x -1)/block.x);
	kernelPrintInt<<<grid,block>>>(dArray,sizedArray);

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize getPointer failed",cudaStatus);
		goto Error;
	}

Error:
	return cudaStatus;
}

//Tạo và cập nhật ColumnQ
hipError_t makeColumnQ(Embedding *dQ,int sizedQ,Embedding **&pdQ,int &sizepdQ,int *&d_arrSizedQ,int &sized_arrSizedQ,int *&d_arrPrevQ,int &sized_arrPrevQ,int iPrevQ,int &first){
	hipError_t cudaStatus;

	//Tạo nội dung cho các phần tử của dQ

	cudaStatus=createEmbeddingElement(dQ,sizedQ,first);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n cudaStatuscreateEmbeddingElement in makeColumnQ failed",stderr);
		goto Error;
	}

	////In nội dung dQ
	//cudaStatus=print(dQ,sizedQ);
	//if(cudaStatus!=hipSuccess){
	//	fprintf(stderr,"\n print of kernel.cu failed",stderr);
	//	goto Error;
	//}


	//Lấy con trỏ của dQ lưu vào pdQ
	cudaStatus = getPointer(pdQ,sizepdQ,dQ);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n getPointer in makeColumnQ failed",stderr);
		goto Error;
	}

	//Lấy kích thước của dQ lưu vào mảng d_arrSizedQ
	cudaStatus = getSizedQ(d_arrSizedQ,sized_arrSizedQ,sizedQ);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n getSizedQ  in makeColumnQ failed",stderr);
		goto Error;
	}

	//Lấy prevQ
	cudaStatus = getSizedQ(d_arrPrevQ,sized_arrPrevQ,iPrevQ);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n getSizedQ of kernel.cu failed",stderr);
		goto Error;
	}

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() in makeColumnQ failed");
		goto Error;
	}

Error:
	return cudaStatus;
}


//kernel in nội dung embedding thứ i.
__global__ void kernelPrintEmbedding(Embedding **pdQ,int *d_arrSizedQ,int *d_arrPrevQ,int sizepdQ,int firstEmbedding,int lastColumnQ){
	Embedding *dQ = pdQ[lastColumnQ];
	int vid = dQ[firstEmbedding].vid;
	int idx = dQ[firstEmbedding].idx;
	int prevQ = d_arrPrevQ[lastColumnQ];	
	printf("\n Q%d: (idx:%d, vid:%d) prevQ:%d",lastColumnQ,idx,vid,prevQ);
	while (true)
	{
		dQ=pdQ[prevQ];
		vid = dQ[idx].vid;
		idx = dQ[idx].idx;		
		printf("\n Q%d: (idx:%d, vid:%d)",prevQ,idx,vid);	

		prevQ = d_arrPrevQ[prevQ];
		if(prevQ==-1){ 
			printf("\nEnd of Embedding\n");
			return;
		}

	}
}

//In embedding thứ i. Cần phải biết cột Q cuối để truy xuất Embedding ngược về phía trước
inline hipError_t printEmbedding(Embedding **pdQ,int *d_arrSizedQ,int *d_arrPrevQ,int sizepdQ,int firstEmbedding,int lastColumnQ){
	hipError_t cudaStatus;

	kernelPrintEmbedding<<<1,1>>>(pdQ,d_arrSizedQ,d_arrPrevQ,sizepdQ,firstEmbedding,lastColumnQ);

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() in printEmbedding failed",cudaStatus);
		goto Error;
	}
Error:
	return cudaStatus;
}

__global__ void kernelSetValueFordQ(Extension *d_ValidExtension,int noElem_d_ValidExtension,Embedding *dQ1,Embedding *dQ2,int *d_scanResult,int li,int lij,int lj){
	int i = blockDim.x *blockIdx.x +threadIdx.x;
	if(i<noElem_d_ValidExtension){
		if(d_ValidExtension[i].li==li &&d_ValidExtension[i].lij == lij && d_ValidExtension[i].lj){
			dQ1[d_scanResult[i]].idx=-1;
			dQ1[d_scanResult[i]].vid=d_ValidExtension[i].vgi;

			dQ2[d_scanResult[i]].idx=d_scanResult[i];
			dQ2[d_scanResult[i]].vid=d_ValidExtension[i].vgj;
		}

	}

}


inline hipError_t createEmbeddingRoot(Embedding **&dArrPointerEmbedding,int &noElem_dArrPointerEmbedding,int *&dArrSizedQ,int &noElem_dArrSizedQ,int *&dArrPrevQ,int &noElem_dArrPrevQ,Extension *d_ValidExtension,int noElem_d_ValidExtension,int li,int lij,int lj){
	hipError_t cudaStatus;

	//Vì đây là lần đầu tiên tạo Embedding, chúng ta tạo 2 cột Q có kích thước bằng nhau và bằng số lượng Embedding tìm thấy trong d_ValidExtension của nhãn cạnh (li,lij,lj)
	//Tạo Q1 và Q2 trên bộ nhớ device, sau đó chép địa chỉ của nó vào biến mảng dArrPointerEmbedding. Do đó, chúng ta không huỷ bộ nhớ của Q1 và Q2 sau khi gọi hàm createEmbeddingRoot.
	Embedding *Q1=nullptr;//embedding dQ.
	Embedding *Q2=nullptr;
	int sizedQ=0;


	//Tạo bảo nhiêu mảng dQ, mỗi mảng có số lượng phần tử là bao nhiêu và nội dung mảng là gì?
	//Tạo 2 mảng dQ
	/*1.Tạo mảng M có kích thước bằng với d_ValidExtension và khởi tạo giá trị cho các phần tử trong M bằng 0.*/
	int* d_M;
	cudaStatus=hipMalloc((int**)&d_M,noElem_d_ValidExtension*sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc M failed");
		//exit(1);
		goto Error;
	}
	else
	{
		hipMemset(d_M,0,noElem_d_ValidExtension*sizeof(int));
	}

	/*//2. Tạo noElem_d_ValidExtension threads. Mỗi thread sẽ kiểm tra phần tử tương ứng trong mảng d_ValidExtension xem có bằng cạnh (li,lij,lj) 
	Nếu bằng thì bậc vị trí tại M lên giá trị là 1*/
	//printf("\nMang d_ValidExtension");
	//printfExtension(d_ValidExtension,noElem_d_ValidExtension);
	//hipDeviceSynchronize();
	dim3 block(blocksize);
	dim3 grid((noElem_d_ValidExtension+block.x-1)/block.x);

	kernelMarkExtension<<<grid,block>>>(d_ValidExtension,noElem_d_ValidExtension,d_M,li,lij,lj);
	hipDeviceSynchronize();
	/*printf("\n\nMang d_ValidExtension");
	printfExtension(d_ValidExtension,noElem_d_ValidExtension);
	hipDeviceSynchronize();
	printf("\nMang d-M:");
	printInt(d_M,noElem_d_ValidExtension);*/

	/* 3. Exclusive Scan d_M
	Kết quả scan lưu vào mảng d_scanResult
	*/
	int* d_scanResult;
	cudaStatus=hipMalloc((int**)&d_scanResult,noElem_d_ValidExtension*sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc M failed");
		//exit(1);
		goto Error;
	}
	else
	{
		hipMemset(d_scanResult,0,noElem_d_ValidExtension*sizeof(int));
	}

	cudaStatus=scanV(d_M,noElem_d_ValidExtension,d_scanResult);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\nscanV() d_M createForwardEmbedding failed");
		//exit(1);
		goto Error;
	}

	/*
	4. Tạo mảng Q1 và Q2 có kích thước là (scanM[LastIndex]) nếu phần tử cuối cùng của d_ValidExtension không phải là (li,lij,lj).
	Ngược lại thì Q có kích thước là (scanM[LastIndex]+1). 
	Mỗi phần tử của Q có cấu trúc là {int idx, int vid}
	*/
	bool same = false;
	kernelMatchLastElement<<<1,1>>>(d_ValidExtension,noElem_d_ValidExtension,li,lij,lj,same);
	hipDeviceSynchronize();

	int noElem_d_Q=0;

	cudaStatus=getLastElement(d_scanResult,noElem_d_ValidExtension,noElem_d_Q);

	if (same==true){
		noElem_d_Q++;
	}

	sizedQ=noElem_d_Q;

	printf("\nnoElem_d_Q1:%d",noElem_d_Q);

	//Tạo Embedding dQ1, khi đã biết kích thước của chúng	

	cudaStatus = hipMalloc((void**)&Q1,sizedQ*sizeof(Embedding));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc dQ1 in createEmbeddingRoot() failed",cudaStatus);
		goto Error;
	}
	else
	{
		hipMemset(Q1,-1,sizedQ*sizeof(Embedding));
	}

	cudaStatus = hipMalloc((void**)&Q2,sizedQ*sizeof(Embedding));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc dQ1 in createEmbeddingRoot() failed",cudaStatus);
		goto Error;
	}
	else
	{
		hipMemset(Q2,-1,sizedQ*sizeof(Embedding));
	}
	kernelSetValueFordQ<<<grid,block>>>(d_ValidExtension,noElem_d_ValidExtension,Q1,Q2,d_scanResult,li,lij,lj);
	hipDeviceSynchronize();

	getPointer(dArrPointerEmbedding,noElem_dArrPointerEmbedding,Q1);
	getPointer(dArrPointerEmbedding,noElem_dArrPointerEmbedding,Q2);

	int iPrevQ=-1;
	for (int j = 0; j < 2; j++)
	{
		//Lấy kích thước của dQ lưu vào mảng d_arrSizedQ
		cudaStatus = getSizedQ(dArrSizedQ,noElem_dArrSizedQ,sizedQ);
		if(cudaStatus!=hipSuccess){
			fprintf(stderr,"\n getSizedQ  in makeColumnQ failed",stderr);
			goto Error;
		}

		//Lấy prevQ
		cudaStatus = getSizedQ(dArrPrevQ,noElem_dArrPrevQ,iPrevQ);
		if(cudaStatus!=hipSuccess){
			fprintf(stderr,"\n getSizedQ of kernel.cu failed",stderr);
			goto Error;
		}
		iPrevQ++;
	}


	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() in createEmbeddingRoot() failed",cudaStatus);
		goto Error;
	}
Error:
	return cudaStatus;

}

inline hipError_t createEmbeddingRoot1(Embedding **&dArrPointerEmbedding,int &noElem_dArrPointerEmbedding,int *&dArrSizedQ,int &noElem_dArrSizedQ,Extension *d_ValidExtension,int noElem_d_ValidExtension,int li,int lij,int lj){
	hipError_t cudaStatus;
	//Vì đây là lần đầu tiên tạo Embedding, chúng ta tạo 2 cột Q có kích thước bằng nhau và bằng số lượng Embedding tìm thấy trong d_ValidExtension của nhãn cạnh (li,lij,lj)
	//Tạo Q1 và Q2 trên bộ nhớ device, sau đó chép địa chỉ của nó vào biến mảng dArrPointerEmbedding. Do đó, chúng ta không huỷ bộ nhớ của Q1 và Q2 sau khi gọi hàm createEmbeddingRoot.
	Embedding *Q1=nullptr;//embedding dQ.
	Embedding *Q2=nullptr;
	int sizedQ=0;


	//Tạo bảo nhiêu mảng dQ, mỗi mảng có số lượng phần tử là bao nhiêu và nội dung mảng là gì?
	//Tạo 2 mảng dQ
	/*1.Tạo mảng M có kích thước bằng với d_ValidExtension và khởi tạo giá trị cho các phần tử trong M bằng 0.*/
	int* d_M;
	cudaStatus=hipMalloc((int**)&d_M,noElem_d_ValidExtension*sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc M failed");
		//exit(1);
		goto Error;
	}
	else
	{
		hipMemset(d_M,0,noElem_d_ValidExtension*sizeof(int));
	}

	/*//2. Tạo noElem_d_ValidExtension threads. Mỗi thread sẽ kiểm tra phần tử tương ứng trong mảng d_ValidExtension xem có bằng cạnh (li,lij,lj) 
	Nếu bằng thì bậc vị trí tại M lên giá trị là 1*/
	//printf("\nMang d_ValidExtension");
	//printfExtension(d_ValidExtension,noElem_d_ValidExtension);
	//hipDeviceSynchronize();
	dim3 block(blocksize);
	dim3 grid((noElem_d_ValidExtension+block.x-1)/block.x);

	kernelMarkExtension<<<grid,block>>>(d_ValidExtension,noElem_d_ValidExtension,d_M,li,lij,lj);
	hipDeviceSynchronize();
	/*printf("\n\nMang d_ValidExtension");
	printfExtension(d_ValidExtension,noElem_d_ValidExtension);
	hipDeviceSynchronize();
	printf("\nMang d-M:");
	printInt(d_M,noElem_d_ValidExtension);*/

	/* 3. Exclusive Scan d_M
	Kết quả scan lưu vào mảng d_scanResult
	*/
	int* d_scanResult;
	cudaStatus=hipMalloc((int**)&d_scanResult,noElem_d_ValidExtension*sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc M failed");
		//exit(1);
		goto Error;
	}
	else
	{
		hipMemset(d_scanResult,0,noElem_d_ValidExtension*sizeof(int));
	}

	cudaStatus=scanV(d_M,noElem_d_ValidExtension,d_scanResult);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\nscanV() d_M createForwardEmbedding failed");
		//exit(1);
		goto Error;
	}

	/*
	4. Tạo mảng Q1 và Q2 có kích thước là (scanM[LastIndex]) nếu phần tử cuối cùng của d_ValidExtension không phải là (li,lij,lj).
	Ngược lại thì Q có kích thước là (scanM[LastIndex]+1). 
	Mỗi phần tử của Q có cấu trúc là {int idx, int vid}
	*/
	bool same = false;
	kernelMatchLastElement<<<1,1>>>(d_ValidExtension,noElem_d_ValidExtension,li,lij,lj,same);
	hipDeviceSynchronize();

	int noElem_d_Q=0;

	cudaStatus=getLastElement(d_scanResult,noElem_d_ValidExtension,noElem_d_Q);

	if (same==true){
		noElem_d_Q++;
	}

	sizedQ=noElem_d_Q;

	printf("\nnoElem_d_Q1:%d",noElem_d_Q);

	//Tạo Embedding dQ1, khi đã biết kích thước của chúng	

	cudaStatus = hipMalloc((void**)&Q1,sizedQ*sizeof(Embedding));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc dQ1 in createEmbeddingRoot1() failed",cudaStatus);
		goto Error;
	}
	else
	{
		hipMemset(Q1,-1,sizedQ*sizeof(Embedding));
	}

	cudaStatus = hipMalloc((void**)&Q2,sizedQ*sizeof(Embedding));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc dQ1 in createEmbeddingRoot1() failed",cudaStatus);
		goto Error;
	}
	else
	{
		hipMemset(Q2,-1,sizedQ*sizeof(Embedding));
	}
	kernelSetValueFordQ<<<grid,block>>>(d_ValidExtension,noElem_d_ValidExtension,Q1,Q2,d_scanResult,li,lij,lj);
	hipDeviceSynchronize();

	getPointer(dArrPointerEmbedding,noElem_dArrPointerEmbedding,Q1);
	getPointer(dArrPointerEmbedding,noElem_dArrPointerEmbedding,Q2);


	for (int j = 0; j < 2; j++)
	{
		//Lấy kích thước của dQ lưu vào mảng d_arrSizedQ
		cudaStatus = getSizedQ(dArrSizedQ,noElem_dArrSizedQ,sizedQ);
		if(cudaStatus!=hipSuccess){
			fprintf(stderr,"\n getSizedQ  in makeColumnQ failed",stderr);
			goto Error;
		}	
	}


	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() in createEmbeddingRoot() failed",cudaStatus);
		goto Error;
	}
Error:
	return cudaStatus;
}


//Kernel khởi tạo giá trị cho right most path trên device */
__global__ void kernelInitializeValueForRMPath(int *dRMPath,int noElem_dRMPath){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElem_dRMPath){
		dRMPath[i]=i;		
	}

}

/* Tạo một right most path trên device */
inline hipError_t createRMPath(int *&dRMPath,int &noElem_dRMPath){
	hipError_t cudaStatus;
	//Khởi tạo kích thước ban đầu của dRMPath bằng 2
	noElem_dRMPath=2;
	cudaStatus = hipMalloc((void**)&dRMPath,noElem_dRMPath*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc dRMPath failed",cudaStatus);
		goto Error;
	}

	kernelInitializeValueForRMPath<<<1,2>>>(dRMPath,noElem_dRMPath);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n CudaDeviceSynchronize() in createRMPath() failed",cudaStatus);
		goto Error;
	}

Error:
	return cudaStatus;
}

__global__ void kernelPrintRMPath(int *dRMPath,int noElem_dRMPath){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElem_dRMPath){
		printf("\n dRMPath[%d]: %d",i,dRMPath[i]);
	}

}


//Hàm hiển thị nội dung dRMPath trên device
inline hipError_t printRMPath(int *dRMPath,int noElem_dRMPath){
	hipError_t cudaStatus;

	dim3 block(blocksize);
	dim3 grid((noElem_dRMPath + block.x - 1)/block.x);

	kernelPrintRMPath<<<grid,block>>>(dRMPath,noElem_dRMPath);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n CudaDeviceSynchronize() in createRMPath() failed",cudaStatus);
		goto Error;
	}

Error:
	return cudaStatus;
}

//kernel tìm số lượng embedding hiện tại
__global__ void kernelGetNumberOfEmbedding(int *dArrSizedQ,int noElem_dArrSizedQ,int *dNumberOfEmbedding){
	dNumberOfEmbedding[0] = dArrSizedQ[noElem_dArrSizedQ-1];
}

//Hàm tìm số lượng embedding hiện tại
inline hipError_t findNumberOfEmbedding(int *dArrSizedQ,int noElem_dArrSizedQ,int &noElem_dArrPointerdHO){
	hipError_t	cudaStatus;

	noElem_dArrPointerdHO=0;
	int *dNumberOfEmbedding;
	cudaStatus = hipMalloc((void**)&dNumberOfEmbedding,sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc dNumberOfEmbedding in findNumberOfEmbedding() failed",cudaStatus);
		goto Error;
	}
	else
	{
		hipMemset(dNumberOfEmbedding,0,sizeof(int));
	}

	kernelGetNumberOfEmbedding<<<1,1>>>(dArrSizedQ,noElem_dArrSizedQ,dNumberOfEmbedding);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() in createGraphHistory() failed",cudaStatus);
		goto Error;
	}

	hipMemcpy(&noElem_dArrPointerdHO,dNumberOfEmbedding,sizeof(int),hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() in createGraphHistory() failed",cudaStatus);
		goto Error;
	}
Error:
	hipFree(dNumberOfEmbedding);
	return cudaStatus;
}

inline hipError_t createElementdHO(int *&dHO,int maxOfVer){
	hipError_t	cudaStatus;

	cudaStatus = hipMalloc((void**)&dHO,maxOfVer*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc() for dHO in createElementdHO() failed",cudaStatus);
		goto Error;
	}
	else
	{
		hipMemset(dHO,0,maxOfVer*sizeof(int));
	}

	//hipDeviceSynchronize();
	//cudaStatus=hipGetLastError();
	//if(cudaStatus!=hipSuccess){
	//	fprintf(stderr,"\n hipDeviceSynchronize() in createElementdHO() failed",cudaStatus);
	//	goto Error;
	//}
Error:
	return cudaStatus;
}

//kernel lấy pointer trỏ đến bộ nhớ mảng ở device rồi gán cho dArrPointerdHO
__global__ void	kernelAssignPointer(int **dArrPointerdHO,int pos,int *dHO){
	dArrPointerdHO[pos]=dHO;
}

//Hàm lấy pointer trỏ đến bộ nhớ mảng ở device rồi gán cho dArrPointerdHO
inline hipError_t assignPointer(int **&dArrPointerdHO,int pos,int *dHO){
	hipError_t cudaStatus;

	kernelAssignPointer<<<1,1>>>(dArrPointerdHO,pos,dHO);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() in assignPointer() failed",cudaStatus);
		goto Error;
	}
Error:
	return cudaStatus;
}

//Kernel in mảng double pointer Int trên device
__global__ void kernelPrintDoublePointerInt(int **dArrPointerdHO,int noElem_dArrPointerdHO,unsigned int maxOfVer){
	int i = blockDim.x *blockIdx.x + threadIdx.x;
	if(i<noElem_dArrPointerdHO){
		for (int j = 0; j < maxOfVer; j++)
		{
			printf("\n Thread %d: j:%d V[%d]:%d",i,j,j,dArrPointerdHO[i][j]);;
		}
	}

}


/* Hàm in mảng double pointer int (dArrPointerdHO) khi biết số lượng phần tử mảng (noElem_dArrPointerdHO) và
* Kích thước của mỗi phần tử mảng */
inline hipError_t printDoublePointerInt(int **dArrPointerdHO,int noElem_dArrPointerdHO,unsigned int maxOfVer){
	hipError_t cudaStatus;

	dim3 block(blocksize);
	dim3 grid((noElem_dArrPointerdHO + block.x - 1)/block.x);
	kernelPrintDoublePointerInt<<<grid,block>>>(dArrPointerdHO,noElem_dArrPointerdHO,maxOfVer);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() in printDoublePointerInt() failed",cudaStatus);
		goto Error;
	}
Error:
	return cudaStatus;
}

//Hàm tạo mảng double pointer Int trên device (dArrPointerdHO) khi biết trước số lượng phần tử cần tạo và kích thước của mỗi mảng.
inline hipError_t createdArrPointerdHO(int **&dArrPointerdHO,int noElem_dArrPointerdHO,unsigned int maxOfVer){
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&dArrPointerdHO,noElem_dArrPointerdHO*sizeof(int*));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc() for dArrPointerdHO in createdArrPointerdHO() failed",cudaStatus);
		goto Error;
	}
	for (int i = 0; i < noElem_dArrPointerdHO; i++)
	{
		int noElem_dHO=maxOfVer;
		int *dHO=nullptr;
		cudaStatus = createElementdHO(dHO,maxOfVer);
		if(cudaStatus!=hipSuccess){
			fprintf(stderr,"\n createElementdHO() in createdArrPointerdHO() failed",cudaStatus);
			goto Error;
		}

		int pos = i;
		assignPointer(dArrPointerdHO,pos,dHO);
	}

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() in createdArrPointerdHO() failed",cudaStatus);
		goto Error;
	}
Error:
	return cudaStatus;

}

//Hàm tạo phần tử dHLN trên device
inline hipError_t createElementdHLN(int *&dHLN,int noElem_dHLN){
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&dHLN,noElem_dHLN*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc() dHLN in createElementdHLN() failed",cudaStatus);
		goto Error;
	}
	else
	{
		hipMemset(dHLN,0,noElem_dHLN*sizeof(int));
	}

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() in createElementdHLN() failed",cudaStatus);
		goto Error;
	}
Error:
	return cudaStatus;
}

//Hàm tạo mảng double pointer int dHLN
inline hipError_t createdArrPointerdHLN(int **&dArrPointerdHLN,int noElem_dArrPointerdHO,int *hNumberEdgeInEachGraph,int *hArrGraphId){
	hipError_t cudaStatus;
	//Cấp phát bộ nhớ trên device cho dArrpointerdHLN theo số lượng embedding, cũng chính bằng số lượng phần tử của mảng dArrPointerdHO (noElem_dArrPointerdHO)
	cudaStatus = hipMalloc((void**)&dArrPointerdHLN, noElem_dArrPointerdHO*sizeof(int*));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc() dArrPointerdHLN in createdArrPointerdHLN() failed",cudaStatus);
		goto Error;
	}	


	for (int i = 0; i < noElem_dArrPointerdHO; i++)
	{
		int index = hArrGraphId[i];
		int *dHLN=nullptr;
		cudaStatus = createElementdHLN(dHLN,hNumberEdgeInEachGraph[index]);
		if(cudaStatus!=hipSuccess){
			fprintf(stderr,"\n createElementdHLN() in createdArrPointerdHLN() failed",cudaStatus);
			goto Error;
		}		
		cudaStatus = assignPointer(dArrPointerdHLN,i,dHLN);
		if(cudaStatus!=hipSuccess){
			fprintf(stderr,"\n assignPointer() in createdArrPointerdHLN() failed",cudaStatus);
			goto Error;
		}
	}

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() in createdArrPointerdHLN() failed",cudaStatus);
		goto Error;
	}
Error:
	return cudaStatus;
}

//kernel tìm graphid của tất cả các embedding và lưu kết quả vào mảng
__global__ void kernelFindGraphIdOfAllEmbedding(Embedding **dArrPointerEmbedding,int noElem_dArrPointerEmbedding,int *dArrGraphId,unsigned int maxOfVer,int noElemOfEmbedding,int *dArrSizedQ){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemOfEmbedding){
		int vid =dArrPointerEmbedding[noElem_dArrPointerEmbedding-1][i].vid;
		int graphId=vid/maxOfVer;
		dArrGraphId[i]=graphId;
		//printf("\nThread %d: vid:%d graphId:%d maxOfVer:%d",i,vid,graphId,maxOfVer);
	}

}

//Hàm tìm graphid của tất cả các embedding và lưu kết quả vào mảng
inline hipError_t findGraphIdOfAllEmbedding(Embedding **dArrPointerEmbedding,int noElem_dArrPointerEmbedding,int *&hArrGraphId,unsigned int maxOfVer,int *&dArrGraphId,int noElemOfEmbedding,int *dArrSizedQ){
	hipError_t cudaStatus;

	hArrGraphId = (int*)malloc(noElemOfEmbedding*sizeof(int));
	if(hArrGraphId==NULL){
		printf("\nMalloc hArrGraphId in findGraphIdOfAllEmbedding() failed\n");
		exit(1);
	}


	//int *dArrGraphId=nullptr;
	cudaStatus = hipMalloc((void**)&dArrGraphId,noElemOfEmbedding*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc() dArrGraphId in findGraphIdOfAllEmbedding() failed",cudaStatus);
		goto Error;
	}

	dim3 block(blocksize);
	dim3 grid((noElemOfEmbedding + block.x -1)/block.x);

	kernelFindGraphIdOfAllEmbedding<<<grid,block>>>(dArrPointerEmbedding,noElem_dArrPointerEmbedding,dArrGraphId,maxOfVer,noElemOfEmbedding,dArrSizedQ);
	hipDeviceSynchronize();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() kernelFindGraphIdOfAllEmbedding in findGraphIdOfAllEmbedding() failed",cudaStatus);
		goto Error;
	}

	hipMemcpy(hArrGraphId,dArrGraphId,noElemOfEmbedding*sizeof(int),hipMemcpyDeviceToHost);

	/*printf("\n**********hArrGraphId ***********\n");
	for (int j = 0; j < noElemOfEmbedding; j++)
	{
	printf("\n hArrGraphId[%d]:%d",j,hArrGraphId[j]);
	}*/

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() kernelFindGraphIdOfAllEmbedding in findGraphIdOfAllEmbedding() failed",cudaStatus);
		goto Error;
	}
Error:
	return cudaStatus;
}

//kernel in mảng double pointer int dArrPointerdHLN
__global__ void kernelPrintdArrPointerdHLN(int **dArrPointerdHLN,int noElem_dArrPointerdHO,int *dNumberEdgeInEachGraph,int *dArrGraphId){
	int i = blockDim.x *blockIdx.x + threadIdx.x;
	if(i<noElem_dArrPointerdHO){
		int n =dNumberEdgeInEachGraph[dArrGraphId[i]];
		for (int j = 0; j < n; j++)
		{
			printf("\n Thread %d: j:%d dArrPointerdHLN[%d][%d]:%d",i,j,i,j,dArrPointerdHLN[i][j]);
		}
	}

}

//Hàm in mảng double pointer int dArrPointerdHLN
inline hipError_t printdArrPointerdHLN(int **dArrPointerdHLN,int noElem_dArrPointerdHO,int *dNumberEdgeInEachGraph,int *dArrGraphId){
	hipError_t cudaStatus;
	dim3 block(blocksize);
	dim3 grid((noElem_dArrPointerdHO+block.x - 1)/block.x);
	kernelPrintdArrPointerdHLN<<<grid,block>>>(dArrPointerdHLN,noElem_dArrPointerdHO,dNumberEdgeInEachGraph,dArrGraphId);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() kernelFindGraphIdOfAllEmbedding in findGraphIdOfAllEmbedding() failed",cudaStatus);
		goto Error;
	}
Error:
	return cudaStatus;
}

//kernel tạo mảng dArrNumberEdgeOfEachdHLN dựa vào graphId đã thu thập được theo thứ tự của từng embedding lưu trong mảng dArrGraphId
__global__ void kernelCreatedArrNumberEdgeOfEachdHLN(int *dArrNumberEdgeOfEachdHLN,int noElemOfEmbedding,int *dArrGraphId,int *dNumberEdgeInEachGraph){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemOfEmbedding){
		dArrNumberEdgeOfEachdHLN[i]= dNumberEdgeInEachGraph[dArrGraphId[i]];
	}

}


//Hàm tạo mảng dArrNumberEdgeOfEachdHLN dựa vào graphId đã thu thập được theo thứ tự của từng embedding lưu trong mảng dArrGraphId
inline hipError_t createdArrNumberEdgeOfEachdHLN(int *&dArrNumberEdgeOfEachdHLN,int noElemOfEmbedding,int *dArrGraphId,int *dNumberEdgeInEachGraph){
	hipError_t cudaStatus;

	//Cấp phát bộ nhớ cho mảng dArrNumberEdgeOfEachdHLN với số lượng phần tử bằng với số lượng embedding
	cudaStatus = hipMalloc((void**)&dArrNumberEdgeOfEachdHLN, noElemOfEmbedding*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc() dArrNumberEdgeOfEachdHLN in createdArrPointerdHLN() failed",cudaStatus);
		goto Error;
	}

	dim3 block(blocksize);
	dim3 grid((noElemOfEmbedding+block.x - 1)/block.x);
	kernelCreatedArrNumberEdgeOfEachdHLN<<<grid,block>>>(dArrNumberEdgeOfEachdHLN,noElemOfEmbedding,dArrGraphId,dNumberEdgeInEachGraph);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() kernelCreatedArrNumberEdgeOfEachdHLN in createdArrNumberEdgeOfEachdHLN() failed",cudaStatus);
		goto Error;
	}
Error:
	return cudaStatus;
}

//kernel in nội dung của mảng dArrPointerdHLN khi biết số lượng cạnh của mỗi phần tử tương ứng của embedding được lưu trong mảng dArrNumberEdgeOfEachdHLN
__global__ void kernelprintDoublePointerInt(int **dArrPointerdHLN,int noElemOfEmbedding,int *dArrNumberEdgeOfEachdHLN){
	int i = blockDim.x *blockIdx.x + threadIdx.x;
	if(i<noElemOfEmbedding){
		int length = dArrNumberEdgeOfEachdHLN[i];
		for (int j = 0; j < length; j++)
		{
			printf("\n Thread %d: j:%d dArrPointerdHLN[%d][%d]:%d",i,j,i,j,dArrPointerdHLN[i][j]);
		}
	}

}


//Overloading function printDoublePointerInt() để in nội dung mảng dArrPointerdHLN dựa vào số lượng embedding và số lượng cạnh trong mỗi phần tử
inline hipError_t printDoublePointerInt(int **dArrPointerdHLN,int noElemOfEmbedding,int *dArrNumberEdgeOfEachdHLN){
	hipError_t cudaStatus;

	dim3 block(blocksize);
	dim3 grid((noElemOfEmbedding + block.x -1)/block.x);

	kernelprintDoublePointerInt<<<grid,block>>>(dArrPointerdHLN,noElemOfEmbedding,dArrNumberEdgeOfEachdHLN);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize()  in printDoublePointerInt() failed",cudaStatus);
		goto Error;
	}
Error:
	return cudaStatus;
}

__global__ void kernelAssignValueForGraphHistory(Embedding **dArrPointerEmbedding,int noElem_dArrPointerEmbedding,int *dArrPrevQ,int noElemOfEmbedding,int *d_O,int *d_N,unsigned int maxOfVer,int **dArrPointerdHO,int **dArrPointerdHLN,int *dArrNumberEdgeOfEachdHLN){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	//Mỗi một embedding sẽ cập nhật graphHistory tương ứng của nó (gồm 2 mảng: dArrPointerdHO(mảng các đỉnh của embedding mà thread i đang xử lý) và dArrPointerdHLN(mảng các cạnh tương ứng với ánh xạ đỉnh).)
	if(i<noElemOfEmbedding){
		int vid = dArrPointerEmbedding[noElem_dArrPointerEmbedding-1][i].vid; //Từ cột Q cuối cùng, chúng ta lấy ra được vid của 6 embedding tương ứng
		int indexOfFirstVertexInGraph = vid-(vid%maxOfVer); //the first global id vertex in graph
		int toVid = vid;//đỉnh to của cạnh thuộc embedding
		int idxOfVertex= (vid%maxOfVer); //Vị trí của phần tử đỉnh cần cập nhật trong mảng dArrPointerdHO[i][idxOfVertex];
		dArrPointerdHO[i][idxOfVertex]=2; //Cập nhật đỉnh đã thuộc right most path của embedding trong mảng dArrPointerdHO tương ứng.
		int prevQ= dArrPrevQ[noElem_dArrPointerEmbedding-1]; 
		int newi=dArrPointerEmbedding[noElem_dArrPointerEmbedding-1][i].idx; //lấy index gán cho newi

		while (true)
		{			

			vid = dArrPointerEmbedding[prevQ][newi].vid; //truy xuất phần tử phía trước theo prevQ và newi
			int fromVid=vid; //đỉnh from của cạnh thuộc embedding


			int idxEdge = d_O[vid]-d_O[indexOfFirstVertexInGraph]; //vị trí cạnh cần cập nhật được khởi tạo bằng giá trị index của vid đang xét trừ đi giá trị index của đỉnh đầu tiên trong đồ thị đó.
			int indexOfdN=d_O[fromVid];

			while (d_N[indexOfdN]!=toVid){
				idxEdge=idxEdge+1;
				indexOfdN++;
			}

			int fromVidR=toVid;
			int toVidR=fromVid;
			int indexOfEdgeR=d_O[fromVidR]-d_O[indexOfFirstVertexInGraph];
			indexOfdN=d_O[fromVidR];
			while(d_N[indexOfdN]!=toVidR){
				indexOfEdgeR++;
				indexOfdN++;
			}


			//Nếu không phải là đỉnh đầu tiên thì phải cộng vào idxEdge một lượng bằng tổng bậc của các đỉnh trước đó
			//Tổng bậc của các đỉnh trước đó chính bằng 

			idxOfVertex = (vid%maxOfVer); //Đánh dấu đỉnh thuộc Embedding
			dArrPointerdHO[i][idxOfVertex]=2;


			dArrPointerdHLN[i][idxEdge]=2;//Đánh dấu cạnh thuộc Embedding. vì đây là đơn đồ thị vô hướng nên cạnh AB cũng bằng cạnh BA,do đó ta phải đánh dấu cạnh BA cũng thuộc embedding.
			dArrPointerdHLN[i][indexOfEdgeR]=2;


			if(dArrPrevQ[prevQ]==-1) return; //nếu là cột Q đầu tiên thì dừng lại vì đã duyệt xong embedding
			newi=dArrPointerEmbedding[prevQ][newi].idx; //ngược lại thì lấy index của cột Q phía trước
			prevQ=dArrPrevQ[prevQ]; //Lấy Q phía trước
			toVid=fromVid; //cập nhật lại đỉnh to.
		}


	}
}

//Xây dựng graphHistory cho tất cả các embedding
inline hipError_t createGraphHistory(Embedding **dArrPointerEmbedding,int *dArrSizedQ,int *dArrPrevQ,int noElem_dArrPointerEmbedding,int noElem_dArrSizedQ,int noElem_dArrPrevQ,int *d_O,int *d_LO,int numberOfElementd_O,int *d_N,int *d_LN,int numberOfElementd_N,unsigned int maxOfVer,int **&dArrPointerdHO,int &noElem_dArrPointerdHO,int **&dArrPointerdHLN,int *&dArrNumberEdgeOfEachdHLN,int *hNumberEdgeInEachGraph,int noElem_hNumberEdgeInEachGraph,int *dNumberEdgeInEachGraph){
	hipError_t cudaStatus;

	//số lượng embedding chính bằng giá trị của biến noElem_dArrPointerdHO
	cudaStatus = findNumberOfEmbedding(dArrSizedQ,noElem_dArrSizedQ,noElem_dArrPointerdHO);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n findNumberOfEmbedding() in createGraphHistory() failed",cudaStatus);
		goto Error;
	}
	int noElemOfEmbedding=noElem_dArrPointerdHO;
	//In nội dung số lượng phần tử embedding vừa tìm được
	//printf("\nNumber Of Embedding: %d",noElem_dArrPointerdHO);

	/* Tạo graphHistory
	*	1. Tạo mảng dArrPointerdHO
	*	2. Tạo mảng dArrPointerdHLN
	*	3. Tạo mảng dArrNumberEdgeOfEachdHLN: mảng này mô tả số cạnh của mỗi phần tử trong mảng dArrPointerdHLN
	*	Bước 2 và 3 có thể được thực hiện một cách độc lập, nên có thể xử lý song song ở bước này.
	*	4. Cập nhật nội dung cho 3 mảng trên.
	*/

	//1. Tạo 5 mảng có số lượng phần tử là  maxOfVer trên device, và chép pointer của các mảng bỏ vào phần tử dArrPointerEmbedding
	cudaStatus = createdArrPointerdHO(dArrPointerdHO,noElem_dArrPointerdHO,maxOfVer);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n createdArrPointerdHO() in createGraphHistory() failed",cudaStatus);
		goto Error;
	}
	//In nội dung của mảng vừa tạo được
	/*printf("\n ********** dArrPointerdHO *****************\n");
	printDoublePointerInt(dArrPointerdHO,noElem_dArrPointerdHO,maxOfVer);
	if(cudaStatus!=hipSuccess){
	fprintf(stderr,"\n printDoublePointerInt() in createGraphHistory() failed",cudaStatus);
	goto Error;
	}*/

	//2. Tạo dArrPointerHLN
	/* Tìm số lượng cạnh của mỗi embedding
	* Biết được global vertex id của embedding thì chúng ta biết được graphId của embedding đó
	* Biết được graphID thì suy ra được số lượng cạnh của embedding.
	* Trước tiên nên tính số lượng cạnh của mỗi đồ thị trong CSDL và lưu chúng vào một mảng <-- Làm được
	* Sau đó duyệt qua các vid của embedding ở last column Q để biết được graphID mà embedding thuộc vào
	*/
	//Tính graphId của từng embedding và lưu vào mảng 
	int *dArrGraphId=nullptr; //Mảng này dùng để in nội dung của mảng dArrPointerdHLN
	int *hArrGraphId=nullptr; //Lấy graphId ở mảng này mang đi tra trong mảng hNumberEdgeInEachGraph để lấy số lượng cạnh cho embedding đó để tạo dArrPointerdHLN
	cudaStatus = findGraphIdOfAllEmbedding(dArrPointerEmbedding,noElem_dArrPointerEmbedding,hArrGraphId,maxOfVer,dArrGraphId,noElemOfEmbedding,dArrSizedQ);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n findGraphIdOfAllEmbedding() in createGraphHistory() failed",cudaStatus);
		goto Error;
	}
	cudaStatus =createdArrPointerdHLN(dArrPointerdHLN,noElem_dArrPointerdHO,hNumberEdgeInEachGraph,hArrGraphId);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n createdArrPointerdHLN() in createGraphHistory() failed",cudaStatus);
		goto Error;
	}

	//In nội dung mảng dArrPointerdHLN
	/*printf("\n***************** dArrPointerdHLN ***************\n");

	cudaStatus = printdArrPointerdHLN(dArrPointerdHLN,noElem_dArrPointerdHO,dNumberEdgeInEachGraph,dArrGraphId);
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
	fprintf(stderr,"\n printdArrPointerdHLN() in createGraphHistory() failed",cudaStatus);
	goto Error;
	}*/

	//3. Tạo mảng dArrNumberEdgeOfEachdHLN

	cudaStatus = createdArrNumberEdgeOfEachdHLN(dArrNumberEdgeOfEachdHLN,noElemOfEmbedding,dArrGraphId,dNumberEdgeInEachGraph);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n createdArrNumberEdgeOfEachdHLN() in createGraphHistory() failed",cudaStatus);
		goto Error;
	}

	//printf("\n**************dArrNumberEdgeOfEachdHLN**************\n");
	//printInt(dArrNumberEdgeOfEachdHLN,noElemOfEmbedding);

	//printf("\n**************dArrNumberEdgeOfEachdHLN**************\n");
	//printDoublePointerInt(dArrPointerdHLN,noElemOfEmbedding,dArrNumberEdgeOfEachdHLN);

	//4.1 Cập nhật nội dung cho graphHistory
	/* Cần có cơ sở dữ liệu để ánh xạ đỉnh và cạnh phù hợp vào mảng dArrPointerEmbedding (chứa idx và vid), dArrPointerdHLN (chứa cạnh)
	*	Mỗi một thread sẽ chịu trách nhiệm cập nhật dữ liệu cho 1 embedding
	*/


	kernelAssignValueForGraphHistory<<<1,noElemOfEmbedding>>>(dArrPointerEmbedding,noElem_dArrPointerEmbedding,dArrPrevQ,noElemOfEmbedding,d_O,d_N,maxOfVer,dArrPointerdHO,dArrPointerdHLN,dArrNumberEdgeOfEachdHLN);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() in createGraphHistory() failed",cudaStatus);
		goto Error;
	}
Error:
	return cudaStatus;
}

//kernel tính số cạnh trong mỗi đồ thị trong CSDL và lưu vào biến mảng tương ứng.
__global__ void kernelGetNumberOfEdgeInGraph(int *d_O,int numberOfElementd_N,unsigned int numberOfGraph,unsigned int maxOfVer,int *dNumberEdgeInEachGraph){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<numberOfGraph){
		int graphId=i;
		int idxFrom = graphId*maxOfVer;		
		int idxFirstNext = (graphId+1)*maxOfVer;
		int r=0;
		if (graphId!=(numberOfGraph-1)){
			r=d_O[idxFirstNext]-d_O[idxFrom];
		}else
		{
			r=numberOfElementd_N-d_O[idxFrom];
		}
		dNumberEdgeInEachGraph[i]=r;	
	}
}

//Hàm tính số cạnh của tất cả các đồ thị trong CSDL, kết quả lưu vào một mảng tương ứng
inline hipError_t getNumberOfEdgeInGraph(int *d_O,int numberOfElementd_N,unsigned int maxOfVer,int *&hNumberEdgeInEachGraph,int *&dNumberEdgeInEachGraph,unsigned int numberOfGraph){
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&dNumberEdgeInEachGraph,numberOfGraph*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc() in dNumberEdgeInEachGraph createGraphHistory() failed",cudaStatus);
		goto Error;
	}

	dim3 block(blocksize);
	dim3 grid((numberOfGraph + block.x-1)/block.x);

	kernelGetNumberOfEdgeInGraph<<<grid,block>>>(d_O,numberOfElementd_N,numberOfGraph,maxOfVer,dNumberEdgeInEachGraph);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize()  kernelGetNumberOfEdgeInGraph in getNumberOfEdgeInGraph() failed",cudaStatus);
		goto Error;
	}

	//printf("\n *************dNumberEdgeInEachGraph********\n" );
	//printInt(dNumberEdgeInEachGraph,numberOfGraph);

	hNumberEdgeInEachGraph = (int*)malloc(numberOfGraph*sizeof(int));
	if(hNumberEdgeInEachGraph==NULL){
		printf("\n Malloc hNumberEdgeInEachGraph in getNumberOfEdgeInGraph() failed" );
		exit(1);
	}

	hipMemcpy(hNumberEdgeInEachGraph,dNumberEdgeInEachGraph,numberOfGraph*sizeof(int),hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() in createGraphHistory() failed",cudaStatus);
		goto Error;
	}
Error:
	return cudaStatus;
}

//kernel in tất cả column Q của embedding
__global__ void kernelprintAllEmbeddingColumn(Embedding **dArrPointerEmbedding,int *dArrSizedQ,int noElem_dArrPointerEmbedding){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElem_dArrPointerEmbedding){
		Embedding* Q = dArrPointerEmbedding[i];
		int lenght = dArrSizedQ[i];
		for (int j = 0; j < lenght; j++)
		{
			printf("\n Thread %d: j:%d (idx:%d vid:%d)",i,j,Q[j].idx,Q[j].vid);
		}
	}

}


//Hàm in tất cả các column Q của embedding
inline hipError_t printAllEmbeddingColumn(Embedding **dArrPointerEmbedding,int *dArrSizedQ,int noElem_dArrPointerEmbedding){
	hipError_t cudaStatus;

	dim3 block(blocksize);
	dim3 grid((noElem_dArrPointerEmbedding + block.x - 1)/block.x);
	printf("\n****************** All Columm in Embedding dArrPointerEmbedding *************\n");
	kernelprintAllEmbeddingColumn<<<grid,block>>>(dArrPointerEmbedding,dArrSizedQ,noElem_dArrPointerEmbedding);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() in printAllEmbeddingColumn() failed",cudaStatus);
		goto Error;
	}
Error:
	return cudaStatus;
}

//kernel in một embedding khi biết vị trí Row của nó trong last column Q.
__global__ void kernelprintEmbeddingFromPos(Embedding **dArrPointerEmbedding,int posColumn,int posRow){
	Embedding *Q =dArrPointerEmbedding[posColumn];
	printf("\n Q[%d] Row[%d] (idx:%d vid:%d)",posColumn,posRow,Q[posRow].idx,Q[posRow].vid);
	while (true)
	{
		posRow = Q[posRow].idx;
		posColumn=posColumn-1;		
		Q=dArrPointerEmbedding[posColumn];
		printf("\n Q[%d] Row[%d] (idx:%d vid:%d)",posColumn,posRow,Q[posRow].idx,Q[posRow].vid);
		posRow=Q[posRow].idx;
		if(posRow==-1) return;
	}
}


//Hàm in một embedding khi biết vị trí Row của nó trong last column Q.
inline hipError_t printEmbeddingFromPos(Embedding **dArrPointerEmbedding,int posColumn,int posRow){
	hipError_t cudaStatus;
	printf("\n ****Embeding from posColumn: %d posRow:%d **************\n",posColumn,posRow);
	kernelprintEmbeddingFromPos<<<1,1>>>(dArrPointerEmbedding,posColumn,posRow);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() in printEmbeddingFromPos() failed",cudaStatus);
		goto Error;
	}
Error:
	return cudaStatus;
}

//kernel tìm bậc của các vid trên cột Q và lưu kết quả vào mảng dArrDegreeOfVid
__global__ void kernelCalDegreeOfVid(Embedding **dArrPointerEmbedding,int idxQ,int *d_O, int numberOfElementd_O,int noElem_Embedding,int numberOfElementd_N,unsigned int maxOfVer,float *dArrDegreeOfVid){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElem_Embedding){
		int vid = dArrPointerEmbedding[idxQ][i].vid;
		float degreeOfV =0;
		int nextVid;
		int graphid;
		int lastGraphId=(numberOfElementd_O-1)/maxOfVer;
		if (vid==numberOfElementd_O-1){ //nếu như đây là đỉnh cuối cùng trong d_O
			degreeOfV=numberOfElementd_N-d_O[vid]; //thì bậc của đỉnh vid chính bằng tổng số cạnh trừ cho giá trị của d_O[vid].
		}
		else
		{
			nextVid = vid+1; //xét đỉnh phía sau có khác 1 hay không?
			graphid=vid/maxOfVer;
			if(d_O[nextVid]==-1 && graphid==lastGraphId){
				degreeOfV=numberOfElementd_N-d_O[vid];
			}
			else if(d_O[nextVid]==-1 && graphid!=lastGraphId){
				nextVid=(graphid+1)*maxOfVer;
				degreeOfV=d_O[nextVid]-d_O[vid];
			}
			else
			{
				degreeOfV=d_O[nextVid]-d_O[vid];
			}							
		}
		dArrDegreeOfVid[i]=degreeOfV;
	}

}

//Hàm tìm bậc của các đỉnh trên column Q và lưu kết quả vào mảng dArrDegreeOfVid
inline hipError_t findDegreeOfVer(Embedding **dArrPointerEmbedding,int idxQ,int *d_O, int numberOfElementd_O,int noElem_Embedding,int numberOfElementd_N, unsigned int maxOfVer,float *&dArrDegreeOfVid){
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&dArrDegreeOfVid,noElem_Embedding*sizeof(float));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc dArrDegreeOfVid in findMaxDegreeOfVer() failed");
		goto Error;
	}
	else
	{
		hipMemset(dArrDegreeOfVid,0,noElem_Embedding*sizeof(float));
	}

	dim3 block(blocksize);
	dim3 grid((noElem_Embedding + block.x -1)/block.x);
	kernelCalDegreeOfVid<<<grid,block>>>(dArrPointerEmbedding,idxQ,d_O, numberOfElementd_O,noElem_Embedding,numberOfElementd_N, maxOfVer,dArrDegreeOfVid);	
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();	
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() of kernelCalDegreeOfVid in findDegreeOfVer() failed",cudaStatus);
		goto Error;
	}

Error:

	return cudaStatus;
}


//Hàm tìm bậc lớn nhất của các đỉnh vid trong cột Q và lưu kết quả vào biến maxDegreeOfVer và float *dArrDegreeOfVid
inline hipError_t findMaxDegreeOfVer(Embedding **dArrPointerEmbedding,int idxQ,int *d_O, int numberOfElementd_O,int noElem_Embedding,int numberOfElementd_N,unsigned int maxOfVer,int &maxDegreeOfVer,float *&dArrDegreeOfVid){
	hipError_t cudaStatus;

	//Lấy bậc của các đỉnh vid trong cột Q và lưu vào mảng dArrDegreeOfVid có số lượng phần tử bằng số lượng phần tử của embedding

	cudaStatus = findDegreeOfVer(dArrPointerEmbedding,idxQ,d_O, numberOfElementd_O,noElem_Embedding,numberOfElementd_N, maxOfVer,dArrDegreeOfVid);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\findDegreeOfVer() in findMaxDegreeOfVer() failed");
		goto Error;
	}

	printf("\n*******dArrDegreeOfVid*************\n");
	printFloat(dArrDegreeOfVid,noElem_Embedding);

	//Tìm bậc lớn nhất và lưu kết quả vào biến maxDegreeOfVer
	float *h_max;
	h_max = (float*)malloc(sizeof(float));
	if(h_max==NULL){
		printf("\nMalloc h_max failed");
		exit(1);
	}

	float *d_max;
	int *d_mutex;
	cudaStatus=hipMalloc((void**)&d_max,sizeof(float));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc d_max failed");
		goto Error;
	}
	else
	{
		hipMemset(d_max,0,sizeof(float));
	}

	cudaStatus=hipMalloc((void**)&d_mutex,sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc d_mutex failed");
		goto Error;
	}
	else
	{
		hipMemset(d_mutex,0,sizeof(int));
	}

	dim3 gridSize = 256;
	dim3 blockSize = 256;
	find_maximum_kernel<<<gridSize, blockSize>>>(dArrDegreeOfVid, d_max, d_mutex, noElem_Embedding);

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaDeviceSynchronize find_maximum_kernel in findMaxDegreeOfVer() failed");
		goto Error;
	}

	// copy from device to host
	hipMemcpy(h_max, d_max, sizeof(float), hipMemcpyDeviceToHost);

	//report results
	maxDegreeOfVer = (int)(*h_max); //bậc lớn nhất của các đỉnh trong 1 cột Q
	printf("\nMax degree of vid in Q column is: %d",maxDegreeOfVer);




Error:
	free(h_max);
	hipFree(d_max);
	//hipFree(dArrDegreeOfVid); Giữ lại bậc của các đỉnh trong cột Q để thuận lợi cho việc tìm các mở rộng ở bước kế tiếp
	return cudaStatus;
}

//kernel tìm các mở rộng hợp lệ và ghi nhận vào mảng dArrV và dArrExtension tương ứng.
__global__ void kernelFindValidForwardExtension(Embedding **dArrPointerEmbedding,int noElem_dArrPointerEmbedding,int noElem_Embedding,int *d_O,int *d_LO,int *d_N,int *d_LN,float *dArrDegreeOfVid,int maxDegreeOfVer,struct_V *dArrV,EXT *dArrExtension,int idxQ,int minLabel,int maxid){
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if(i<noElem_Embedding){
		int posColumn =noElem_dArrPointerEmbedding-1;
		int posRow=i;
		int col = posColumn;
		int row = posRow;
		Embedding *Q=dArrPointerEmbedding[idxQ];
		int vid = Q[i].vid;
		int degreeVid=__float2int_rn(dArrDegreeOfVid[i]);
		//Duyệt qua các đỉnh kề với đỉnh vid dựa vào số lần duyệt là bậc
		int indexToVidIndN=d_O[vid];
		int labelFromVid = d_LO[vid];
		int toVid;
		int labelToVid;
		bool b=true;
		for (int j = 0; j < degreeVid; j++,indexToVidIndN++) //Duyệt qua tất cả các đỉnh kề với đỉnh vid, nếu đỉnh không thuộc embedding thì --> cạnh cũng không thuộc embedding vì đây là Q cuối
		{			
			toVid=d_N[indexToVidIndN]; //Lấy vid của đỉnh cần kiểm tra
			labelToVid = d_LO[toVid]; //lấy label của đỉnh cần kiểm tra
			posColumn=col;
			posRow=row;
			Q=dArrPointerEmbedding[posColumn];
			printf("\nThread %d, j: %d has ToVidLabel:%d",i,j,labelToVid);
			//1. Trước tiên kiểm tra nhãn của labelToVid có nhỏ hơn minLabel hay không. Nếu nhỏ hơn thì return
			if(labelToVid<minLabel) continue;
			//2. kiểm tra xem đỉnh toVid có tồn tại trong embedding hay không nếu tồn tại thì return
			//Duyệt qua embedding column từ Q cuối đến Q đầu, lần lượt lấy vid so sánh với toVid

			//printf("\n Q[%d] Row[%d] (idx:%d vid:%d)",posColumn,posRow,Q[posRow].idx,Q[posRow].vid);//Q[1][0]
			if(toVid==Q[posRow].vid) continue;
			//printf("\nj:%d toVid:%d Q.vid:%d",j,toVid,Q[posRow].vid);

			while (true)
			{
				posRow = Q[posRow].idx;//0
				posColumn=posColumn-1;		//0
				Q=dArrPointerEmbedding[posColumn];
				//printf("\n posColumn[%d] Row[%d] (idx:%d vid:%d)",posColumn,posRow,Q[posRow].idx,Q[posRow].vid);//Q[0][0]
				//printf("\nj:%d toVid:%d Q.vid:%d",j,toVid,Q[posRow].vid);
				if(toVid==Q[posRow].vid) {
					b=false; break;
				}
				posRow=Q[posRow].idx;//-1
				//printf("\nposRow:%d",posRow);
				if(posRow==-1) break;
			}
			if (b==false){b=true; continue;}
			int indexOfd_arr_V=i*maxDegreeOfVer+j;
			//printf("\nThread %d: m:%d",i,maxDegreeOfVer);
			int indexOfd_LN=indexToVidIndN;
			dArrV[indexOfd_arr_V].valid=1;
			printf("\ndArrV[%d].valid:%d",indexOfd_arr_V,dArrV[indexOfd_arr_V].valid);
			//cập nhật dữ liệu cho mảng dArrExtension
			dArrExtension[indexOfd_arr_V].vgi=vid;
			dArrExtension[indexOfd_arr_V].vgj=toVid;
			dArrExtension[indexOfd_arr_V].lij=d_LN[indexOfd_LN];
			printf("\n");
			printf("d_LN[%d]:%d ",indexOfd_LN,d_LN[indexOfd_LN]);
			dArrExtension[indexOfd_arr_V].li=labelFromVid;
			dArrExtension[indexOfd_arr_V].lj=labelToVid;
			dArrExtension[indexOfd_arr_V].vi=idxQ;
			dArrExtension[indexOfd_arr_V].vj=maxid+1;
			dArrExtension[indexOfd_arr_V].posColumn=col;
			dArrExtension[indexOfd_arr_V].posRow=row;
		}
	}
}

//kernel in mảng struct_V *dArrV trên device
__global__ void kernelprintdArrV(struct_V *dArrV,int noElem_dArrV,EXT *dArrExtension){
	int i = blockDim.x *blockIdx.x + threadIdx.x;
	if(i<noElem_dArrV){
		int vi = dArrExtension[i].vi;
		int vj = dArrExtension[i].vj;
		int li = dArrExtension[i].li;
		int lij = dArrExtension[i].lij;
		int lj = dArrExtension[i].lj;
		printf("\n dArrV[%d].backward:%d ,dArrV[%d].valid:%d Extension:(vgi:%d,vgj:%d) (vi:%d vj:%d li:%d lij:%d lj:%d)",i,dArrV[i].backward,i,dArrV[i].valid,dArrExtension[i].vgi,dArrExtension[i].vgj,vi,vj,li,lij,lj);
	}

}

//Hàm in mảng struct_V *dArrV
inline hipError_t printdArrV(struct_V *dArrV,int noElem_dArrV,EXT *dArrExtension){
	hipError_t cudaStatus;
	dim3 block(blocksize);
	dim3 grid((noElem_dArrV + block.x -1 )/block.x);
	kernelprintdArrV<<<grid,block>>>(dArrV,noElem_dArrV,dArrExtension);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() in printdArrV() failed", cudaStatus);
		goto Error;
	}
Error:
	return cudaStatus;
}
//kernel trích phần tử valid từ mảng dArrV và lưu vào mảng dArrValid
__global__ void kernelExtractValidFromdArrV(struct_V *dArrV,int noElem_dArrV,int *dArrValid){
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if(i<noElem_dArrV){
		dArrValid[i]=dArrV[i].valid;
	}
}

//kernel trích các mở rộng hợp lệ từ mảng dArrExtension sang mảng dExt
__global__ void kernelExtractValidExtensionTodExt(EXT *dArrExtension,int *dArrValid,int *dArrValidScanResult,int noElem_dArrV,EXT *dExt,int noElem_dExt){
	int i =blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElem_dArrV){
		if(dArrValid[i]==1){
			dExt[dArrValidScanResult[i]].vi = dArrExtension[i].vi;
			dExt[dArrValidScanResult[i]].vj = dArrExtension[i].vj;
			dExt[dArrValidScanResult[i]].li = dArrExtension[i].li;
			dExt[dArrValidScanResult[i]].lij = dArrExtension[i].lij;
			dExt[dArrValidScanResult[i]].lj = dArrExtension[i].lj;
			dExt[dArrValidScanResult[i]].vgi = dArrExtension[i].vgi;
			dExt[dArrValidScanResult[i]].vgj = dArrExtension[i].vgj;
			dExt[dArrValidScanResult[i]].posColumn = dArrExtension[i].posColumn;
			dExt[dArrValidScanResult[i]].posRow = dArrExtension[i].posRow;
		}

	}

}

//Kernel in nội dung mảng EXT *dExt
__global__ void kernelPrintdExt(EXT *dExt,int noElem_dExt){
	int i = blockDim.x *blockIdx.x + threadIdx.x;
	if(i<noElem_dExt){		
		int vi=dExt[i].vi;
		int vj=dExt[i].vj;
		int li= dExt[i].li;
		int lij=dExt[i].lij;
		int lj=dExt[i].lj;
		int vgi=dExt[i].vgi;
		int vgj=dExt[i].vgj;
		int posColumn= dExt[i].posColumn;
		int posRow=dExt[i].posRow;
		printf("\n Thread %d (vi:%d vj:%d li:%d lij:%d lj:%d) (vgi:%d vgj:%d) (posColumn:%d posRow:%d)",i,vi,vj,li,lij,lj,vgi,vgj,posColumn,posRow);
	}

}

//Hàm in dExt
inline hipError_t printdExt(EXT *dExt,int noElem_dExt){
	hipError_t cudaStatus;

	dim3 block(blocksize);
	dim3 grid((noElem_dExt+block.x -1)/block.x);
	kernelPrintdExt<<<grid,block>>>(dExt,noElem_dExt);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() kernelPrintdExt in printdExt() failed", cudaStatus);
		goto Error;
	}

Error:
	return cudaStatus;
}

//Hàm trích các mở rộng hợp lệ từ mảng dArrExtension sang mảng dExt
inline hipError_t extractValidExtensionTodExt(EXT *dArrExtension,struct_V *dArrV,int noElem_dArrV,EXT *&dExt,int &noElem_dExt){
	hipError_t cudaStatus;
	//1. Trích dữ liệu ra mảng dArrvalid
	int *dArrValid = nullptr;

	cudaStatus = hipMalloc((void**)&dArrValid, noElem_dArrV*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc dArrValid in extractValidExtensionTodExt() failed", cudaStatus);
		goto Error;
	}
	dim3 block(blocksize);
	dim3 grid((noElem_dArrV + block.x -1)/block.x);
	kernelExtractValidFromdArrV<<<grid,block>>>(dArrV,noElem_dArrV,dArrValid);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() kernelExtractValidFromdArrV in extractValidExtensionTodExt() failed", cudaStatus);
		goto Error;
	}
	//In nội dung dArrValid
	printf("\n********dArrValid******\n");
	printInt(dArrValid,noElem_dArrV);

	//2. Scan mảng dArrValid để lấy kích thước của mảng cần tạo
	int *dArrValidScanResult = nullptr;
	
	cudaStatus = hipMalloc((void**)&dArrValidScanResult,sizeof(int)*noElem_dArrV);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\n CudaMalloc dArrValidScanResult in extractValidExtensionToExt() failed");
		goto Error;
	}
	else
	{
		hipMemset(dArrValidScanResult,0,sizeof(int)*noElem_dArrV);
	}


	cudaStatus = scanV(dArrValid,noElem_dArrV,dArrValidScanResult);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\n scanV dArrValid in extractValidExtensionToExt() failed");
		goto Error;
	}

	//In nội dung kết quả dArrValidScanResult
	printf("\n********dArrValidScanResult******\n");
	printInt(dArrValidScanResult,noElem_dArrV);

	//3. Lấy kích thước của mảng EXT *dExt;
	noElem_dExt=0;
	cudaStatus=getSizeBaseOnScanResult(dArrValid,dArrValidScanResult,noElem_dArrV,noElem_dExt);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\n getSizeBaseOnScanResult in extractValidExtensionToExt() failed");
		goto Error;
	}

	//In nội dung noElem_dExt
	printf("\n******** noElem_dExt ******\n");
	printf("\n noElem_dExt:%d",noElem_dExt);


	//4. Khởi tạo mảng dExt có kích thước noElem_dExt rồi trích dữ liệu từ dArrExtension sang dựa vào dArrValid.
	cudaStatus = hipMalloc((void**)&dExt,sizeof(EXT)*noElem_dExt);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc dExt in extractValidExtensionTodExt() failed", cudaStatus);
		goto Error;
	}
	else
	{
		hipMemset(dExt,0,sizeof(EXT)*noElem_dExt);
	}
	dim3 blockb(blocksize);
	dim3 gridb((noElem_dArrV+blockb.x -1)/blockb.x);
	kernelExtractValidExtensionTodExt<<<gridb,blockb>>>(dArrExtension,dArrValid,dArrValidScanResult,noElem_dArrV,dExt,noElem_dExt);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() kernelExtractValidExtensionTodExt in extractValidExtensionTodExt() failed", cudaStatus);
		goto Error;
	}
	
	//In mảng dExt;
	printf("\n********** dExt **********\n");
	cudaStatus =printdExt(dExt,noElem_dExt);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n printdExt() in extractValidExtensionTodExt() failed", cudaStatus);
		goto Error;
	}
		
Error:
	hipFree(dArrValid);
	hipFree(dArrValidScanResult);
	return cudaStatus;
}

//Hàm Tìm tất cả các mở rộng hợp lệ forward từ các đỉnh trên cột Q và lưu vào mảng dExt và noElem_dExt
inline hipError_t forwardExtensionQ(Embedding **dArrPointerEmbedding,int noElem_dArrPointerEmbedding,int *dArrSizedQ,int noElem_dArrSizedQ,int noElem_Embedding,int idxQ,EXT *&dExt,int &noElem_dExt,int *d_O,int *d_LO,int *d_N,int *d_LN,int numberOfElementd_O,int numberOfElementd_N,unsigned int maxOfVer,int minLabel,int maxid){
	hipError_t cudaStatus;

	//Tìm bậc lớn nhất của các đỉnh vid trong cột Q
	int maxDegreeOfVer=0;
	float *dArrDegreeOfVid=nullptr; //Được sử dụng để tìm các mở rộng từ các vid trên column Q
	cudaStatus = findMaxDegreeOfVer(dArrPointerEmbedding,idxQ,d_O,numberOfElementd_O,noElem_Embedding, numberOfElementd_N,maxOfVer,maxDegreeOfVer,dArrDegreeOfVid);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n findMaxDegreeOfVer() in forwardExtensionQ() failed",cudaStatus);
		goto Error;
	}

	//Tạo mảng dArrV có số lượng phần tử bằng số lượng embedding nhân với bậc lớn nhất của các vid vừa tìm được
	//Tạo mảng d_arr_V có kích thước: maxDegree_vid_Q * |Q|
	//	Lưu ý, mảng d_arr_V phải có dạng cấu trúc đủ thể hiện cạnh mở rộng có hợp lệ hay không và là forward extension hay backward extension.
	//	struct struct_V
	//	{
	//		int valid; //default: 0, valid: 1
	//		int backward; //default: 0- forward; backward: 1
	//	}

	struct_V *dArrV;
	int noElem_dArrV=maxDegreeOfVer*noElem_Embedding;
	cudaStatus=hipMalloc((void**)&dArrV,noElem_dArrV*sizeof(struct_V));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc dArrV in  failed");
		goto Error;
	}
	else
	{
		hipMemset(dArrV,0,noElem_dArrV*sizeof(struct_V));
	}

	//Các mở rộng hợp lệ sẽ được ghi nhận vào mảng dArrV, đồng thời thông tin của cạnh mở rộng gồm dfscode, vgi, vgj và row pointer của nó cũng được xây dựng
	//và lưu trữ trong mảng EXT *dExtension, mảng này có số lượng phần tử bằng với mảng dArrV. Sau đó chúng ta sẽ rút trích những mở rộng hợp lệ này và lưu vào dExt. 
	//Để xây dựng dfscode (vi,vj,li,lij,lj) thì chúng ta cần:
	// - Dựa vào giá trị của right most path để xác định vi
	// - Dựa vào maxid để xác định vj
	// - Dựa vào CSDL để xác định các thành phần còn lại.
	//Chúng ta có thể giải phóng bộ nhớ của dExtension sau khi đã trích các mở rộng hợp lệ thành công.


	EXT *dArrExtension= nullptr;
	cudaStatus = hipMalloc((void**)&dArrExtension,noElem_dArrV*sizeof(EXT));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc dArrExtension forwardExtensionQ() failed",cudaStatus);
		goto Error;
	}
	else
	{
		hipMemset(dArrExtension,0,noElem_dArrV*sizeof(EXT));
	}

	printf("\nnoElem_dArrV:%d",noElem_dArrV );


	//Gọi kernel với các đối số: CSDL, bậc của các đỉnh, dArrV, dArrExtension,noElem_Embedding,maxDegreeOfVer,idxQ,dArrPointerEmbedding,minLabel,maxid
	dim3 block(blocksize);
	dim3 grid((noElem_Embedding+block.x - 1)/block.x);
	kernelFindValidForwardExtension<<<grid,block>>>(dArrPointerEmbedding,noElem_dArrPointerEmbedding,noElem_Embedding,d_O,d_LO,d_N,d_LN,dArrDegreeOfVid,maxDegreeOfVer,dArrV,dArrExtension,idxQ,minLabel,maxid);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() kernelFindValidForwardExtension in forwardExtensionQ() failed",cudaStatus);
		goto Error;
	}
	//In mảng dArrV để kiểm tra thử
	/*printf("\n****************dArrV*******************\n");
	cudaStatus = printdArrV(dArrV,noElem_dArrV,dArrExtension);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n printdArrV() in forwardExtensionQ() failed",cudaStatus);
		goto Error;
	}*/
	//Chép kết quả từ dArrExtension sang dExt

	cudaStatus =extractValidExtensionTodExt(dArrExtension,dArrV,noElem_dArrV,dExt,noElem_dExt);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n extractValidExtensionTodExt() in forwardExtensionQ() failed");
		goto Error;
	}

Error:

	hipFree(dArrExtension);
	hipFree(dArrV);
	return cudaStatus;
}

//kernel lấy chép địa chỉ của dExt lưu vào dArrPointerExt
__global__ void kernelGetPointerExt(EXT **dArrPointerExt,EXT *dExt,int pos){
	dArrPointerExt[pos]=dExt;
}


//Tìm tất cả các mở rộng hợp lệ forward và lưu vào mảng dArrPointerExt
inline hipError_t forwardExtension(Embedding **dArrPointerEmbedding,int noElem_dArrPointerEmbedding,int *dArrSizedQ,int noElem_dArrSizedQ,int *dRMPath,int noElem_dRMPath,int *d_O,int *d_LO,int *d_N,int *d_LN,int numberOfElementd_O,int numberOfElementd_N,unsigned int maxOfVer,EXT **&dArrPointerExt,int &noElem_dArrPointerExt,int minLabel,int maxid,int *&dArrNoElemPointerExt){
	hipError_t cudaStatus;

	//Lấy số lượng embedding
#pragma region "get noElem_Embedding"
	int noElem_Embedding = 0;
	cudaStatus = findNumberOfEmbedding(dArrSizedQ,noElem_dArrSizedQ,noElem_Embedding);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n findNumberOfEmbedding() in forwardExtension() failed",cudaStatus);
		goto Error;
	}

	//printf("\n noElem_Embedding:%d",noElem_Embedding);
#pragma endregion

	//Duyệt qua các column Q thuộc dRMPath và tìm các mở rộng hợp lệ từ chúng
	int *hRMPath =(int*)malloc(sizeof(int)*noElem_dRMPath);
	if (hRMPath==NULL){
		printf("\n malloc hRMPath in forwardExtension() failed");
		exit(1);
	}

	cudaStatus = hipMemcpy(hRMPath,dRMPath,sizeof(int)*noElem_dRMPath,hipMemcpyDeviceToHost);
	if (cudaStatus !=hipSuccess){
		fprintf(stderr,"\n hipMemcpy dRMPath --> hRMPath failed",cudaStatus);
		goto Error;
	}

	printf("\n ***************** hRMPath **************\n");
	for (int i = 0; i < noElem_dRMPath; i++)
	{
			printf("\n hRMPath[%d]:%d",i,hRMPath[i]);

	}

	cudaStatus = hipMalloc((void**)&dArrPointerExt,noElem_dRMPath*sizeof(EXT*));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc dArrPointerExt in forwardExtension() failed",cudaStatus);
		goto Error;		
	}
	else
	{
		hipMemset(dArrPointerExt,0,noElem_dRMPath*sizeof(EXT*));
	}

	int *hArrNoElemPointerExt;
	hArrNoElemPointerExt = (int*)malloc(sizeof(int)*noElem_dRMPath);
	if(hArrNoElemPointerExt==NULL){
		printf("\nMalloc hArrNoElemPointerExt in kernel.cu failed");
		goto Error;
	}
	else
	{
		memset(hArrNoElemPointerExt,0,sizeof(int)*noElem_dRMPath);
	}

	cudaStatus = hipMalloc((void**)&dArrNoElemPointerExt,sizeof(int)*noElem_dRMPath);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc dArrNoElemPointerExt in forwardExtension() failed",cudaStatus);
		goto Error;		
	}
	else
	{
		hipMemset(dArrNoElemPointerExt,0,sizeof(int)*noElem_dRMPath);
	}

	for (int i = noElem_dRMPath-1; i>=0  ; i--)
	{
		int idxQ=hRMPath[i];
		printf("\n*********idxQ:%d***************\n",idxQ);
		EXT *dExt=nullptr; //Những mở rộng hợp lệ sẽ được trích sang mảng dExt
		int noElem_dExt=0;
		cudaStatus = forwardExtensionQ(dArrPointerEmbedding,noElem_dArrPointerEmbedding,dArrSizedQ,noElem_dArrSizedQ,noElem_Embedding,idxQ,dExt,noElem_dExt,d_O,d_LO,d_N,d_LN, numberOfElementd_O, numberOfElementd_N, maxOfVer,minLabel,maxid);
		if(cudaStatus!=hipSuccess){
			fprintf(stderr,"\n forwardExtensionQ() in forwardExtension() failed",cudaStatus);
			goto Error;		
		}
		//Chép pointer của dExt bỏ vào mảng dArrPointerExt
		hArrNoElemPointerExt[i]=noElem_dExt;
		kernelGetPointerExt<<<1,1>>>(dArrPointerExt,dExt,i);
		hipDeviceSynchronize();
		cudaStatus=hipGetLastError();
		if(cudaStatus!=hipSuccess){
			fprintf(stderr,"\n hipDeviceSynchronize() kernelGetPointerExt in forwardExtension() failed",cudaStatus);
			goto Error;
		}
	}

	//chép dữ liệu từ hArrNoElemPointerExt sang dArrNoElemPointerExt
	cudaStatus = hipMemcpy(dArrNoElemPointerExt,hArrNoElemPointerExt,sizeof(int)*noElem_dRMPath,hipMemcpyHostToDevice);
	if(cudaStatus!=hipSuccess){
			fprintf(stderr,"\n hipMemcpy() hArrNoElemPointerExt sang dArrNoElemPointerExt in forwardExtension() failed",cudaStatus);
			goto Error;
		}


	//hipDeviceSynchronize();
	//cudaStatus=hipGetLastError();
	//if(cudaStatus!=hipSuccess){
	//	fprintf(stderr,"\n hipDeviceSynchronize() in forwardExtension() failed",cudaStatus);
	//	goto Error;
	//}
Error:
	return cudaStatus;

}


//kernel in mảng dArrPointerExt
__global__ void kernelprintdArrPointerExt(EXT **dArrPointerExt,int *dArrNoElemPointerExt,int noElem_dArrPointerExt){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<noElem_dArrPointerExt){
		int noElem_dExt=dArrNoElemPointerExt[i];
		printf("\nThread %d: noElem_dExt:%d",i,noElem_dExt);
		if(noElem_dExt>0){
			EXT* dExt= dArrPointerExt[i];
			printf("\n dExt_value:%p dExt_address:%p ",dExt,&dExt);
			
			int length = dArrNoElemPointerExt[i];
			for (int i = 0; i < length; i++)
			{
				int vi=dExt[i].vi;
				int vj=dExt[i].vj;
				int li= dExt[i].li;
				int lij=dExt[i].lij;
				int lj=dExt[i].lj;
				int vgi=dExt[i].vgi;
				int vgj=dExt[i].vgj;
				int posColumn= dExt[i].posColumn;
				int posRow=dExt[i].posRow;
				printf("\n Thread %d (vi:%d vj:%d li:%d lij:%d lj:%d) (vgi:%d vgj:%d) (posColumn:%d posRow:%d)",i,vi,vj,li,lij,lj,vgi,vgj,posColumn,posRow);
			}
		}
	}
}

//Hàm in mảng dArrPointerExt
inline hipError_t printdArrPointerExt(EXT **dArrPointerExt,int *dArrNoElemPointerExt,int noElem_dArrPointerExt){
	hipError_t cudaStatus;
	
	dim3 block(blocksize);
	dim3 grid((noElem_dArrPointerExt + block.x - 1)/block.x);
	kernelprintdArrPointerExt<<<grid,block>>>(dArrPointerExt,dArrNoElemPointerExt,noElem_dArrPointerExt);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaDeviceSynchronize() in printArrPointerdExt() failed");
		goto Error;
	}
Error:
	return cudaStatus;
}

//Hàm giải phóng bộ nhớ Ext** dArrPointerExt và dArr
inline hipError_t cudaFreeArrPointerExt(EXT **&dArrPointerExt,int *&dArrNoElemPointerExt,int noElem_dArrPointerExt){
	hipError_t cudaStatus;
	EXT **hArrPointerExt=nullptr;
	hArrPointerExt = (EXT**)malloc(sizeof(EXT*)*noElem_dArrPointerExt);
	if(hArrPointerExt==NULL){
		printf("\n malloc hArrPointerExt in cudaFreeArrpointerExt failed"),
		exit(1);
	}

	cudaStatus = hipMemcpy(hArrPointerExt,dArrPointerExt,noElem_dArrPointerExt*sizeof(EXT*),hipMemcpyDeviceToHost);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\hipMemcpy() in printArrPointerdExt() failed");
		goto Error;
	}

	int length = noElem_dArrPointerExt;
	for (int i = 0; i < length; i++)
	{
		if (hArrPointerExt[i]!=NULL){
			hipFree(hArrPointerExt[i]);
		}
	}
	hipFree(dArrPointerExt);
	hipFree(dArrNoElemPointerExt);

	
Error:
	return cudaStatus;
}


//Hàm giải phóng bộ nhớ Embedding *dArrPointerEmbedding và dArrSizeQ
inline hipError_t cudaFreeArrPointerEmbedding(Embedding **&dArrPointerEmbedding,int *&dArrSizedQ,int noElem_dArrPointerEmbedding){
	hipError_t cudaStatus;
	Embedding **hArrPointerExt=nullptr;
	hArrPointerExt = (Embedding**)malloc(sizeof(Embedding*)*noElem_dArrPointerEmbedding);
	if(hArrPointerExt==NULL){
		printf("\n malloc hArrPointerExt in cudaFreeArrPointerEmbedding() failed"),
		exit(1);
	}

	cudaStatus = hipMemcpy(hArrPointerExt,dArrPointerEmbedding,noElem_dArrPointerEmbedding*sizeof(Embedding*),hipMemcpyDeviceToHost);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\hipMemcpy() in cudaFreeArrPointerEmbedding() failed");
		goto Error;
	}

	int length = noElem_dArrPointerEmbedding;
	for (int i = 0; i < length; i++)
	{
		if (hArrPointerExt[i]!=NULL){
			hipFree(hArrPointerExt[i]);
		}
	}
	hipFree(dArrPointerEmbedding);
	hipFree(dArrSizedQ);	
Error:
	return cudaStatus;
}

//Kernel ánh xạ nhãn cạnh sang vị trí tương ứng trong dArrAllPossibleExtension và set giá trị tại đó bằng 1
__global__ void kernelassigndAllPossibleExtension(EXT **dArrPointerExt,int posdArrPointerExt,int Lv,int Le,int *dArrAllPossibleExtension,int noElem_PointerExt){
	int i = blockDim.x *blockIdx.x + threadIdx.x;
	if(i<noElem_PointerExt){
		int lij,lj;
		lij=dArrPointerExt[posdArrPointerExt][i].lij;
		lj=dArrPointerExt[posdArrPointerExt][i].lj;
		int idx=lij*Lv+lj;
		dArrAllPossibleExtension[idx]=1;
	}
}

//Hàm duyệt qua các phần tử trong mảng dExt và set giá trị 1 tại vị trí tương ứng trong mảng kết quả dArrAllPossibleExtension
inline hipError_t assigndAllPossibleExtension(EXT **dArrPointerExt,int posdArrPointerExt,int Lv,int Le,int *dArrAllPossibleExtension,int noElem_PointerExt){
	hipError_t cudaStatus;

	dim3 block(blocksize);
	dim3 grid((noElem_PointerExt+block.x -1)/block.x);
	kernelassigndAllPossibleExtension<<<grid,block>>>(dArrPointerExt, posdArrPointerExt, Lv, Le,dArrAllPossibleExtension,noElem_PointerExt);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize kernelassigndAllPossibleExtension in assigndAllPossibleExtension() failed");
		goto Error;
	}

	//In nội dung dArrAllPossibleExtension
	cudaStatus = printInt(dArrAllPossibleExtension,Lv*Le);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n printInt(dArrAllPossibleExtension,Lv*Le) in assigndAllPossibleExtension() failed");
		goto Error;
	}

	
Error:
	return cudaStatus;
}

//Kernel gán giá trị cho mảng dArrUniEdge
__global__ void kernelassigndArrUniEdge(int *dArrAllPossibleExtension,int *dArrAllPossibleExtensionScanResult,int noElem_dArrAllPossibleExtension,UniEdge *dArrUniEdge,int Lv,int *dFromLi){
	int i = blockDim.x*blockIdx.x +threadIdx.x;
	if(i<noElem_dArrAllPossibleExtension){
		if(dArrAllPossibleExtension[i]==1){
			int li,lij,lj;
			li=dFromLi[0];
			lij = i/Lv;
			lj=i%Lv;
			dArrUniEdge[dArrAllPossibleExtensionScanResult[i]].li=li;
			dArrUniEdge[dArrAllPossibleExtensionScanResult[i]].lij=lij;
			dArrUniEdge[dArrAllPossibleExtensionScanResult[i]].lj=lj;
		}
	}
}

//Hàm gán giá trị cho mảng dArrUniEdge
inline hipError_t assigndArrUniEdge(int *dArrAllPossibleExtension,int *dArrAllPossibleExtensionScanResult,int noElem_dArrAllPossibleExtension,UniEdge *&dArrUniEdge,int Lv,int *dFromLi){
	hipError_t cudaStatus;

	dim3 block(blocksize);
	dim3 grid((noElem_dArrAllPossibleExtension+block.x-1)/block.x);

	kernelassigndArrUniEdge<<<grid,block>>>(dArrAllPossibleExtension,dArrAllPossibleExtensionScanResult,noElem_dArrAllPossibleExtension,dArrUniEdge,Lv,dFromLi);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() in assigndArrUniEdge() failed");
		goto Error;
	}	
Error:
	return cudaStatus;}

//kernel lấy nhãn from Li
__global__ void kernelGetFromLabel(EXT **dArrPointerExt,int pos,int *dFromLi){
	dFromLi[0]	= dArrPointerExt[pos][0].li;
}

//kernel getPointerUniEdge
__global__ void kernelGetPointerUniEdge(UniEdge **dArrPointerUniEdge,UniEdge *dArrUniEdge,int pos){
	dArrPointerUniEdge[pos]=dArrUniEdge;
}


//Hàm trích các mở rộng duy nhất và lưu kết quả vào mảng dArrPointerUniEdge, mỗi phần tử của nó là một pointer trỏ đến mảng dArrUniEdge trên device
inline hipError_t extractUniExtension(EXT **dArrPointerExt,int noElem_dArrPointerExt,int Lv,int Le,UniEdge **&dArrPointerUniEdge,int noElem_dArrPointerUniEdge,int *&dArrNoELemPointerUniEdge,int *hArrNoElemPointerExt,int *dArrNoElemPointerExt){
	hipError_t cudaStatus;
	/*Duyệt qua từng EXTk để thực hiện rút trích và lưu kết quả vào UniEdge **dArrPointerUniEdge
	* Mỗi phần tử của mảng UniEdge **dArrPointerUniEdge là một pointer, chính là kết quả của 1 lần xử lý EXTk
	*	Trích các unique forward extention lưu vào dUniqueEdgeForward
	*	Trích các unique backward extension lưu vào dUniqueEdgeBackward (Backward Extension chỉ tồn tại ở EXTk cuối)
	*/

	//1. Khởi tạo mảng UniEdge **dArrPointerUniEdge với số lượng phần tử bằng kích thước dRMPath				
	//Cấp phát bộ nhớ cho mảng dArrPointerUniEdge
	cudaStatus=hipMalloc((void**)&dArrPointerUniEdge,sizeof(UniEdge*)*noElem_dArrPointerUniEdge);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc dArrPointerUniEdge in extractUniExtension() failed",cudaStatus);
		goto Error;
	}
	else
	{
		hipMemset(dArrPointerUniEdge,0,sizeof(UniEdge*)*noElem_dArrPointerUniEdge);
	}
	//Cấp phát bộ nhớ cho mảng dArrNoELemPointerUniEdge
	cudaStatus=hipMalloc((void**)&dArrNoELemPointerUniEdge,sizeof(int)*noElem_dArrPointerUniEdge);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc dArrPointerUniEdge in extractUniExtension() failed",cudaStatus);
		goto Error;
	}
	else
	{
		hipMemset(dArrNoELemPointerUniEdge,0,sizeof(int)*noElem_dArrPointerUniEdge);
	}
	
	int *hArrNoELemPointerUniEdge=(int*)malloc(sizeof(int)*noElem_dArrPointerUniEdge); //Nơi lưu trữ tạm thời phải được giải phóng cuối hàm này, dữ liệu sẽ được chép sang bộ nhớ dArrNoELemPointerUniEdge
	if(hArrNoELemPointerUniEdge==NULL){
		printf("\n Malloc hArrNoELemPointerUniEdge in extractUniExtension() failed");
		exit(1);
	}
	else
	{
		memset(hArrNoELemPointerUniEdge,0,sizeof(int)*noElem_dArrPointerUniEdge);
	}


	for (int i = 0; i < noElem_dArrPointerExt; i++)
	{		
		//Khai báo bộ nhớ dArrAllPossibleExtension và số lượng phần tử của nó
		int *dArrAllPossibleExtension =nullptr; //Phải được giải phóng bên trong vòng for sau khi dùng xong
		int noElem_dArrAllPossibleExtension = Lv*Le;

		//printf("\n hArrNoElemPointerExt:%d",hArrNoElemPointerExt[i]);
		//Nếu số lượng phần tử tại EXTk lớn hơn bằng minsup thì mới duyệt.
		//Ngon hơn nữa thì xét Số lượng phần tử phân biệt trong EXTk >= minsup thì mới duyệt
		if(hArrNoElemPointerExt[i]>0){

			int *dFromLi;
			cudaStatus = hipMalloc((void**)&dFromLi,sizeof(int));
			if(cudaStatus!=hipSuccess){
				fprintf(stderr,"\n hipMalloc dFromLi in extractUniExtension failed");
				goto Error;
			}
			else
			{
				hipMemset(dFromLi,0,sizeof(int));
			}


			//lấy nhãn Li lưu vào biến dFromLi
			kernelGetFromLabel<<<1,1>>>(dArrPointerExt,i,dFromLi);
			hipDeviceSynchronize();
			cudaStatus=hipGetLastError();
			if(cudaStatus!=hipSuccess){
				fprintf(stderr,"\n hipDeviceSynchronize()  kernelGetFromLabel in extracUniExtension failed");
				goto Error;
			}
						
			//Hiển thị nội dung nhãn dFromLi
			printf("\n ****dFrom *******\n");
			cudaStatus =printInt(dFromLi,1);
			if(cudaStatus!=hipSuccess){
				fprintf(stderr,"\n printInt(dFromLi,1) in extracUniExtension failed");
				goto Error;
			}


			UniEdge * dArrUniEdge=nullptr;
			int noElem_dArrUniEdge=0;

			//Khởi tạo một mảng dArrAllPossileExtension có kích thước bằng Lv*Le với giá trị là zero
			cudaStatus=hipMalloc((void**)&dArrAllPossibleExtension,noElem_dArrAllPossibleExtension*sizeof(int));
			if(cudaStatus!=hipSuccess){
				fprintf(stderr,"\n hipMalloc((void**)&dArrAllPossibleExtension in extractUniExtension() failed",cudaStatus);
				goto Error;
			}
			else
			{
				hipMemset(dArrAllPossibleExtension,0,noElem_dArrAllPossibleExtension*sizeof(int));
			}
			//Gọi hàm assigndAllPossibleExtension để ánh xạ (li,lij,lj) sang vị trí trên mảng dArrAllPossibleExtension và set 1 value tại index đó.
			//Gọi kernel gồm hArrNoElemPointerExt[i] threads, mỗi thread sẽ đọc nhãn li,lij,lj và ánh xạ thành vị trí tương ứng trên mảng dArrAllPossibleExtension
			//đồng thời set giá trị 1 tại vị trí trên mảng dArrAllPossibleExtension.
			//
			cudaStatus = assigndAllPossibleExtension(dArrPointerExt,i,Lv,Le,dArrAllPossibleExtension,hArrNoElemPointerExt[i]);
			if(cudaStatus!=hipSuccess){
				fprintf(stderr,"\n assigndAllPossibleExtension in extractUniExtension() failed",cudaStatus);
				goto Error;
			}
			
			//Scan mảng dArrAllPossibleExtension để biết kích thước của mảng dArrUniEdge và ánh xạ từ vị trí trong dArrAllPossibleExtension thành nhãn để lưu vào dArrUniEdge
			int *dArrAllPossibleExtensionScanResult =nullptr;
			cudaStatus = hipMalloc((void**)&dArrAllPossibleExtensionScanResult,noElem_dArrAllPossibleExtension*sizeof(int));
			if(cudaStatus!=hipSuccess){
				fprintf(stderr,"\n hipMalloc dArrAllPossibleExtensionScanResult in extractUniExtension() failed",cudaStatus);
				goto Error;
			}
			cudaStatus = scanV(dArrAllPossibleExtension,noElem_dArrAllPossibleExtension,dArrAllPossibleExtensionScanResult);
			if(cudaStatus!=hipSuccess){
				fprintf(stderr,"\n scanV dArrAllPossibleExtension in extractUniExtension() failed",cudaStatus);
				goto Error;
			}

			//Tính kích thước của dArrUniEdge và lưu vào noElem_dArrUniEdge
			cudaStatus =getSizeBaseOnScanResult(dArrAllPossibleExtension,dArrAllPossibleExtensionScanResult,noElem_dArrAllPossibleExtension,noElem_dArrUniEdge);
			if(cudaStatus!=hipSuccess){
				fprintf(stderr,"\n scanV dArrAllPossibleExtension in extractUniExtension() failed",cudaStatus);
				goto Error;
			}

			//HIển thị giá trị của noElem_dArrUniEdge
			printf("\n******noElem_dArrUniEdge************\n");
			//printf("\n noElem_dArrUniEdge:%d",noElem_dArrUniEdge);

			//Cấp phát bộ nhớ cho dArrUniEdge
			cudaStatus = hipMalloc((void**)&dArrUniEdge,noElem_dArrUniEdge*sizeof(UniEdge));
			if(cudaStatus!=hipSuccess){
				fprintf(stderr,"\n hipMalloc dArrUniEdge in extractUniExtension() failed",cudaStatus);
				goto Error;
			}

			//Gọi hàm để ánh xạ dữ liệu từ dArrAllPossibleExtension sang mảng dArrUniEdge
			/* Input Data:	dArrAllPossibleExtension, dArrAllPossibleExtensionScanResult,  */
			cudaStatus =assigndArrUniEdge(dArrAllPossibleExtension,dArrAllPossibleExtensionScanResult,noElem_dArrAllPossibleExtension,dArrUniEdge,Lv,dFromLi);
			if(cudaStatus!=hipSuccess){
				fprintf(stderr,"\n assigndArrUniEdge in extractUniExtension() failed",cudaStatus);
				goto Error;
			}

			//In nội dung mảng dArrUniEdge
			printf("\n**********printf************");
			printfUniEdge(dArrUniEdge,noElem_dArrUniEdge);

			//Lưu lại số lượng cạnh duy nhất
			hArrNoELemPointerUniEdge[i]=noElem_dArrUniEdge;

			kernelGetPointerUniEdge<<<1,1>>>(dArrPointerUniEdge,dArrUniEdge,i);
			hipDeviceSynchronize();

			
			hipFree(dArrAllPossibleExtensionScanResult);
			hipFree(dFromLi);
		}		//end if
		hipFree(dArrAllPossibleExtension);
	} //end for 

	hipMemcpy(dArrNoELemPointerUniEdge,hArrNoELemPointerUniEdge,sizeof(int)*noElem_dArrPointerUniEdge,hipMemcpyHostToDevice);
	
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() in extractUniExtension() failed");
		goto Error;
	}
Error:
	free(hArrNoELemPointerUniEdge);
	return cudaStatus;
}

//kernel in nội dung mảngdArrPointerUniEdge
__global__ void kernelprintArrPointerUniEdge(UniEdge **dArrPointerUniEdge,int *dArrNoELemPointerUniEdge,int noElem_dArrPointerUniEdge){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElem_dArrPointerUniEdge){
		if(dArrNoELemPointerUniEdge[i]!=0){
			UniEdge * dArrUniEdge = dArrPointerUniEdge[i];
			int n = dArrNoELemPointerUniEdge[i];
			for (int j = 0; j < n; j++)
			{
				printf("\n Thread %d: j:%d (li:%d lij:%d lj:%d)",i,j,dArrUniEdge[j].li,dArrUniEdge[j].lij,dArrUniEdge[j].lj);
			}
		}
	}
}


//Hàm in nội dung mảngdArrPointerUniEdge
inline hipError_t printArrPointerUniEdge(UniEdge **dArrPointerUniEdge,int *dArrNoELemPointerUniEdge,int noElem_dArrPointerUniEdge){
	hipError_t cudaStatus;
	dim3 block(blocksize);
	dim3 grid((noElem_dArrPointerUniEdge+block.x-1)/block.x);
	
	kernelprintArrPointerUniEdge<<<grid,block>>>(dArrPointerUniEdge,dArrNoELemPointerUniEdge,noElem_dArrPointerUniEdge);
	hipDeviceSynchronize();
	cudaStatus= hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() kernelprintArrPointerUniEdge in printArrPointerUniEdge() failed");
		goto Error;
	}
Error:
	return cudaStatus;
}

//Hàm giải phóng bộ nhớ Ext** dArrPointerUniEdge và dArrNoELemPointerUniEdge
inline hipError_t cudaFreeArrPointerUniEdge(UniEdge **&dArrPointerUniEdge,int *&dArrNoELemPointerUniEdge,int noElem_dArrPointerUniEdge){
	hipError_t cudaStatus;
	UniEdge **hArrPointerUniEdge=nullptr;
	hArrPointerUniEdge = (UniEdge**)malloc(sizeof(EXT*)*noElem_dArrPointerUniEdge);
	if(hArrPointerUniEdge==NULL){
		printf("\n malloc hArrPointerExt in cudaFreeArrpointerExt failed"),
		exit(1);
	}

	cudaStatus = hipMemcpy(hArrPointerUniEdge,dArrPointerUniEdge,noElem_dArrPointerUniEdge*sizeof(EXT*),hipMemcpyDeviceToHost);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\hipMemcpy() in printArrPointerdExt() failed");
		goto Error;
	}

	int length = noElem_dArrPointerUniEdge;
	for (int i = 0; i < length; i++)
	{
		if (hArrPointerUniEdge[i]!=NULL){
			hipFree(hArrPointerUniEdge[i]);
		}
	}
	hipFree(dArrPointerUniEdge);
	hipFree(dArrNoELemPointerUniEdge);

Error:
	return cudaStatus;
}




__global__ void kernelExtractPointerUniEdge(UniEdge **dPointerArrUniEdge,UniEdge **dArrPointerUniEdge,int pos){
	dPointerArrUniEdge[0] = dArrPointerUniEdge[pos];
	printf("\nPointer UniEdge:%p",dArrPointerUniEdge[pos]);
}

__global__ void kernelExtractPointerExt(EXT **dPointerArrExt,EXT **dArrPointerExt,int pos,unsigned int noElemdArrExt){
	dPointerArrExt[0] = dArrPointerExt[pos];
	printf("\nPointer:%p",dArrPointerExt[pos]);
}

__global__ void kernelfindBoundary(EXT **dPointerArrExt,unsigned int noElemdArrExt,unsigned int *dArrBoundary,unsigned int maxOfVer){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	EXT *dArrExt = dPointerArrExt[0];
	if(i<noElemdArrExt-1){		
		unsigned int graphIdAfter=dArrExt[i+1].vgi/maxOfVer;
		unsigned int graphIdCurrent=dArrExt[i].vgi/maxOfVer;
		if(graphIdAfter!=graphIdCurrent){
			dArrBoundary[i]=1;
		}
	}
}

inline hipError_t findBoundary(EXT **dPointerArrExt,unsigned int noElemdArrExt,unsigned int *&dArrBoundary,unsigned int maxOfVer){
	hipError_t cudaStatus;
	dim3 block(blocksize);
	dim3 grid((noElemdArrExt+block.x-1)/block.x);


	kernelfindBoundary<<<grid,block>>>(dPointerArrExt,noElemdArrExt,dArrBoundary,maxOfVer);

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() in findBoundary() failed",cudaStatus);
		goto Error;
	}
Error:
	
	return cudaStatus;
}

__global__ void kernelPrint(EXT **dArrExt,unsigned int noElemdArrExt){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noElemdArrExt){
		EXT *arrExt = dArrExt[0];
		printf("\nPointer ext:%p",dArrExt[0]);
		printf("\n vgi:%d vgj:%d",arrExt[i].vgi,arrExt[i].vgj);
	}
}

__global__ void kernelPrintUE(UniEdge **dPointerArrUniEdge,unsigned int noElem){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if(i<noElem){
		UniEdge *arrUniEdge = dPointerArrUniEdge[0];
		printf("\nPointer ue:%p",dPointerArrUniEdge[0]);
		printf("\n UniEdge: li:%d, lij:%d, lj:%d)",arrUniEdge[i].li,arrUniEdge[i].lij,arrUniEdge[i].lj);
	}

}



__global__ void kernelFilldF(UniEdge **dPointerArrUniEdge,unsigned int pos,EXT **dPointerArrExt,unsigned int noElemdArrExt,unsigned int *dArrBoundaryScanResult,float *dF){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if(i<noElemdArrExt){
		UniEdge *dUniEdge = dPointerArrUniEdge[0];
		int li = dUniEdge[pos].li;
		int lij = dUniEdge[pos].lij;
		int lj = dUniEdge[pos].lj;
		EXT *dArrExt = dPointerArrExt[0];
		int Li = dArrExt[i].li;
		int Lij = dArrExt[i].lij;
		int Lj = dArrExt[i].lj;
		printf("\nThread %d: UniEdge(li:%d lij:%d lj:%d) (Li:%d Lij:%d Lj:%d)",i,li,lij,lj,Li,Lij,Lj);

		if(li==Li && lij==Lij && lj==Lj){
			dF[dArrBoundaryScanResult[i]]=1;
		}
	}
}


inline hipError_t calcSupport(UniEdge **dPointerArrUniEdge,unsigned int pos,EXT **dPointerArrExt,unsigned int noElemdArrExt,unsigned int *dArrBoundaryScanResult,float *dF,unsigned int noElemdF,float &support,unsigned int noElemdArrUniEdge){
	hipError_t cudaStatus;
	dim3 block(blocksize);
	dim3 grid((noElemdArrExt+block.x-1)/block.x);

	printf("\n**********dPointerArrExt***********\n");
	kernelPrint<<<1,noElemdArrExt>>>(dPointerArrExt,noElemdArrExt);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n kernelPrintExt  in computeSupportv2() failed");
		goto Error;
	}

	printf("\n**********dPointerArrUniEdge***********\n");
	kernelPrintUE<<<1,noElemdArrUniEdge>>>(dPointerArrUniEdge,noElemdArrUniEdge);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n kernelPrintUE  in computeSupportv2() failed");
		goto Error;
	}

	kernelFilldF<<<grid,block>>>(dPointerArrUniEdge,pos,dPointerArrExt,noElemdArrExt,dArrBoundaryScanResult,dF);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n 	hipDeviceSynchronize() kernelFilldF in calcSupport() failed",cudaStatus);
		goto Error;
	}
	printf("\n**********dF****************\n");
	printFloat(dF,noElemdF);

	reduction(dF,noElemdF,support);

	printf("\n******support********");
	printf("\n Support:%f",support);
	
	hipMemset(dF,0,noElemdF*sizeof(float));
Error:				
	return cudaStatus;
}


//Hàm tính độ hỗ trợ computeSupportv2
inline hipError_t computeSupportv2(EXT **dArrPointerExt,int *dArrNoElemPointerExt,int *hArrNoElemPointerExt,int noElem_dArrPointerExt,UniEdge **dArrPointerUniEdge,int *dArrNoELemPointerUniEdge,int *hArrNoELemPointerUniEdge,int noElem_dArrPointerUniEdge,unsigned int **&hArrPointerSupport,unsigned int *&hArrNoElemPointerSupport,unsigned int noElem_hArrPointerSupport,unsigned int maxOfVer){
	hipError_t cudaStatus;

	//Cấp phát bộ nhớ cho hArrPointerSupport. Mỗi phần tử là một địa chỉ trỏ đến 1 mảng kiểu unsigned int
	hArrPointerSupport = (unsigned int**)malloc(sizeof(unsigned int*)*noElem_hArrPointerSupport);
	if(hArrPointerSupport==NULL){
		printf("\n malloc hArrPointerSupport in kernel.cu failed");
		exit(1);
	}
	else
	{
		memset(hArrPointerSupport,0,sizeof(unsigned int*)*noElem_hArrPointerSupport);
	}

	hArrNoElemPointerSupport = (unsigned int*)malloc(sizeof(unsigned int)*noElem_hArrPointerSupport);
	if(hArrNoElemPointerSupport==NULL){
		printf("\n malloc hArrNoelemPointerSupport in computeSupportv2() failed");
		exit(1);
	}
	else
	{
		memset(hArrNoElemPointerSupport,0,sizeof(unsigned int)*noElem_hArrPointerSupport);
	}
	
	//Duyệt qua mảng các pointer trỏ đến mảng chứa các cạnh duy nhất. Mỗi vòng lặp j sẽ ứng với một segment EXTk, và mỗi EXTk sẽ có một boundary 
	for (int j = 0; j < noElem_dArrPointerUniEdge ; j++)
	{
		
		//Mảng dArrBoundary dùng để lưu trữ boundary của EXTk (ở đây là EXT thứ j theo như vòng lặp for bên dưới)
		unsigned int *dArrBoundary=nullptr;
		unsigned int *dArrBoundaryScanResult=nullptr;
		unsigned int noElemdArrBoundary=0; //Bằng với hArrNoElemPointerExt[j]
		if(hArrNoELemPointerUniEdge[j]>0){ //Nếu tồn tại unique edge tại dArrPointerUniEdge j đang xét thì tìm boundary của EXTk j tương ứng
			UniEdge **dPointerArrUniEdge=nullptr;
			unsigned int noElemdArrUniEdge = hArrNoELemPointerUniEdge[j];
			cudaStatus = hipMalloc((void**)&dPointerArrUniEdge,sizeof(UniEdge*));
			if(cudaStatus!=hipSuccess){
				fprintf(stderr,"\ncudaMalloc dPointerArrUniEdge in computeSupportv2() failed",cudaStatus);
				goto Error;
			}

			
			EXT **dPointerArrExt = nullptr;
			unsigned int noElemdArrExt = hArrNoElemPointerExt[j];
			cudaStatus = hipMalloc((void**)&dPointerArrExt,sizeof(EXT*));
			if(cudaStatus!=hipSuccess){
				fprintf(stderr,"\ncudaMalloc dPointerArrExt in computeSupportv2() failed",cudaStatus);
				goto Error;
			}
			//Hoạt động rút trích diễn ra song song
						
			kernelExtractPointerUniEdge<<<1,1>>>(dPointerArrUniEdge,dArrPointerUniEdge,j); //Trích phần tử  trong mảng dArrPointerUniEdge lưu vào biến dArrUniEdge để tiện tính toán
			hipDeviceSynchronize();
			kernelExtractPointerExt<<<1,1>>>(dPointerArrExt,dArrPointerExt,j,noElemdArrExt); //Trích phần tử trong mảng dArrPointerExt lưu vào biến dArrExt để tiện tính toán
			hipDeviceSynchronize();
			cudaStatus = hipGetLastError();
			if(cudaStatus!=hipSuccess){
				fprintf(stderr,"\n hipDeviceSynchronize() kernelExtractPointerExt kernelExtractPointerUniEdge in computeSupportv2() failed",cudaStatus);
				goto Error;
			}

			printf("\n**********dPointerArrExt***********\n");
			kernelPrint<<<1,noElemdArrExt>>>(dPointerArrExt,noElemdArrExt);
			hipDeviceSynchronize();
			cudaStatus=hipGetLastError();
			if(cudaStatus!=hipSuccess){
				fprintf(stderr,"\n kernelPrintExt  in computeSupportv2() failed");
				goto Error;
			}

			printf("\n**********dPointerArrUniEdge***********\n");
			kernelPrintUE<<<1,noElemdArrUniEdge>>>(dPointerArrUniEdge,noElemdArrUniEdge);
			hipDeviceSynchronize();
			cudaStatus=hipGetLastError();
			if(cudaStatus!=hipSuccess){
				fprintf(stderr,"\n kernelPrintUE  in computeSupportv2() failed");
				goto Error;
			}

			
					
#pragma region "find Boundary and scan Boundary"
			noElemdArrBoundary = noElemdArrExt;
			cudaStatus=hipMalloc((void**)&dArrBoundary,sizeof(unsigned int)*noElemdArrBoundary);
			if(cudaStatus!=hipSuccess){
				fprintf(stderr,"\n hipMalloc dArrBoundary in computeSupportv2() failed");
				goto Error;
			}
			else
			{
				hipMemset(dArrBoundary,0,sizeof(unsigned int)*noElemdArrBoundary);
			}

			cudaStatus=hipMalloc((void**)&dArrBoundaryScanResult,sizeof(unsigned int)*noElemdArrBoundary);
			if(cudaStatus!=hipSuccess){
				fprintf(stderr,"\n hipMalloc dArrBoundary in computeSupportv2() failed");
				goto Error;
			}
			else
			{
				hipMemset(dArrBoundaryScanResult,0,sizeof(unsigned int)*noElemdArrBoundary);
			}

			//Tìm boundary của EXTk và lưu kết quả vào mảng dArrBoundary
			cudaStatus = findBoundary(dPointerArrExt,noElemdArrExt,dArrBoundary,maxOfVer);
			if(cudaStatus!=hipSuccess){
				fprintf(stderr,"\n findBoundary() in computeSupportv2() failed");
				goto Error;
			}

			printf("\n ************* dArrBoundary ************\n");
			cudaStatus=printUnsignedInt(dArrBoundary,noElemdArrBoundary);
			if(cudaStatus!=hipSuccess){
				fprintf(stderr,"\n printUnsignedInt in computeSupportv2() failed", cudaStatus);
				goto Error;
			}

			//Scan dArrBoundary lưu kết quả vào dArrBoundaryScanResult
			cudaStatus=scanV(dArrBoundary,noElemdArrBoundary,dArrBoundaryScanResult);
			if(cudaStatus!=hipSuccess){
				fprintf(stderr,"\n Exclusive scan dArrBoundary in computeSupportv2() failed",cudaStatus);
				goto Error;
			}

			printf("\n**************dArrBoundaryScanResult****************\n");
			printUnsignedInt(dArrBoundaryScanResult,noElemdArrBoundary);

			float *dF=nullptr;
			unsigned int noElemdF = 0;

			cudaStatus = hipMemcpy(&noElemdF,&dArrBoundaryScanResult[noElemdArrBoundary-1],sizeof(unsigned int),hipMemcpyDeviceToHost);
			if(cudaStatus !=hipSuccess){
				fprintf(stderr,"\n cudamemcpy dF failed",cudaStatus);
				goto Error;
			}
			noElemdF++;
			printf("\n*****noElemdF******\n");
			printf("noElemdF:%d",noElemdF);

			cudaStatus = hipMalloc((void**)&dF,sizeof(unsigned int)*noElemdF);
			if(cudaStatus!=hipSuccess){
				fprintf(stderr,"\ncudaMalloc dF failed",cudaStatus);
				goto Error;
			}
			else
			{
				hipMemset(dF,0,sizeof(float)*noElemdF);
			}
#pragma endregion "end of finding Boundary"

			hArrNoElemPointerSupport[j]=noElemdArrUniEdge;
			unsigned int * hArrSupport = (unsigned int*)malloc(sizeof(unsigned int)*noElemdArrUniEdge);
			if(hArrSupport==NULL){
				printf("\n Malloc hArrSupport in computeSupportv2() failed");
				exit(1);
			}
			else
			{
				memset(hArrSupport,0,sizeof(unsigned int)*noElemdArrUniEdge);
			}
			//Duyệt và tính độ hỗ trợ của các cạnh
			for (int i = 0; i < noElemdArrUniEdge; i++)
			{					
				float support=0;
				cudaStatus =calcSupport(dPointerArrUniEdge,i,dPointerArrExt,noElemdArrExt,dArrBoundaryScanResult,dF,noElemdF,support,noElemdArrUniEdge);
				if(cudaStatus !=hipSuccess){
					fprintf(stderr,"\n calcSupport failed",cudaStatus);
					goto Error;
				}				
				hArrSupport[i]=support;
			}			
		    hArrPointerSupport[j]=hArrSupport;
			/*printf("\n***************hArrPointerSupport*************\n");
			for (int i = 0; i < noElemdArrUniEdge; i++)
			{
				printf("\n support:%d ",hArrSupport[i]);
			}*/
		}			
	}


	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() in computeSupportv2() failed",cudaStatus);
		goto Error;
	}
Error:
	
	return cudaStatus;
}
