#include "hip/hip_runtime.h"
#include "kernelCountLabelInGraphDB.h"

__global__ void kernelCountLabelInGraphDB(int *LO,int *result,unsigned int sizeLO,unsigned int sizeResult){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if(i<sizeResult){
		for(int j=0;j<sizeLO;++j){
			if(LO[j]==i) ++result[i];
		}

	}

}