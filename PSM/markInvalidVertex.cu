#include "markInvalidVertex.h"
#include "kernelMarkInvalidVertex.h"

//********Đếm số đỉnh song song và loại nhỏ những đỉnh nhỏ hơn minsup****
	//Nếu số đỉnh nhỏ hơn minSup thì đánh dấu đỉnh đó là -1 trong mảng O và mảng LO và các cạnh liên quan đến đỉnh đó cũng được đánh dấu là -1
	//1. Cấp phát mảng trên bộ nhớ GPU có kích thước =|LV| 
	//cấp phát vùng nhớ trên GPU
inline hipError_t  markInvalidVertex(int *d_O,int *d_LO,int sizeOfarrayO,unsigned int minsup){
	printf("\ncall markInvalidVertex.cu\n");
	hipError_t cudaStatus;
	int grid, block_x=32;
	int n=5;
	size_t nBytesd_labelAmount=n*sizeof(int);
	int *d_labelAmount;

	cudaStatus = hipMalloc((int**) &d_labelAmount,nBytesd_labelAmount);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return cudaStatus;
	}else{	hipMemset(d_labelAmount,0,nBytesd_labelAmount); }

	grid=(nBytesd_labelAmount+block_x-1/block_x);
	
	printf("\nValue of d_labelAmount is set zero all:");
	kernelPrintf<<<grid,block_x>>>(d_labelAmount,n);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus= hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	printf("\nCount label in d_LO and store the result in d_labelAmount:");
	kernelCountLabelInGraphDB<<<grid,block_x>>>(d_LO,d_labelAmount,sizeOfarrayO,n);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus= hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	printf("\n");
	printf("\nValue of d_labelAmount in result:");
	kernelPrintf<<<grid,block_x>>>(d_labelAmount,n);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus= hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	printf("\nValue of d_O:");

	kernelPrintf<<<grid,block_x>>>(d_O,sizeOfarrayO);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus= hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	//Những đỉnh nhỏ hơn minsup sẽ được đánh dấu là -1 trong mảng d_O
	//kernelMarkInvalidVertex(int *d_O,int *LO,unsigned int sizeLO,int *d_labelAmount,unsigned int sizeLabelAmount,unsigned int minsup=2){
	printf("\nProcess to mark vertices that have frequency less than minsup is:",minsup);
	kernelMarkInvalidVertex<<<grid,block_x>>>(d_O,d_LO,sizeOfarrayO,d_labelAmount,n,minsup);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus= hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	printf("\nCheck d_O in result:");
	kernelPrintf<<<grid,block_x>>>(d_O,sizeOfarrayO);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus= hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

Error:
	hipFree(d_labelAmount);
	return cudaStatus;
}




