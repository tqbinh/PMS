#include "hip/hip_runtime.h"
#include "calcLabelAndStoreUniqueExtension.h"

__global__ void kernelCalcLabelAndStoreUniqueExtension(int *d_allPossibleExtension,int *d_allPossibleExtensionScanResult,unsigned int noElem_allPossibleExtension,Extension *d_UniqueExtension,unsigned int Le,unsigned int Lv){
	int i=blockIdx.x*blockDim.x + threadIdx.x;	
	if(i<noElem_allPossibleExtension && d_allPossibleExtension[i]==1){
		int li,lj,lij;
		li=i/(Le*Lv);
		lij=(i%(Le*Lv))/Lv;
		lj=(i%(Le*Lv))-((i%(Le*Lv))/Lv)*Lv;
		//printf("\n[%d]:%d li:%d lij:%d lj:%d",i,d_allPossibleExtensionScanResult[i],li,lij,lj);
		d_UniqueExtension[d_allPossibleExtensionScanResult[i]].li=li;
		d_UniqueExtension[d_allPossibleExtensionScanResult[i]].lij=lij;
		d_UniqueExtension[d_allPossibleExtensionScanResult[i]].lj=lj;
	}
}

hipError_t calcLabelAndStoreUniqueExtension(int *d_allPossibleExtension,int *d_allPossibleExtensionScanResult,unsigned int noElem_allPossibleExtension,Extension *d_UniqueExtension,unsigned int noElem_d_UniqueExtension,unsigned int Le,unsigned int Lv){
	hipError_t cudaStatus;


	dim3 block(1024);
	dim3 grid((noElem_allPossibleExtension+block.x-1)/block.x);

	kernelCalcLabelAndStoreUniqueExtension<<<grid,block>>>(d_allPossibleExtension,d_allPossibleExtensionScanResult,noElem_allPossibleExtension,d_UniqueExtension,Le,Lv);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize failed");
		goto Error;
	}

Error:
	return cudaStatus;
}
