#include "access_d_LO_from_idx_of_d_O.h"
#include "kernelPrintf.h"

inline hipError_t access_d_LO_from_idx_of_d_O(int *d_LO,int *d_N,int sizeOfArrayN){
	printf("\n access_d_LO_from_idx_of_d_N \n");
	
	printf("\nValue of d_N:");
	kernelPrintf<<<sizeOfArrayN+32-1,32>>>(d_N,sizeOfArrayN);
	
	hipError_t cudaStatus;
	dim3 block(32);
	dim3 grid((sizeOfArrayN+block.x-1)/block.x);
	kernelaccess_d_LO_from_idx_of_d_O<<<grid,block>>>(d_LO,d_N,sizeOfArrayN);
	cudaStatus= hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "kernelaccess_d_LO_from_idx_of_d_O launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	//Đồng bộ dữ liệu và đảm bảo không có lỗi xảy ra	
	cudaStatus= hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
Error:
	printf("\n");
	return cudaStatus;

}
