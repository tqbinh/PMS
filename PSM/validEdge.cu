#include "hip/hip_runtime.h"
#include "validEdge.h"
#include "kernelPrintf.h"

__global__ void	kernelValidEdge(Extension *d_Extension,int *V,unsigned int numberElementd_Extension){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i<numberElementd_Extension){	
		if(d_Extension[i].li<=d_Extension[i].lj){
			V[i]=1;
		}
	}
}




hipError_t validEdge(Extension *d_Extension,int *V,unsigned int numberElementd_Extension){
	hipError_t cudaStatus;

	dim3 block(512);
	dim3 grid(numberElementd_Extension+block.x-1/block.x);

	kernelValidEdge<<<grid,block>>>(d_Extension,V,numberElementd_Extension);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize kernelValidEdge failed",cudaStatus);
		goto labelError;
	}
	//
	printf("\nV array: ");
	kernelPrintf<<<grid,block>>>(V,numberElementd_Extension);

labelError:

	return cudaStatus;
}
