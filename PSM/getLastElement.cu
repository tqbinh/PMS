#include "hip/hip_runtime.h"
#include "getLastElement.h"
#include "kernelPrintf.h"



__global__ void kernelGetLastElement(int *Arr,unsigned int noArr, int *value){
	value[0]=Arr[noArr-1];
	//printf("\n Value:%d",value[0]);
}


hipError_t getLastElement(int *d_index,unsigned int numberElementd_index,int &numberElementd_UniqueExtension){
	hipError_t cudaStatus;
	dim3 block(512);
	dim3 grid((numberElementd_index+block.x-1)/block.x);

	int *value;
	hipMalloc((int**)&value,1*sizeof(int));
	
	kernelGetLastElement<<<1,1>>>(d_index,numberElementd_index,value);
	hipDeviceSynchronize();
	cudaStatus= hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize failed",cudaStatus);
		goto Error;
	}
	
	hipMemcpy(&numberElementd_UniqueExtension,value,1*sizeof(int),hipMemcpyDeviceToHost);
	//printf("\n\nnumberElementd_UniqueExtension:%d",numberElementd_UniqueExtension);
	

Error:
	hipFree(value);
	//hipFree(d_index);

	return cudaStatus;	
}

/* Kernel này trả về graphid chứa embedding cuối cùng trong mảng d_ValidExtension */
__global__ void kernelGetLastElementExtension(Extension *inputArray,unsigned int noEleInputArray,int *value,int maxOfVer){
	value[0] = (inputArray[noEleInputArray-1].vgi/maxOfVer); /*Lấy global vertex id chia cho tổng số đỉnh của đồ thị (maxOfVer). Ở đây các đồ thị luôn có số lượng đỉnh bằng nhau (maxOfVer) */
}


/* Hàm này trả về graphId chứa Embedding cuối cùng */
inline hipError_t getLastElementExtension(Extension* inputArray,unsigned int numberElementOfInputArray,int &outputValue,unsigned int maxOfVer){
	hipError_t cudaStatus;

	int *value;
	hipMalloc((int**)&value,sizeof(int));
	/* Lấy graphId chứa embedding cuối cùng */
	kernelGetLastElementExtension<<<1,1>>>(inputArray,numberElementOfInputArray,value,maxOfVer);
	hipDeviceSynchronize();
	cudaStatus= hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize failed",cudaStatus);
		goto Error;
	}
	
	hipMemcpy(&outputValue,value,sizeof(int),hipMemcpyDeviceToHost);
	//printf("\n\nnumberElementd_UniqueExtension:%d",numberElementd_UniqueExtension);
	

Error:
	hipFree(value);
	//hipFree(d_index);

	return cudaStatus;	

}


inline hipError_t getSizeBaseOnScanResult(int *d_allPossibleExtension,int *d_allPossibleExtensionScanResult,unsigned int noElem_allPossibleExtension,int &noElem_d_UniqueExtension){
	hipError_t cudaStatus;

	cudaStatus=getLastElement(d_allPossibleExtensionScanResult,noElem_allPossibleExtension,noElem_d_UniqueExtension);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n getLastElement() of getSizeBaseOnScanResult failed",cudaStatus);
		goto Error;
	}

	printf("\n noElem_d_UniqueExtension inside function:%d",noElem_d_UniqueExtension);

	int valueOfLast=0; //giá trị phần tử cuối cùng của mảng d_allPossibleExtension
	cudaStatus=getLastElement(d_allPossibleExtension,noElem_allPossibleExtension,valueOfLast);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n getLastElement() of getSizeBaseOnScanResult failed",cudaStatus);
		goto Error;
	}

	printf("\nValue of Last Element:%d ",valueOfLast);
	if (valueOfLast==1){
		noElem_d_UniqueExtension=noElem_d_UniqueExtension+1;
	}


	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize() of getSizeBaseOnScanResult failed",cudaStatus);
		goto Error;
	}
Error:
	return cudaStatus;
}
