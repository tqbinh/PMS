#include "hip/hip_runtime.h"
#include "getLastElement.h"
#include "kernelPrintf.h"

//__global__ void kernelGetLastElement(int *Arr,unsigned int noArr,int *value){
//	int i = blockIdx.x*blockDim.x + threadIdx.x;
//	if (i<noArr){
//		if(i==noArr-1){
//			value[0] = Arr[i];			
//		}
//	}
//}

__global__ void kernelGetLastElement2(int *Arr,unsigned int noArr, int *value){
	value[0]=Arr[noArr-1];
	//printf("\n Value:%d",value[0]);
}


hipError_t getLastElement(int *d_index,unsigned int numberElementd_index,int &numberElementd_UniqueExtension){
	hipError_t cudaStatus;
	dim3 block(512);
	dim3 grid((numberElementd_index+block.x-1)/block.x);

	int *value;
	hipMalloc((int**)&value,1*sizeof(int));
	
	//kernelGetLastElement<<<grid,block>>>(d_index,numberElementd_index,value);
	kernelGetLastElement2<<<1,1>>>(d_index,numberElementd_index,value);
	hipDeviceSynchronize();
	cudaStatus= hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize failed",cudaStatus);
		goto Error;
	}
	
	hipMemcpy(&numberElementd_UniqueExtension,value,1*sizeof(int),hipMemcpyDeviceToHost);
	//printf("\n\nnumberElementd_UniqueExtension:%d",numberElementd_UniqueExtension);
	

Error:
	hipFree(value);
	//hipFree(d_index);

	return cudaStatus;	
}

__global__ void kernelGetLastElementExtension(Extension *inputArray,unsigned int noEleInputArray,int *value,int maxOfVer){
	value[0] = (inputArray[noEleInputArray-1].vgi/maxOfVer);
}



inline hipError_t getLastElementExtension(Extension* inputArray,unsigned int numberElementOfInputArray,int &outputValue,unsigned int maxOfVer){
	hipError_t cudaStatus;

	int *value;
	hipMalloc((int**)&value,sizeof(int));
	
	
	kernelGetLastElementExtension<<<1,1>>>(inputArray,numberElementOfInputArray,value,maxOfVer);
	hipDeviceSynchronize();
	cudaStatus= hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize failed",cudaStatus);
		goto Error;
	}
	
	hipMemcpy(&outputValue,value,sizeof(int),hipMemcpyDeviceToHost);
	//printf("\n\nnumberElementd_UniqueExtension:%d",numberElementd_UniqueExtension);
	

Error:
	hipFree(value);
	//hipFree(d_index);

	return cudaStatus;	

}
