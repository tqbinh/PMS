#include "hip/hip_runtime.h"
#include "getLastElement.h"
#include "kernelPrintf.h"

__global__ void kernelGetLastElement(int *Arr,unsigned int noArr,int *value){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<noArr){
		if(i==noArr-1){
			value[0] = Arr[i];			
		}
	}
}

hipError_t getLastElement(int *d_index,unsigned int numberElementd_index,int &numberElementd_UniqueExtension){
	hipError_t cudaStatus;
	dim3 block(512);
	dim3 grid((numberElementd_index+block.x-1)/block.x);

	int *value;
	hipMalloc((int**)&value,1*sizeof(int));
	
	kernelGetLastElement<<<grid,block>>>(d_index,numberElementd_index,value);
	hipDeviceSynchronize();
	cudaStatus= hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"hipDeviceSynchronize failed",cudaStatus);
		goto Error;
	}
	
	hipMemcpy(&numberElementd_UniqueExtension,value,1*sizeof(int),hipMemcpyDeviceToHost);
	printf("\n\nnumberElementd_UniqueExtension:%d",numberElementd_UniqueExtension);
	

Error:
	return cudaStatus;
	hipFree(value);
	hipFree(d_index);
}
