#include "hip/hip_runtime.h"
#pragma once
#include "markEmbedding.cuh"


__global__ void kernelMarkEmbedding(cHistory **dH,struct_Q *device_arr_Q,int lastColumn,int n,unsigned int maxOfVer,int *d_O,int *d_N){
	int i= blockDim.x*blockIdx.x + threadIdx.x; //mỗi thread i sẽ xử lý một embedding
	if(i<n){
		int vid = device_arr_Q[lastColumn]._d_arr_Q[i].vid; // Từ cột Q cuối cùng, mỗi thread i sẽ xử lý embedding thứ i
		int indexOfFirstVertexInGraph = vid-(vid%maxOfVer);
		int toVid = vid;//đỉnh to của cạnh thuộc embedding
		int idxOfdH= (vid%maxOfVer);
		dH[i]->d_arr_HO[idxOfdH]=2;
		int prevQ=device_arr_Q[lastColumn]._prevQ;
		int newi=device_arr_Q[lastColumn]._d_arr_Q[i].idx;
		while (true)
		{			
			//printf("\nd_arr_Q[%d]: (prevQ:%d, idx:%d,vid:%d)",prevQ,device_arr_Q[prevQ]._prevQ,device_arr_Q[prevQ]._d_arr_Q[newi].idx,device_arr_Q[prevQ]._d_arr_Q[newi].vid);		
			
			vid = device_arr_Q[prevQ]._d_arr_Q[newi].vid;
			int fromVid=vid; //đỉnh from của cạnh thuộc embedding
			
			
			int idxEdge = d_O[vid]-d_O[indexOfFirstVertexInGraph]; //vị trí cạnh cần cập nhật được khởi tạo bằng giá trị index của vid đang xét trừ đi giá trị index của đỉnh đầu tiên trong đồ thị đó.
			int indexOfdN=d_O[fromVid];
			
			while (d_N[indexOfdN]!=toVid){
				idxEdge=idxEdge+1;
				indexOfdN++;
			}

			int fromVidR=toVid;
			int toVidR=fromVid;
			int indexOfEdgeR=d_O[fromVidR]-d_O[indexOfFirstVertexInGraph];
			indexOfdN=d_O[fromVidR];
			while(d_N[indexOfdN]!=toVidR){
				indexOfEdgeR++;
				indexOfdN++;
			}

			
			//Nếu không phải là đỉnh đầu tiên thì phải cộng vào idxEdge một lượng bằng tổng bậc của các đỉnh trước đó
			//Tổng bậc của các đỉnh trước đó chính bằng 
						
			idxOfdH = (vid%maxOfVer); //Đánh dấu đỉnh thuộc Embedding
			dH[i]->d_arr_HO[idxOfdH]=2;

			dH[i]->d_arr_HLN[idxEdge]=2;//Đánh dấu cạnh thuộc Embedding. vì đây là đơn đồ thị vô hướng nên cạnh AB cũng bằng cạnh BA,do đó ta phải đánh dấu cạnh BA cũng thuộc embedding.
			dH[i]->d_arr_HLN[indexOfEdgeR]=2;

			if(device_arr_Q[prevQ]._prevQ==-1) return; //nếu là cột Q đầu tiên thì dừng lại vì đã duyệt xong embedding
			newi=device_arr_Q[prevQ]._d_arr_Q[i].idx; //ngược lại thì lấy index của cột Q phía trước
			prevQ=device_arr_Q[prevQ]._prevQ; //Lấy Q phía trước
			toVid=fromVid; //cập nhật lại đỉnh to.
		}
	}

}


hipError_t markEmbedding(cHistory **dH,struct_Q *device_arr_Q,int lastColumn,int n,unsigned int maxOfVer,int *d_O,int *d_N){
	hipError_t cudaStatus;

	dim3 block(1024);
	dim3 grid((n+block.x-1)/block.x);
	/*printf("\****************ndH arr***********"); //kiểm tra thử dữ liệu của mảng dH trên device xem có đúng không
	kernelPrintdeviceH<<<1,1>>>(dH,n);*/
	kernelMarkEmbedding<<<grid,block>>>(dH,device_arr_Q,lastColumn,n,maxOfVer,d_O,d_N);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaDeviceSynchronize markEmbedding failed");
		goto Error;
	}

	//printf("\****************ndH arr***********"); //kiểm tra thử dữ liệu của mảng dH trên device sau khi đã đánh dấu các embedding thuộc right most path
	//kernelPrintdeviceH<<<1,1>>>(dH,n);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaDeviceSynchronize markEmbedding failed");
		goto Error;
	}
Error:

	return cudaStatus;
}
