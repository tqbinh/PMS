#include "hip/hip_runtime.h"
#include "castingIntToFloat.h"
#include <math_functions.h>
#include "hip/device_functions.h"
#include <math.h>
#include <device_types.h>
#include <device_functions_decls.h>
#include <>
#include <deviceaccess.h>
#include <math_functions.h>


__global__ void kernelCastingUnsignedInt2Float(float* d_out,unsigned int* d_in,unsigned int n){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if(i<n){
		d_out[i]= __uint2float_rn(d_in[i]);	
	}
}

__global__ void kernelCastingInt2Float(float* d_out,int* d_in,unsigned int n){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if(i<n){
		d_out[i]= __int2float_rd(d_in[i]);
	}

}

__global__ void kernelCastingFloat2Int(int* d_out,float* d_in,unsigned int n){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if(i<n){
		d_out[i]=__float2int_rd(d_in[i]);
	}
}


__global__ void kernelCastingFloat2UnsignedInt(unsigned int* d_out,float* d_in,unsigned int n){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if(i<n){
		d_out[i]=__float2uint_rd(d_in[i]);
	}
}
