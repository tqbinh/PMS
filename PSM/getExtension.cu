#include "hip/hip_runtime.h"
#include "getExtension.h"


__global__ void kernelUpdateData(ArrHistory *d_arrH,cHistory *h_d_Hi,int i){
	d_arrH->vecA[i]=h_d_Hi;
}


__global__ void kernelPrintd_d_obj(cHistory **d_d_obj,int n,int *d_arr_number_HLN){
	for (int i = 0; i < n; i++)
	{
		printf("\nd_d_obj[%d] n:%d m:%d d_arr_HO:%p",i,d_d_obj[i]->n,d_d_obj[i]->m,d_d_obj[i]->d_arr_HO);
		/*for (int j = 0; j < d_arr_number_HLN[i]; j++)
		{
			printf("\nd_HLN:%d",d_d_obj[i]->d_arr_HO[j]);
		}*/

	}
}


__global__ void kernelPrintd_h(cHistory **d_h,int noEle){
	for (int i = 0; i < noEle; i++)
	{
		d_h[0]->print();
	}

}

__global__ void kernelPrintArrHistory(ArrHistory *d_arrH,int n){	
	printf("\n***inside kernelPrintArrHistory" );
	printf("\nvalue of n:%d",d_arrH->n);
	printf("\npointer vecA:%p",d_arrH->vecA);
	printf("\nvalue n of vecA:%d",d_arrH->vecA[0]->n);
		//d_arrH->print();
	//printf("\nd_arrH:%p",d_arrH);
		//for (int i = 0; i < n; i++)
		//{
		//	//printf("\n vecA[%d].m:%p",i,d_arrH->vecA[i]->m);
		//	//printf("\n vec[%d]:%p",i,d_arrH->vecA[i]->d_arr_HO);
		//	//d_arrH->vecA[i]->print();
		//}
}


void cHistory::print(){
	for (int i = 0; i < n; i++)
	{
		
		printf("\nd_arr_HO[%d]:%d - %p",i,d_arr_HO[i],&d_arr_HO[i]);
	}
	for (int i = 0; i < m; i++)
	{
		printf("\nd_arr_HLN[%d]:%d - %p",i,d_arr_HLN[i],&d_arr_HLN[i]);
	}
}

void cHistory::printmn(){
	printf("\n n:%d m:%d",n,m);
}


cHistory::cHistory(){
	n=0;
	m=0;
	d_arr_HO=NULL;
	d_arr_HLN=NULL;
}


cHistory::cHistory(int _n,int _m){
	n=_n;
	m=_m;

	d_arr_HO = (int*)malloc(sizeof(int)*n);
	if(d_arr_HO==NULL){
		printf("\nMalloc d_arr_HO failed");
		exit(1);
	}
	else
	{
		memset(d_arr_HO,0,sizeof(int)*n);
	}

	d_arr_HLN=(int*)malloc(sizeof(int)*m);
	if(d_arr_HLN==NULL){
		printf("\nMalloc d_arr_HLN failed");
		exit(1);
	}
	else
	{
		memset(d_arr_HLN,0,sizeof(int)*m);
	}
}

ArrHistory::ArrHistory(){
	n=0;
	vecA=NULL;
}

ArrHistory::ArrHistory(int _n){
	
	n=_n; //số lượng phần tử của vecA
	vecA=NULL;
	vecA=new cHistory*[n]; //dynamic array (size = n) of pointer to an object cHistory
	if(vecA==NULL){
		printf("\nMalloc failed");
		exit(1);
	}	
}

void ArrHistory::print()	
{
	printf("\n Number of element of vecA is:%d \n Address of vecA:%p \n Below is the pointer to array of object:",n,vecA);
	for (int i = 0; i < n; i++)
	{
		printf("\n Value of vecA[%d]:%p",i,vecA[i]); //vecA[i] lưu trữ địa chỉ của một đối tượng cHistory. hay nói cách khác là nó trỏ đến đối tượng cHistory
		vecA[i]->print();
	}
}

__global__ void kernelFindNumberOfEdgeInAGraph(int *d_arr_number_HLN,struct_Q *device_arr_Q,int numberEmbedding,int lastColumn,unsigned int maxOfVer,int *d_O,unsigned int numberOfGraph,unsigned int noDeg){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<numberEmbedding){
		//printf("\n vid:%d",device_arr_Q[lastColumn]._d_arr_Q[i].vid);
		int graphId=(device_arr_Q[lastColumn]._d_arr_Q[i].vid)/maxOfVer;
		//printf("\n vid:%d, graphid:%d",device_arr_Q[lastColumn]._d_arr_Q[i].vid,graphId);
		int idxFrom = graphId*maxOfVer;		
		int idxFirstNext = (graphId+1)*maxOfVer;
		int r=0;
		//printf("\n i:%d, vid:%d, r:%d",i,device_arr_Q[lastColumn]._d_arr_Q[i].vid,r);
		if (graphId!=(numberOfGraph-1)){
			//printf("\nidxFirstNext:%d",idxFirstNext);
			r=d_O[idxFirstNext]-d_O[idxFrom];
		}else
		{
			r=noDeg-d_O[idxFrom];
		}
		//printf("\n i:%d, vid:%d, r:%d",i,device_arr_Q[lastColumn]._d_arr_Q[i].vid,r);
		d_arr_number_HLN[i]=r;
	}
}


hipError_t	findNumberOfEdgeInAGraph(int *d_arr_number_HLN,struct_Q *device_arr_Q,int numberEmbedding,int lastColumn,unsigned int maxOfVer,int *d_O,unsigned int numberOfGraph,unsigned int noDeg){
	hipError_t cudaStatus;

	dim3 block(1024);
	dim3 grid((numberEmbedding+block.x-1)/block.x);
	printf("\nCall kernelFindNumberOfEdgeInAGraph");
	kernelFindNumberOfEdgeInAGraph<<<grid,block>>>(d_arr_number_HLN,device_arr_Q,numberEmbedding,lastColumn,maxOfVer,d_O,numberOfGraph,noDeg);

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if(cudaStatus!=hipSuccess)
	{
		fprintf(stderr,"\ncudaDeviceSynchronize findNumberOfEdgeInAGraph failed");
		goto Error;
	}
Error:
	return cudaStatus;
}


__global__ void	kernelGetnoEle_Embedding(struct_Q *device_arr_Q,int lastColumn,int *noEle_Embeddings){
	noEle_Embeddings[0]=device_arr_Q[lastColumn]._size;
	printf("\nInside kernelGetnoEle_Embedding:%d",noEle_Embeddings[0]);
}

__global__ void kernelPrintdeviceH(cHistory **device_H,int numberEmbeddings){
	for (int i = 0; i < numberEmbeddings; i++)
	{
		printf("\n**************dH[%d]:%p**********",i,&device_H[i]);		
		////device_H[i]->print();
		//printf("\n n:%d",device_H[i]->n);
		//printf("\n m:%d",device_H[i]->m);
		//for (int j = 0; j < device_H[i]->n; j++)
		//{
		//	printf("\nHO[%d]:%d",j,device_H[i]->d_arr_HO[j]);
		//}
		//for (int j = 0; j < device_H[i]->m; j++)
		//{
		//	printf("\nHLN[%d]:%d",j,device_H[i]->d_arr_HLN[j]);
		//}
		device_H[i]->printmn();
		device_H[i]->print();
	}
}




hipError_t getExtension(struct_Q *device_arr_Q,int lastColumn,vector<struct_DFS> &P,vector<int> &RMPath,int *d_O,int *d_LO,int numberOfElementd_O,int *d_N,int *d_LN,int numberOfElementd_N,unsigned int Lv,unsigned int Le,unsigned int minsup,unsigned int maxOfVer,unsigned int numberOfGraph,unsigned int noDeg){
	hipError_t cudaStatus;

	/*
		Dữ liệu truyền vào gồm: CSDL(d_O,d_LO,d_N,d_LN), ngưỡng minsup, pattern P và các Embeddings của P(device_arr_Q), Right Most Path (vector<int> RMPath)
		Làm sao để tìm tất cả các mở rộng hợp lệ từ tất cả các đỉnh thuộc RMPath của Embedding 
		Chúng ta thực hiện tìm mở rộng lần lượt từ Qk đến Q0 (Chỉ xét các Q thuộc RMPath của P).
		B1. Tìm bậc lớn nhất của tất cả các vid của Q đang xét (GPU step). Kết quả lưu vào biến maxDegree_vid_Q
		B2. Tạo mảng d_arr_V có kích thước: maxDegree_vid_Q * |Q|
			Lưu ý, mảng d_arr_V phải có dạng cấu trúc đủ thể hiện cạnh mở rộng có hợp lệ hay không và là forward extension hay backward extension.
			struct struct_V
			{
				int valid; //default: 0, valid: 1
				int BK; //default: 0- forward; backward: 1
			}
		B3. Dựa vào CSDL để tìm những mở rộng hợp lệ, thông tin backward và forward được ghi nhận vào d_arr_V
		
		- Valid Forward: cạnh mở rộng luôn phải lớn hơn hoặc bằng cạnh đầu tiên của DFS_CODE
			+ Lớn hơn nếu nó có đỉnh from lớn hơn
			+ Hoặc nhãn cạnh lớn hơn
			+ hoặc nhãn đỉnh "to" lớn hơn
			+ và đỉnh "to" không thuộc embedding
		- Valid Backward: cạnh mở rộng luôn phải lớn hơn hoặc bằng cạnh kết nối với đỉnh "to" của mở rộng.
			+ Đỉnh "to" của mở rộng phải thuộc right most path
			+ Cạnh mở rộng không thuộc embedding và phải có nhãn cạnh lớn hơn hoặc bằng hoặc phải có một đỉnh lớn hơn hoặc bằng cạnh lớn nhất kề với đỉnh "to" của
			mở rộng.
		
		Nhưng vấn đề là làm sao kiểm tra mở rộng đó có thuộc embedding hay không?
		- Cần phải xây dựng một embedding và ánh xạ nó với CSDL hiện có để ghi nhận lại những cạnh và đỉnh đã thuộc embedding và right most path
		- Hoạt động này có thể được thực hiện một cách song song hay không?
		1. Chúng ta biết được số lượng embedding, suy ra chúng ta có thể biết được cần phải xây dựng bao nhiêu ánh xạ (history mapping).
			- Khởi tạo bộ nhớ d_arr_History, mỗi phần tử của d_arr_History là một đồ thị
		Mọi hoạt động cấp phát bộ nhớ đều phải được thực hiện ở host. Do đó, cần phải biết kích thước cần cấp phát bộ nhớ
			+ Duyệt qua embedding
				o Dựa vào vid để biết được embedding thuộc đồ thị nào

	*/

	//1. Lấy số lượng embedding từ device_arr_Q và lưu kết quả vào biến noEle_Embeddings
	//	 Lấy kích thước mảng dO, dLN và lưu vào mảng
	int *noEle_Embeddings=NULL;
	int *noEle_hEmbeddings=(int*)new int[1];
	hipMalloc((void**)&noEle_Embeddings,sizeof(int));
	
	kernelGetnoEle_Embedding<<<1,1>>>(device_arr_Q,lastColumn,noEle_Embeddings);
	
	hipMemcpy(noEle_hEmbeddings,noEle_Embeddings,sizeof(int),hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaDeviceSynchronize() kernelGetnoEle_Embedding failed");
		goto Error;
	} 

	printf("\nnoEle_Embeddings:%d",*noEle_hEmbeddings);
	
	//Tạo mảng số nguyên có kích thước bằng số lượng embedding
	//Mangr d_arr_number_HLN lưu trữ số lượng phần tử của mảng d_HLN trong object cHistory của embedding tương ứng.

	int * d_arr_number_HLN;
	cudaStatus=hipMalloc((void**)&d_arr_number_HLN,sizeof(int)*noEle_hEmbeddings[0]);
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,":\ncudaMalloc d_arr_number_HLN failed");
		goto Error;
	}
	else
	{
		hipMemset(d_arr_number_HLN,0,sizeof(int)*noEle_hEmbeddings[0]);
	}

	findNumberOfEdgeInAGraph(d_arr_number_HLN,device_arr_Q,noEle_hEmbeddings[0],lastColumn,maxOfVer,d_O,numberOfGraph,noDeg);

	//Chép kết quả qua host
	int *h_arr_number_HLN;
	h_arr_number_HLN=(int*)malloc (sizeof(int)*noEle_hEmbeddings[0]);
	hipMemcpy(h_arr_number_HLN,d_arr_number_HLN,(sizeof(int)*noEle_hEmbeddings[0]),hipMemcpyDeviceToHost);

	for (int i = 0; i < noEle_hEmbeddings[0]; i++)
	{
		printf("\nh_arr_number_HLN[%d]:%d",i,h_arr_number_HLN[i]);
	}


	//2. Tạo một con trỏ đối tượng h_arrH, member của nó là h_arrH->vecA một mảng các pointer trỏ đến các đối tượng cHistory (cHistory **h_arrH->vecA)
	ArrHistory h_arrH(noEle_hEmbeddings[0]); //6 là số lượng embedding, giá trị tham số truyền vào sẽ được sử dụng để tạo ra 6 phần tử vecA có kiểu (cHistory**)
	//h_arrH.print();

	////Đứng đây thì làm sao khởi tạo cho các đối tượng bên trong (khởi tạo cho các con trỏ trỏ đến mảng các đối tượng) khi đã biết kích thước m và n của object
	for (int i = 0; i < noEle_hEmbeddings[0]; i++) 
	{
		//printf("\n************ %d ************",i);
		h_arrH.vecA[i]= (cHistory*) new cHistory(maxOfVer,h_arr_number_HLN[i]);
		//h_arrH.vecA[i]->print();
	}
	//h_arrH.print();

	//3. Bắt đầu quá trình sao chép mảng cHistory sang device
	int n=noEle_hEmbeddings[0]; 
	int numberElement_darrHO=maxOfVer;
	//int numberElement_darrHLN[]={2,4}; // chính là mảng h_arr_number_HLN chỉ số lượng phần tử của mảng d_arr_HLN trong đối tượng cHistory
	
	//cHistory **h = (cHistory**)malloc(sizeof(cHistory)*n); // chính là mảng h_arrH
	//for (int i = 0; i < n; i++)
	//{
	//	h[i] = new cHistory(numberElement_darrHO,numberElement_darrHLN[i]);
	//}

	cHistory *h1=new cHistory[n]; //Do các embedding lưu trong h_arrH.vecA[i] là không liên tục nhau trên bộ nhớ, Do đó tạo h1 với các bộ nhớ liên tục và chép dữ liệu của h_arrH.vecA[i] sang h1
	for (int i = 0; i < n; i++)
	{
		h1[i].n=h_arrH.vecA[i]->n;
		h1[i].m=h_arrH.vecA[i]->m;
		h1[i].d_arr_HO = (int*) malloc(sizeof(int)*numberElement_darrHO);
		for (int j = 0; j < numberElement_darrHO; j++)
		{
			h1[i].d_arr_HO[j]=h_arrH.vecA[i]->d_arr_HO[j];
		}
		h1[i].d_arr_HLN=(int*)malloc(sizeof(int)*h_arr_number_HLN[i]);
		for (int j = 0; j < h_arr_number_HLN[i]; j++)
		{
			h1[i].d_arr_HLN[j]=h_arrH.vecA[i]->d_arr_HLN[j];
		}
	}

	for (int i = 0; i < n; i++)
	{
		printf("\n********%d***********",i);
		h1[i].printmn();
	}


	cHistory **dH; //dH dùng để lưu kết quả cuối cùng của mảng cHistory
	hipMalloc((void**)&dH,sizeof(cHistory*)*n);

	//Do không thể cấp phát bộ nhớ cho các member của dH một các trực tiếp trên device nên chúng ta sẽ cấp phát thông qua một biến khác device_H
	cHistory **device_H=(cHistory**)malloc(sizeof(cHistory*)*n);

	for (int j = 0; j < n; j++)
	{
		cHistory h2(numberElement_darrHO,h_arr_number_HLN[j]);
		h2.n=h1[j].n;
		h2.m=h1[j].m;
		for (int i = 0; i < h1[j].n; i++)
		{
			h2.d_arr_HO[i]=h1[j].d_arr_HO[i];
		}
		for (int i = 0; i < h1[j].m; i++)
		{
			h2.d_arr_HLN[i]=h1[j].d_arr_HLN[i];
		}
		//h2.print();	
		//Bây giờ làm sao chép đối tượng này sang bộ nhớ device?
		//Tạo một con trỏ đối tượng 
		
		hipMalloc((void**)&device_H[j],sizeof(cHistory));
		hipMemcpy(device_H[j],&h2,sizeof(cHistory),hipMemcpyHostToDevice); //copy h bỏ vào d_h

		int *temp_dO,*temp_dHLN;	//khởi tạo bộ nhớ tạm trên device, gán dữ liệu cho bộ nhớ tạm này. Sau đó, gán chép bộ nhớ này cho các pointer bên trong.
		hipMalloc((void**)&temp_dO,sizeof(int)*numberElement_darrHO);
		hipMalloc((void**)&temp_dHLN,sizeof(int)*h_arr_number_HLN[j]);


		hipMemcpy(temp_dO,h2.d_arr_HO,sizeof(int)*numberElement_darrHO,hipMemcpyHostToDevice);
		hipMemcpy(temp_dHLN,h2.d_arr_HLN,sizeof(int)*h_arr_number_HLN[j],hipMemcpyHostToDevice);


		hipMemcpy(&(device_H[j]->d_arr_HO),&(temp_dO),sizeof(int*),hipMemcpyHostToDevice);
		hipMemcpy(&(device_H[j]->d_arr_HLN),&(temp_dHLN),sizeof(int*),hipMemcpyHostToDevice);

		/*printf("\naddress of d_h:%p",&device_H[j]);
		printf("\n**********j=%d***********",j);
		kernelPrintd_h<<<1,1>>>(device_H[j]);
		hipDeviceSynchronize();
		if(hipGetLastError()!=hipSuccess){
			fprintf(stderr,"\ncudaDeviceSynchronize kernelPrintd_h has been failed");
			goto Error;
		}*/

	}

	hipMemcpy(dH,device_H,sizeof(cHistory*)*n,hipMemcpyHostToDevice);

	//kernelPrintdeviceH<<<1,1>>>(dH,n);

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaDeviceSynchronize kernelPrintdeviceH d has been failed");
		goto Error;
	}

	//Cách chép 1 đối tượng cHistory sang device
		/* 
	//Rắc rối quá. Bây giờ mình chỉ làm cho 1 đối tượng cHistory
	cHistory h(6,12); //n=6 và m=12 đây là 2 giá trị tham số mà chúng ta cần phải trích ra từ embedding device_arr_Q. Sẽ được viết hàm sau.
	
	//h.print();	
	//Bây giờ làm sao chép đối tượng này sang bộ nhớ device?
	//Tạo một con trỏ đối tượng 
	cHistory *d_h;
	hipMalloc((void**)&d_h,sizeof(cHistory));
	hipMemcpy(d_h,&h,sizeof(cHistory),hipMemcpyHostToDevice); //copy h bỏ vào d_h

	int *temp_dO,*temp_dHLN;	//khởi tạo bộ nhớ tạm trên device, gán dữ liệu cho bộ nhớ tạm này. Sau đó, gán chép bộ nhớ này cho các pointer bên trong.
	hipMalloc((void**)&temp_dO,sizeof(int)*6);
	hipMalloc((void**)&temp_dHLN,sizeof(int)*12);

	hipMemcpy(temp_dO,h.d_arr_HO,sizeof(int)*6,hipMemcpyHostToDevice);
	hipMemcpy(temp_dHLN,h.d_arr_HLN,sizeof(int)*12,hipMemcpyHostToDevice);

	// hipMemcpy(&(d_c->data), &hostdata, sizeof(int *), hipMemcpyHostToDevice);

	hipMemcpy(&(d_h->d_arr_HO),&(temp_dO),sizeof(int*),hipMemcpyHostToDevice);
	hipMemcpy(&(d_h->d_arr_HLN),&(temp_dHLN),sizeof(int*),hipMemcpyHostToDevice);

	printf("\nsizeof(h):%d",sizeof(h));
	printf("\nsizeof(d_h):%d",sizeof(d_h));


	kernelPrintd_h<<<1,1>>>(d_h);
	*/

/*
//4. Đã có dH
	Bây giờ chúng ta duyệt qua các embedding và đánh dấu những đỉnh vào cạnh thuộc Embedding là 1
	Những đỉnh và cạnh nào thuộc Embedding thì đánh dấu là 2.
	Input: cHistory **dH, structQ *device_arr_Q, int lastColumn,vector<int> RMPath
*/	


cudaStatus = markEmbedding(dH,device_arr_Q,lastColumn,RMPath,n,maxOfVer,d_O,d_N);
	

	

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaDeviceSynchronize() has been failed");
		goto Error;
	}
Error:

	return cudaStatus;
}
