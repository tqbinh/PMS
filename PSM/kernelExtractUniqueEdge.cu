#include "hip/hip_runtime.h"
#include "kernelExtractUniqueEdge.h"

__global__ void kernelExtractUniqueEdge(int *d_O,int *d_LO,unsigned int numberElementOfd_O,int *d_N,int *d_LN,unsigned int numberElementOfd_N,int *d_singlePattern,unsigned int Lv,unsigned int Le){

	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i<numberElementOfd_O){
		/*printf("\nThread:%d",i);	*/
		if(d_O[i]!=-1){ 
			int j;
			//printf("\nThread:%d",i);	
			for(j=i+1;j<numberElementOfd_O;++j){					
				if(d_O[j]!=-1) {break;}
				
			}			
			int ek;
			if (j==numberElementOfd_O) {
				ek=numberElementOfd_N;

			}
			else
			{
				ek=d_O[j];
			}

			
			int Li=d_LO[i];				
			int startIndex=((Lv+(Lv-(Li-1)))*(Lv-(Lv-(Li-1))+1)/2)*Le;				
			for (int k=d_O[i];k<ek;++k){
				int Lj, Lij;					
				Lij=d_LN[k];					
				Lj=d_LO[d_N[k]]; 					
				if(Lj<Li) continue;
				startIndex=startIndex+Lij*(Lv-Li) + (Lj-Li);
				d_singlePattern[startIndex]=1;
				//printf("\nThread:%d Li:%d Lj:%d Le:%d  index:%d d_signlePattern:%d\n",i,Li,Lj,Le,startIndex,d_singlePattern[startIndex]);
				startIndex=startIndex-(Lj-Li);
				//printf("index:%d [%d] ",index,d_singlePattern[index]);
			}

		}
	}
}
