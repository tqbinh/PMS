#include "hip/hip_runtime.h"
#include "getSatisfyEdge.h"

__global__ void kernelGetSatisfyEdge(UniEdge *d_UniqueExtension,int indexOfSatisfyEdge,int *d_li,int *d_lij,int *d_lj,int *d_arr_labelEdge){

	d_li[0]=d_UniqueExtension[indexOfSatisfyEdge].li;
	d_lij[0]=d_UniqueExtension[indexOfSatisfyEdge].lij;
	d_lj[0]=d_UniqueExtension[indexOfSatisfyEdge].lj;

	d_arr_labelEdge[0]=d_UniqueExtension[indexOfSatisfyEdge].li;
	d_arr_labelEdge[1]=d_UniqueExtension[indexOfSatisfyEdge].lij;
	d_arr_labelEdge[2]=d_UniqueExtension[indexOfSatisfyEdge].lj;

	printf("\n d_arr_labelEdge[0]:%d", d_arr_labelEdge[0]);
	printf("\n d_arr_labelEdge[1]:%d", d_arr_labelEdge[1]);
	printf("\n d_arr_labelEdge[2]:%d", d_arr_labelEdge[2]);
}



inline hipError_t getSatisfyEdge(UniEdge *d_UniqueExtension,int number,int indexOfSatisfyEdge,int &li,int &lij,int &lj,int* &d_arr_edgeLabel){
	hipError_t cudaStatus;

	int *d_li=nullptr;
	int *d_lij=nullptr;
	int *d_lj=nullptr;

	hipMalloc((void**)&d_arr_edgeLabel,sizeof(int)*3);
	hipMemset(d_arr_edgeLabel,0,sizeof(int)*3);

	cudaStatus = hipMalloc((void**)&d_li,sizeof(int));
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"\n hipMalloc d_li failed");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_lij,sizeof(int));
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"\n hipMalloc d_lij failed");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_lj,sizeof(int));
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"\n hipMalloc d_lj failed");
		goto Error;
	}
	

	kernelGetSatisfyEdge<<<1,1>>>(d_UniqueExtension,indexOfSatisfyEdge,d_li,d_lij,d_lj,d_arr_edgeLabel);
	hipDeviceSynchronize();

	hipMemcpy(&li,d_li,sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(&lij,d_lij,sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(&lj,d_lj,sizeof(int),hipMemcpyDeviceToHost);



	hipDeviceSynchronize();
	cudaStatus= hipGetLastError();
	if(cudaStatus != hipSuccess){
		fprintf(stderr,"\n hipDeviceSynchronize getSatisfyEdge failed");
		goto Error;
	}
Error:
	return cudaStatus;
}
