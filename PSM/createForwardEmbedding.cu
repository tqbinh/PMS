#include "hip/hip_runtime.h"
#include "createForwardEmbedding.h"

//__global__ void kernelPrintdArrayQ(struct_QQ Q){
//	printf("\nInside kernelPrintArrayQ:");
//	struct_Q temp;
//	for (int i = 0; i < Q.size; i++)
//	{		
//		temp=Q.structQ[i];
//		printf("\ntemp.size:%d",temp._size);
//		printf("\ntemp._prevQ:%d",temp._prevQ);
//		printf("\ntemp._d_arr_Q:%p",temp._d_arr_Q);
//		printf("\ntemp._d_arr_Q:%p",temp._d_arr_Q);
//		struct_Embedding **temp2=temp._d_arr_Q;
//		printf("\ntemp2[0][0]: (idx:%p",temp2);
//		//for (int j = 0; j < 4; j++)
//		//{
//		//	printf("\ntemp2[0][%d]: (idx:%d",j,temp2);
//		//}
//	}
//}

//struct_QQ convertToStruct(thrust::device_vector<struct_Q> &dArray){
//	struct_QQ Q;
//	Q.structQ=thrust::raw_pointer_cast(&dArray[0]);
//	Q.size=(int)dArray.size();
//	return Q;
//}

//convert thrust device vector to struct_Q
//struct_Q convertDeviceVectorToStruct(thrust::device_vector<struct_Embedding*>&dVecQ,struct_Embedding *d_Q1,int noElemOfd_Q1,int prevQ){
//	struct_Q Q;
//	//Q._d_arr_Q = thrust::raw_pointer_cast(&dVecQ[0]);
//	Q._d_arr_Q = &d_Q1;
//	Q._prevQ=prevQ;
//	Q._size=noElemOfd_Q1;
//	printf("\nInside converDeviceVectorToStruct function:");
//	printf("\nQ._d_arr_Q:%p",*Q._d_arr_Q);
//	return Q;
//}


//__global__ void kernelVector(struct_Q Q,struct_Embedding *d_Q1,int noElem_d_Q){
//	Q._d_arr_Q=&(d_Q1);
//	Q._prevQ=d_Q1->prevQ;
//	Q._size=noElem_d_Q;
//	printf("\nInside kernelVector:");
//	printf("\nd_Q1:%p",d_Q1);
//	printf("\nQ._d_arr_Q:%p",*(Q._d_arr_Q));
//	printf("\nElement of array d_Q:");
//	//printf("\nQ._d_arr_Q[0] value :(idx:%d, vid:%d)",(*(Q._d_arr_Q))->idx,(*(Q._d_arr_Q))->vid);
//	//printf("\nQ._d_arr_Q[1] value:(idx:%d, vid:%d)",((*(Q._d_arr_Q))+1)->idx,((*(Q._d_arr_Q))+1)->vid);
//	//printf("\nQ._d_arr_Q[2] value:(idx:%d, vid:%d)",((*(Q._d_arr_Q))+2)->idx,((*(Q._d_arr_Q))+2)->vid);
//	//printf("\nQ._d_arr_Q[3] value:(idx:%d, vid:%d)",((*(Q._d_arr_Q))+3)->idx,((*(Q._d_arr_Q))+3)->vid);
//	for (int i = 0; i < noElem_d_Q; i++)
//	{
//		printf("\nQ._d_arr_Q[%d] value:(idx:%d, vid:%d)",i,((*(Q._d_arr_Q))+i)->idx,((*(Q._d_arr_Q))+i)->vid);		
//	}
//}


//__global__ void kernelPrintVectorQ(thrust::device_vector<struct_Embedding**> vecQ,int sizeVecQ,int noElem_d_Q){
//
//	int i=blockDim.x*blockIdx.x + threadIdx.x;
//	if (i<sizeVecQ){
//	printf("\nArray is:%p",thrust::raw_pointer_cast(&*vecQ[i]));
//		//cout<<vecQ[0][0]->idx;
//	}
//
//	//int t = blockDim.x*blockIdx.x + threadIdx.x;
//	//if (t<noElem_d_Q){
//	//	for (int j = 0; j < vecQ.size(); j++)
//	//	{
//	//		for (int k = 0; k < noElem_d_Q; k++)
//	//		{
//	//			printf("\nvecQ[%d][%d]: (idx:%d",j,k,vecQ[j][k]->idx);
//	//		}
//	//	}
//
//	//}
//
//}



__global__ void kernelPrintEmbeddingFromLastQ(struct_Embedding **d_arr_Q,int position,int noElem_LastQ){
	int i= threadIdx.x + blockIdx.x * blockDim.x;
	if(i<noElem_LastQ){
		printf("\nd_arr_Q[%d][%d]: (prevQ:%d, idx:%d,vid:%d)",position,i,d_arr_Q[position][i].prevQ,d_arr_Q[position][i].idx,d_arr_Q[position][i].vid);
		//int newi=d_arr_Q[position][i].idx;
		//printf("\nd_arr_Q[%d][%d]: (prevQ:%d, idx:%d,vid:%d)",position,newi,d_arr_Q[position][newi].prevQ,d_arr_Q[position][newi].idx,d_arr_Q[position][newi].vid);		
		int prevQ=d_arr_Q[position][i].prevQ;
		int newi=d_arr_Q[position][i].idx;
		while (true)
		{
			
			printf("\nd_arr_Q[%d][%d]: (prevQ:%d, idx:%d,vid:%d)",prevQ,newi,d_arr_Q[prevQ][newi].prevQ,d_arr_Q[prevQ][newi].idx,d_arr_Q[prevQ][newi].vid);		
			
			if(d_arr_Q[prevQ][newi].prevQ==-1) return;
			newi=d_arr_Q[prevQ][newi].idx;
			prevQ=d_arr_Q[prevQ][newi].prevQ;
		}
	}

}


__global__ void kernelCopy(struct_Embedding *d_Q1,struct_Embedding *d_Q2,struct_Embedding **d_arr_Q){
	d_arr_Q[0]=d_Q1;
	d_arr_Q[1]=d_Q2;
	printf("\n\nInside KernelCopy:");
	printf("\nd_Q1:%p",d_Q1);
	printf("\nd_Q2:%p",d_Q2);
	printf("\nd_arr_Q[0]:%p",d_arr_Q[0]);
	printf("\nd_arr_Q[1]:%p",d_arr_Q[1]);

}



__global__ void kernelPrintArrayEmbedding(struct_Embedding **d_arr_Q,int n,int noElem_Embedding){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if(i<n){
		printf("\nd_arr_Q[%d]:%p",i,d_arr_Q[i]);
		for(int j=0;j<noElem_Embedding;j++){
			printf("\n[%d][%d]: (prevQ:%d, idx:%d,vid:%d)",i,j,d_arr_Q[i][j].prevQ,d_arr_Q[i][j].idx,d_arr_Q[i][j].vid);
		}
	}

}


__global__ void kernelMarkExtension(Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,int *d_M,int li,int lij,int lj){
	int i= blockIdx.x*blockDim.x + threadIdx.x;
	if(i<noElem_d_ValidExtension){
		if(d_ValidExtension[i].li==li && d_ValidExtension[i].lij==lij && d_ValidExtension[i].lj==lj){
			d_M[i]=1;
		}		
	}
}


__global__ void kernelMatchLastElement(Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,int li,int lij,int lj,bool same){
	int lastIndex=noElem_d_ValidExtension-1;
	if(d_ValidExtension[lastIndex].li==li && d_ValidExtension[lastIndex].lij==lij && d_ValidExtension[lastIndex].lj==lj){
		same=true;
	}

}

__global__ void kernelCreateForwardEmbedding(Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,int *d_scanResult,int li,int lij,int lj,struct_Embedding *d_Q1,struct_Embedding *d_Q2){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i<noElem_d_ValidExtension){
		if(d_ValidExtension[i].li==li && d_ValidExtension[i].lij==lij && d_ValidExtension[i].lj==lj){
			d_Q1[d_scanResult[i]].prevQ=-1;
			d_Q1[d_scanResult[i]].vid=d_ValidExtension[i].vgi;

			d_Q2[d_scanResult[i]].prevQ=0;
			d_Q2[d_scanResult[i]].idx=d_scanResult[i];
			d_Q2[d_scanResult[i]].vid=d_ValidExtension[i].vgj;
		}
	}

}

__global__ void kernelPrintd_array_Q(struct_Embedding** d_arr_Q){
	printf("\nd_arr_Q:%p",d_arr_Q);
	printf("\nd_arr_Q[0]:%p",d_arr_Q[0]);
	printf("\nd_arr_Q[1]:%p",d_arr_Q[1]);
}

__global__ void kernelcp(struct_Q *device_arr_Q,int noElem_device_arr_Q,int positionUpdate,struct_Embedding *d_Q,int noElem_d_Q,int prevQ){
	if(positionUpdate<noElem_device_arr_Q && positionUpdate>=0 ){
	device_arr_Q[positionUpdate]._size=noElem_d_Q;
	device_arr_Q[positionUpdate]._prevQ=prevQ;
	device_arr_Q[positionUpdate]._d_arr_Q=d_Q;
	}
}

__global__ void printStructQ(struct_Q *device_arr_Q,int noElem){
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i<noElem){
		printf("\ndevice_arr_Q[%d]._size:%d",i,device_arr_Q[i]._size);
		printf("\ndevice_arr_Q[%d]._prevQ:%d",i,device_arr_Q[i]._prevQ);
		printf("\ndevice_arr_Q[%d]._d_arr_Q:%p",i,device_arr_Q[i]._d_arr_Q);
		for (int j = 0; j < device_arr_Q[i]._size; j++)
		{
			printf("\n(idx:%d, vid:%d)",(device_arr_Q[i]._d_arr_Q)[j].idx,(device_arr_Q[i]._d_arr_Q)[j].vid);
		}
	}
}


__global__ void PrintAllEmbedding(struct_Q *device_arr_Q,int position,int noElemOfLastColumn){
	int i= threadIdx.x + blockIdx.x * blockDim.x;
	if(i<device_arr_Q[position]._size && position!=0){
		printf("\ndevice_arr_Q[%d]: (prevQ:%d, idx:%d,vid:%d)",position,device_arr_Q[position]._prevQ,device_arr_Q[position]._d_arr_Q[i].idx,device_arr_Q[position]._d_arr_Q[i].vid);
		int prevQ=device_arr_Q[position]._prevQ;
		int newi=device_arr_Q[position]._d_arr_Q[i].idx;
		while (true)
		{			
			printf("\nd_arr_Q[%d]: (prevQ:%d, idx:%d,vid:%d)",prevQ,device_arr_Q[prevQ]._prevQ,device_arr_Q[prevQ]._d_arr_Q[newi].idx,device_arr_Q[prevQ]._d_arr_Q[newi].vid);		
			
			if(device_arr_Q[prevQ]._prevQ==-1) return;
			newi=device_arr_Q[prevQ]._d_arr_Q[i].idx;
			prevQ=device_arr_Q[prevQ]._prevQ;
		}
	}

}

__global__ void kernelGetInformationLastElement(struct_Q *d_arr_Q,int positionLastElement,int *sizeOfLastElement){
	sizeOfLastElement[0]=d_arr_Q[positionLastElement]._size;
	printf("\nsizeOfLastElement:%d",sizeOfLastElement[0]);
}


hipError_t createForwardEmbedding(Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,int li,int lij,int lj){
	hipError_t cudaStatus;

	thrust::device_vector<struct_Embedding*> dVecQ(1);
	thrust::device_vector<struct_Q> dArrayQ(1);
	

	/*//GPU step: Duyệt qua mảng d_ValidExtension và đánh dấu 1 tại những vị trí có cạnh bằng (li,lij,lj) trong mảng M tương ứng
		1. Tạo mảng M có kích thước bằng với d_ValidExtension và gán giá trị ban đầu cho các phần tử trong M bằng 0.
		2. Tạo noElem_d_ValidExtension threads. Mỗi thread sẽ kiểm tra phần tử tương ứng trong mảng d_ValidExtension xem có bằng cạnh (li,lij,lj) 
			Nếu bằng thì bậc vị trí tại M lên giá trị là 1
		3. Exclusive Scan M để thu được vị trí index cũng như kích thước của mảng Q1 và Q2
		4. Tạo mảng Q1 và Q2 có kích thước là (scanM[LastIndex]) nếu phần tử cuối cùng của d_ValidExtension không phải là (li,lij,lj).
			Ngược lại thì Q có kích thước là (scanM[LastIndex]+1). 
			Mỗi phần tử của Q có cấu trúc là {int idx, int vid}
		5. Tạo mảng các cấu trúc Q1 và Q2 với kích thước tìm được đồng thời gán giá trị cho các phần tử của mảng là -1.
		6. Lưu các embedding của cạnh (li,lij,lj) vào Q1 và Q2, cụ thể như sau:
			6.1. vgi vào vid của Q1
			6.2. vgj vào vid của Q2
			6.3. d_scanResult[i] vào idx Q2
		7. Làm sao duyệt qua được tất cả các Embedding khi có Q2?
	*/

	/*1.Tạo mảng M có kích thước bằng với d_ValidExtension và gán giá trị ban đầu cho các phần tử trong M bằng 0.*/
	int* d_M;
	cudaStatus=hipMalloc((int**)&d_M,noElem_d_ValidExtension*sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc M failed");
		exit(1);
	}
	else
	{
		hipMemset(d_M,0,noElem_d_ValidExtension*sizeof(int));
	}

	/*//2. Tạo noElem_d_ValidExtension threads. Mỗi thread sẽ kiểm tra phần tử tương ứng trong mảng d_ValidExtension xem có bằng cạnh (li,lij,lj) 
			Nếu bằng thì bậc vị trí tại M lên giá trị là 1*/
	//printf("\nMang d_ValidExtension");
	//printfExtension(d_ValidExtension,noElem_d_ValidExtension);
	//hipDeviceSynchronize();
	dim3 block(1024);
	dim3 grid((noElem_d_ValidExtension+block.x-1)/block.x);
	
	kernelMarkExtension<<<grid,block>>>(d_ValidExtension,noElem_d_ValidExtension,d_M,li,lij,lj);
	hipDeviceSynchronize();
	printf("\n\nMang d_ValidExtension");
	printfExtension(d_ValidExtension,noElem_d_ValidExtension);
	hipDeviceSynchronize();
	printf("\nMang d-M:");
	printInt(d_M,noElem_d_ValidExtension);

	/* 3. Exclusive Scan d_M
		Kết quả scan lưu vào mảng d_scanResult
	*/
	int* d_scanResult;
	cudaStatus=hipMalloc((int**)&d_scanResult,noElem_d_ValidExtension*sizeof(int));
	if (cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc M failed");
		exit(1);
	}
	else
	{
		hipMemset(d_scanResult,0,noElem_d_ValidExtension*sizeof(int));
	}

	cudaStatus=scanV(d_M,noElem_d_ValidExtension,d_scanResult);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\nscanV() d_M createForwardEmbedding failed");
		exit(1);
	}

	/*
	4. Tạo mảng Q1 và Q2 có kích thước là (scanM[LastIndex]) nếu phần tử cuối cùng của d_ValidExtension không phải là (li,lij,lj).
			Ngược lại thì Q có kích thước là (scanM[LastIndex]+1). 
			Mỗi phần tử của Q có cấu trúc là {int idx, int vid}
	*/
	bool same = false;
	kernelMatchLastElement<<<1,1>>>(d_ValidExtension,noElem_d_ValidExtension,li,lij,lj,same);

	int noElem_d_Q=0;
	
	cudaStatus=getLastElement(d_scanResult,noElem_d_ValidExtension,noElem_d_Q);

	if (same==true){
		noElem_d_Q++;
	}

	
	printf("\nnoElem_d_Q1:%d",noElem_d_Q);

	/*
		5. Tạo mảng các cấu trúc Q1 và Q2 với kích thước tìm được đồng thời gán giá trị cho các phần tử của mảng là -1.
	*/
	struct_Embedding *d_Q1=NULL;
	cudaStatus=hipMalloc((struct_Embedding**)&d_Q1,noElem_d_Q*sizeof(struct_Embedding));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc Embedding failed");
		exit(1);
	}
	else
	{
		hipMemset(d_Q1,-1,noElem_d_Q*sizeof(struct_Embedding));
	}

	struct_Embedding *d_Q2=NULL;
	cudaStatus=hipMalloc((struct_Embedding**)&d_Q2,noElem_d_Q*sizeof(struct_Embedding));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc Embedding failed");
		exit(1);
	}
	else
	{
		hipMemset(d_Q2,-1,noElem_d_Q*sizeof(struct_Embedding));
	}
	
	/*
		6. Lưu các embedding của cạnh (li,lij,lj) vào Q1 và Q2, cụ thể như sau:
			6.1. vgi vào vid của Q1
			6.2. vgj vào vid của Q2
			6.3. d_scanResult[i] vào idx Q2
	*/

	kernelCreateForwardEmbedding<<<grid,block>>>(d_ValidExtension,noElem_d_ValidExtension,d_scanResult,li,lij,lj,d_Q1,d_Q2);
	hipDeviceSynchronize();

	printf("\nEmbedding:\nd_Q1");
	printEmbedding(d_Q1,noElem_d_Q);
	printf("\nd_Q2:");
	printEmbedding(d_Q2,noElem_d_Q);

	//wrap_pointer from raw_pointer to device pointer
	thrust::device_ptr<struct_Embedding> dev_ptr(d_Q1);
	thrust::device_vector<struct_Embedding> Vec(dev_ptr,dev_ptr+noElem_d_Q);
	printf("\nSo luong phan tu cua Vec:%d",Vec.size());
	
	printf("\nd_Q1:%p",d_Q1);
	printf("\ndev_prt:%p",dev_ptr);
	for (int i = 0; i < Vec.size(); i++)
	{
		//printf("\nVec[%d]: (raw_pointer:%p",i,(thrust::raw_pointer_cast(&Vec[i])));
		printf("\nVec[%d]:%p",i,Vec[i]);
	}

	//unwrap pointer from device pointer to raw pointer
	struct_Embedding *raw_ptr = thrust::raw_pointer_cast(&Vec[0]);
	//printf("\nraw_ptr:%p",raw_ptr);
	//printf("\n\nValue of raw_ptr:");
	//printEmbedding(raw_ptr,noElem_d_Q);
	 
	
	//Tạo mảng d_arr_Q, mỗi phần tử của d_arr_Q sẽ trỏ tới địa chỉ của vùng nhớ được trỏ tới bởi d_Q1 và d_Q2
	struct_Embedding **d_arr_Q=NULL;
	cudaStatus=hipMalloc((void**)&d_arr_Q,sizeof(struct_Embedding*)*2);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMalloc d_arr_Q failed");
		exit(1);
	}
	
	//kernelCopy<<<1,1>>>(d_Q1,d_Q2,d_arr_Q); 
	
	cudaStatus=hipMemcpy(d_arr_Q,&d_Q1,sizeof(struct_Embedding*)*1,hipMemcpyHostToDevice);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMemcpy d_arr_Q failed");
		//goto Error;
		exit(1);
	}
	else
	{
		printf("\nCopy successful");
	}
	
	cudaStatus=hipMemcpy(d_arr_Q+1,&(d_Q2),sizeof(struct_Embedding*)*1,hipMemcpyHostToDevice);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMemcpy d_arr_Q failed");
		//goto Error;
		exit(1);
	}
	else
	{
		printf("\nCopy successful");
	}

	//printf("\nd_Q1:%p",d_Q1);
	//printf("\nd_Q2:%p",d_Q2);
	//kernelPrintd_array_Q<<<1,1>>>(d_arr_Q);

	//hipDeviceSynchronize();

	//Copy d_arr_Q to d_arr_new_Q: from device to device memory
	//printf("\n\n d_arr_Q:");
	//kernelPrintArrayEmbedding<<<1,2>>>(d_arr_Q,2,noElem_d_Q);
	
	/*
	//Tạo một mảng mới có kích thước bằng d_arr_Q và sao chép d_arr_Q sang mảng mới
	struct_Embedding **d_arr_new_Q=NULL;
	cudaStatus=hipMalloc((void**)&d_arr_new_Q,sizeof(struct_Embedding*)*2);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaMallocd_arr_new_Q failed");
		//goto Error;
		exit(1);
	}

	
	cudaStatus=hipMemcpy(d_arr_new_Q,d_arr_Q,sizeof(struct_Embedding*)*2,hipMemcpyDeviceToDevice);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\\ncudaMemcpy failed");
		//goto Error;
		exit(1);
	}
	hipDeviceSynchronize();
	printf("\nd_arr_new_Q:");
	kernelPrintd_array_Q<<<1,1>>>(d_arr_new_Q);
	hipFree(d_arr_Q);
	kernelPrintArrayEmbedding<<<1,2>>>(d_arr_new_Q,2,noElem_d_Q);
	*/
	
	//Tạo một mảng device_Array_Q có kiểu là struct_Q
	struct_Q *device_arr_Q=NULL;
	hipMalloc((void**)&device_arr_Q,sizeof(struct_Q)*2);
	//Vì device_arr_Q là một device pointer nên để truy cập phần tử của chúng thì chúng ta cần phải sử dụng kernel
	//Chúng ta tạo kernel để chép dữ liệu từ d_Q1 vào device_array_Q
	int prevQ=-1;	
	int positionUpdate=0;
	kernelcp<<<1,1>>>(device_arr_Q,2,positionUpdate,d_Q1,noElem_d_Q,prevQ);
	//hipMemset(d_Q1,0,sizeof(struct_Embedding)*noElem_d_Q); 
	positionUpdate=1;
	prevQ=0;
	kernelcp<<<1,1>>>(device_arr_Q,2,positionUpdate,d_Q2,noElem_d_Q,prevQ);
	//printf("\n\nPrint device_arr_Q:");
	//printStructQ<<<1,2>>>(device_arr_Q,2);



	printf("\nPrint information of size of the last element of d_arr_Q:");	
	int positionLastElement = 1;
	int *dsizeOfLastElement,*hsizeOfLastElement;
	hsizeOfLastElement=(int*)malloc(sizeof(int));
	hipMalloc((void**)&dsizeOfLastElement,sizeof(int));
	hipMemset(dsizeOfLastElement,0,sizeof(int));
	
	kernelGetInformationLastElement<<<1,1>>>(device_arr_Q,positionLastElement,dsizeOfLastElement);
	hipDeviceSynchronize();
	hipMemcpy(hsizeOfLastElement,dsizeOfLastElement,sizeof(int),hipMemcpyDeviceToHost);
	printf("\nhsizeOfLastElement:%d",hsizeOfLastElement[0]);

		//Làm sao để truy xuất tất cả các Embeddings khi truyền vào một mảng cấu trúc struct_Q: device_arr_Q
	printf("\n\nPrint all embedding from the last element of device_arr_Q");
	PrintAllEmbedding<<<1,hsizeOfLastElement[0]>>>(device_arr_Q,1,hsizeOfLastElement[0]);

	////Làm sao mở rộng kích thước của mảng device_arr_Q
	//struct_Q *device_arr_newQ=NULL;
	//hipMalloc((void**)&device_arr_newQ,sizeof(struct_Q)*3);
	//device_arr_newQ=device_arr_Q;
	////printf("\nDevice array of new Q:");
	////printStructQ<<<1,3>>>(device_arr_newQ,3);
	//kernelcp<<<1,1>>>(device_arr_newQ,3,2,d_Q1,noElem_d_Q);




	//Truy xuat tat ca embedding from d_Q2
	//kernelPrintEmbeddingFromLastQ<<<1,noElem_d_Q>>>(d_arr_Q,1,noElem_d_Q);
	//dVecQ[0]=d_Q1;
	//printf("\ndVecQ[0]:%p",dVecQ[0]);
	//convertDeviceVectorToStruct(dVecQ);
	
	
	
	

	//kernelVector<<<1,1>>>(convertDeviceVectorToStruct(dVecQ),d_Q1,noElem_d_Q);
	//Chuyển dVecQ đang chứa d_Q1 thành struct_Q, sau đó đưa nó vào phần tử đầu tiên của dVecQ là dArrayQ[0]
	//dArrayQ[0]=(convertDeviceVectorToStruct(dVecQ,d_Q1,noElem_d_Q,-1));
	//dArrayQ.resize(2);
	//dArrayQ[1]=(convertDeviceVectorToStruct(dVecQ,d_Q2,noElem_d_Q,1));

	//printf("\ndArrayQ[0]:%p",dArrayQ[0]);
	//printf("\ndArrayQ[1]:%p",dArrayQ[1]);
	//convertToStruct(dArrayQ);
	
	//struct_QQ structQ=convertToStruct(dArrayQ);

	//kernelPrintdArrayQ<<<1,1>>>(structQ);

	//dVecQ.resize(2);
	//dVecQ[1]=d_Q2;


	//hipDeviceSynchronize();
	//cudaStatus=hipGetLastError();
	//if(cudaStatus!=hipSuccess){
	//	fprintf(stderr,"\ncudaDeviceSynchronize() failed");
	//	goto Error;
	//}
//Error:
	//hipFree(d_M);
	//hipFree(d_Q1);
	//hipFree(d_Q2);
	return cudaStatus;
}
