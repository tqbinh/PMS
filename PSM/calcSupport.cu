#include "hip/hip_runtime.h"
#include "calcSupport.h"
#include "kernelPrintf.h"

//__device__ int li,lij,lj;


__global__ void kernelCalcSupport(int li,int lij,int lj,Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,int *d_scanB_Result,int *d_F){
	int i= blockIdx.x*blockDim.x + threadIdx.x;
	if(i<noElem_d_ValidExtension){
		if(d_ValidExtension[i].li==li && d_ValidExtension[i].lij==lij && d_ValidExtension[i].lj==lj){
			int index=d_scanB_Result[i];
			d_F[index]=1;
		}		
	}
}



hipError_t calcSupport(Extension *d_UniqueExtension,unsigned int noElem_d_UniqueExtension,Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,int *d_scanB_Result,int *d_F,unsigned int noElem_F){
	hipError_t cudaStatus;

	dim3 block(1024);
	dim3 grid((noElem_d_ValidExtension+block.x-1)/block.x);

	//chép dữ liệu của mảng d_UniqueExtension sang host

	Extension *h_UniqueExtension;
	h_UniqueExtension = new Extension[noElem_d_UniqueExtension];
	if(h_UniqueExtension==NULL){
		printf("\n!!!Memory Problem h_UniqueExtension");
		exit(1);
	}else{
		memset(h_UniqueExtension,0, noElem_d_UniqueExtension*sizeof(Extension));
	}

	hipMemcpy(h_UniqueExtension,d_UniqueExtension,noElem_d_UniqueExtension*sizeof(Extension),hipMemcpyDeviceToHost);

	for (int i=0;i<noElem_d_UniqueExtension;i++){	
		int li,lij,lj;
		li=h_UniqueExtension[i].li;
		lij=h_UniqueExtension[i].lij;
		lj=h_UniqueExtension[i].lj;		
			
		kernelCalcSupport<<<grid,block>>>(li,lij,lj,d_ValidExtension,noElem_d_ValidExtension,d_scanB_Result,d_F);
		hipDeviceSynchronize();
		printf("\n[%d] d_F:",i);
		printInt(d_F,noElem_F);
		hipMemset(d_F,0,noElem_F*sizeof(int));
	}
		
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaDeviceSynchronize function failed");
		goto Error;
	}

Error:
	return cudaStatus;
}
