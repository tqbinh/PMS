#include "hip/hip_runtime.h"
#include "calcSupport.h"
#include "kernelPrintf.h"




__global__ void kernelCalcSupport(int li,int lij,int lj,Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,int *d_scanB_Result,float *d_F){
	int i= blockIdx.x*blockDim.x + threadIdx.x;
	if(i<noElem_d_ValidExtension){
		if(d_ValidExtension[i].li==li && d_ValidExtension[i].lij==lij && d_ValidExtension[i].lj==lj){
			int index=d_scanB_Result[i];
			d_F[index]=1;
		}		
	}
}



hipError_t calcSupport(Extension *d_UniqueExtension,unsigned int noElem_d_UniqueExtension,Extension *d_ValidExtension,unsigned int noElem_d_ValidExtension,int *d_scanB_Result,float *d_F,unsigned int noElem_F,unsigned int minsup,int *d_O,int *d_LO,int numberOfElementd_O,int *d_N,int *d_LN,int numberOfElementd_N,unsigned int Lv,unsigned int Le,unsigned int maxOfVer,unsigned int numberOfGraph,unsigned int noDeg,vector<int> &h_satisfyEdge,vector<int> &h_satisfyEdgeSupport){
	hipError_t cudaStatus;

	dim3 block(blocksize);
	dim3 grid((noElem_d_ValidExtension+block.x-1)/block.x);

	//chép dữ liệu của mảng d_UniqueExtension sang host

	Extension *h_UniqueExtension;
	h_UniqueExtension = new Extension[noElem_d_UniqueExtension];
	if(h_UniqueExtension==NULL){
		printf("\n!!!Memory Problem h_UniqueExtension");
		exit(1);
	}else{
		memset(h_UniqueExtension,0, noElem_d_UniqueExtension*sizeof(Extension));
	}

	hipMemcpy(h_UniqueExtension,d_UniqueExtension,noElem_d_UniqueExtension*sizeof(Extension),hipMemcpyDeviceToHost);
	

	for (int i=0;i<noElem_d_UniqueExtension;i++){	
		int li,lij,lj;
		li=h_UniqueExtension[i].li;
		lij=h_UniqueExtension[i].lij;
		lj=h_UniqueExtension[i].lj;		
			
		kernelCalcSupport<<<grid,block>>>(li,lij,lj,d_ValidExtension,noElem_d_ValidExtension,d_scanB_Result,d_F);
		hipDeviceSynchronize();
		printf("\n[%d] d_F:",i);
		printFloat(d_F,noElem_F);
		float support=0;
		reduction(d_F,noElem_F,support);
		//printf("  Support:%.0f\n",support);

		//Kiểm tra xem độ hỗ trợ có thoả minsup hay không? 
		//Nếu thoả minsup thì kiểm tra xem pattern P có phải là nhỏ nhất hay không? (Đây là hoạt động tuần tự được thực thi trên CPU)
		//Nếu là nhỏ nhất thì mới đi tạo embedding cho pattern P.
		if(support>=minsup){
			//1. Nếu độ hỗ trợ thoả minSup thì xây dựng DFS_Code cho cạnh đó --> cần phải thoả cấu trúc (vi,vj,Li,Lij,Lj)
			//Xây dựng DFS_Code trên device hay trên host? --> xây dựng DFS_Code trên host vì quá trình minDFS_Code diễn tra trên CPU chứ không phải GPU
			//h_frequentEdge[i]=1;
			h_satisfyEdge.push_back(i);		
			h_satisfyEdgeSupport.push_back(support);
			
			//xây dựng embedding cho mở rộng thoả minsup
			//printf("\n***********support of (%d,%d,%d) >= %d --> create embeddings for DFS_CODE************",li,lij,lj,minsup);
			/*cudaStatus=createForwardEmbedding(d_ValidExtension,noElem_d_ValidExtension,li,lij,lj,d_O,d_LO,numberOfElementd_O,d_N,d_LN,numberOfElementd_N,Lv,Le,minsup,maxOfVer,numberOfGraph,noDeg);
			if (cudaStatus!=hipSuccess){
				fprintf(stderr,"\ncreateForwardEmbedding failed");
				goto Error;
			}*/
		}
		
		hipMemset(d_F,0,noElem_F*sizeof(int));
	}
		
	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaDeviceSynchronize function failed");
		goto Error;
	}

Error:
	return cudaStatus;
}


__global__ void kernelGetGraphIdContainEmbedding(int *d_arr_edgeLabel,Extension *d_ValidExtension,int noElem_d_ValidExtension,int *d_arr_graphIdContainEmbedding,unsigned int maxOfVer){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i<noElem_d_ValidExtension){
		if(	d_ValidExtension[i].li == d_arr_edgeLabel[0] && d_ValidExtension[i].lij == d_arr_edgeLabel[1] && 	d_ValidExtension[i].lj == d_arr_edgeLabel[2]){
			int graphid = (d_ValidExtension[i].vgi/maxOfVer);
			d_arr_graphIdContainEmbedding[graphid]=1;
		}
	}
}


__global__ void kernelGetGraph(int *d_arr_graphIdContainEmbedding,int noEle_d_arr_graphIdContainEmbedding,int *d_kq,int *d_scanResult){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i<noEle_d_arr_graphIdContainEmbedding){
		if(d_arr_graphIdContainEmbedding[i]!=0){
			d_kq[d_scanResult[i]]=i;
		}
	}
}


inline hipError_t getGraphIdContainEmbedding(int *d_arr_edgeLabel,Extension *d_ValidExtension,int noElem_d_ValidExtension,int *&h_graphIdContainEmbedding,int &noElem_h_arr_graphIdContainEmbedding,unsigned int maxOfVer){
	hipError_t cudaStatus;

	//Từ global id của đỉnh (vgi hoặc vgj) trong d_ValidExtension chúng ta sẽ tính được graphID chứa mở rộng đó.
	//Các mở rộng trong d_validExtension đã được sắp xếp theo thứ tự từ graphID 0 đến graphId cuối cùng một cách tự nhiên
	//Cần có noElem_d_ValidExtension threads để thực hiện so sánh với nhãn (li,lij,lj), nếu bằng nhau thì sẽ tính graphID=(vgi/maxOfVer) của nó.
	//Set giá trị của mảng d_arr_graphid[graphID]=1;
	//scan mảng d_arr_graphID để thu được index
	//Duyệt qua mảng d_arr_graphid, tại vị trí nào bằng 1 thì ghi giá trị i vào index trong mảng index vừa tính được
	//copy mảng này bỏ vào mảng kết quả.
	dim3 block(blocksize);
	dim3 grid((noElem_d_ValidExtension+block.x)/block.x);
	int noEle_d_arr_graphIdContainEmbedding;
	getLastElementExtension(d_ValidExtension,noElem_d_ValidExtension,noEle_d_arr_graphIdContainEmbedding,maxOfVer);
	noEle_d_arr_graphIdContainEmbedding++;
	//printf("\n noEle_d_arr_graphIdContainEmbedding: %d",noEle_d_arr_graphIdContainEmbedding);

	int *d_arr_graphIdContainEmbedding=NULL;
	cudaStatus=hipMalloc((void**)&d_arr_graphIdContainEmbedding,noEle_d_arr_graphIdContainEmbedding*sizeof(int));
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMalloc d_arr_graphIdContainEmbedding failed");
		goto Error;
	}
	else
	{
		hipMemset(d_arr_graphIdContainEmbedding,0,noEle_d_arr_graphIdContainEmbedding*sizeof(int));
	}

	//Gọi hàm kernelGetGraphIdContainEmbedding để đánh dấu vị trí đồ thị trong mảng d_arr_graphIdContainEmbedding là 1  
	kernelGetGraphIdContainEmbedding<<<grid,block>>>(d_arr_edgeLabel,d_ValidExtension,noElem_d_ValidExtension,d_arr_graphIdContainEmbedding,maxOfVer);
	hipDeviceSynchronize();
	printf("\n*************d_arr_graphIdContainEmbedding***************\n");
	printInt(d_arr_graphIdContainEmbedding,noEle_d_arr_graphIdContainEmbedding);

	int *d_scanResult;
	hipMalloc((void**)&d_scanResult,sizeof(int)*noEle_d_arr_graphIdContainEmbedding);

	scanV(d_arr_graphIdContainEmbedding,noEle_d_arr_graphIdContainEmbedding,d_scanResult);

	printf("\n ************* d_scanResult *************\n");
	printInt(d_scanResult,noEle_d_arr_graphIdContainEmbedding);

	int noElem_kq;	
	getLastElement(d_scanResult,noEle_d_arr_graphIdContainEmbedding,noElem_kq);
	noElem_kq++;

	int *d_kq;
	hipMalloc((void**)&d_kq,sizeof(int)*noElem_kq);
	
	dim3 blocka(blocksize);
	dim3 grida((noEle_d_arr_graphIdContainEmbedding + blocka.x -1)/blocka.x);

	kernelGetGraph<<<grida,blocka>>>(d_arr_graphIdContainEmbedding,noEle_d_arr_graphIdContainEmbedding,d_kq,d_scanResult);
	hipDeviceSynchronize();

	printf("\n*********** d_kq ***********\n");
	printInt(d_kq,noElem_kq);

	h_graphIdContainEmbedding=(int*)malloc(sizeof(int)*noElem_kq);
	if(h_graphIdContainEmbedding==NULL){
		printf("\nMalloc h_graphIdContainEmbedding failed");
		exit(1);
	}


	cudaStatus = hipMemcpy(h_graphIdContainEmbedding,d_kq,sizeof(int)*noElem_kq,hipMemcpyDeviceToHost);
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\n hipMemcpy failed",cudaStatus);
		goto Error;
	}

	noElem_h_arr_graphIdContainEmbedding = noElem_kq;

	hipDeviceSynchronize();
	cudaStatus=hipGetLastError();
	if(cudaStatus!=hipSuccess){
		fprintf(stderr,"\ncudaDeviceSynchronize function failed");
		goto Error;
	}

Error:
	return cudaStatus;
}
